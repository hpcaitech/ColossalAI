/*This code from NVIDIA Megatron:
 *     with minor changes. */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "scaled_masked_softmax.h"
#include "type_shim.h"

namespace multihead_attn {
namespace fused_softmax {
namespace scaled_masked_softmax {

int get_batch_per_block_cuda(int query_seq_len, int key_seq_len, int batches,
                             int attn_heads) {
  return get_batch_per_block(query_seq_len, key_seq_len, batches, attn_heads);
}

torch::Tensor fwd_cuda(torch::Tensor const& input, torch::Tensor const& mask,
                       float scale_factor) {
  // input is a 4d tensor with dimensions [batches, attn_heads, seq_len,
  // seq_len]
  const int batches = input.size(0);
  const int pad_batches = mask.size(0);
  const int attn_heads = input.size(1);
  const int query_seq_len = input.size(2);
  const int key_seq_len = input.size(3);
  TORCH_INTERNAL_ASSERT(key_seq_len <= 2048);
  TORCH_INTERNAL_ASSERT(query_seq_len > 1);
  TORCH_INTERNAL_ASSERT(pad_batches == 1 || pad_batches == batches);
  TORCH_INTERNAL_ASSERT(mask.size(1) == 1);
  TORCH_INTERNAL_ASSERT(mask.size(2) == query_seq_len);
  TORCH_INTERNAL_ASSERT(mask.size(3) == key_seq_len);

  // Output
  auto act_options = input.options().requires_grad(false);
  torch::Tensor softmax_results = torch::empty(
      {batches, attn_heads, query_seq_len, key_seq_len}, act_options);

  // Softmax Intermediate Result Ptr
  void* input_ptr = static_cast<void*>(input.data_ptr());
  void* mask_ptr = static_cast<void*>(mask.data_ptr());
  void* softmax_results_ptr = static_cast<void*>(softmax_results.data_ptr());

  DISPATCH_HALF_AND_BFLOAT(
      input.scalar_type(), "dispatch_scaled_masked_softmax_forward",
      dispatch_scaled_masked_softmax_forward<scalar_t, scalar_t, float>(
          reinterpret_cast<scalar_t*>(softmax_results_ptr),
          reinterpret_cast<const scalar_t*>(input_ptr),
          reinterpret_cast<const uint8_t*>(mask_ptr), scale_factor,
          query_seq_len, key_seq_len, batches, attn_heads, pad_batches););
  return softmax_results;
}

torch::Tensor bwd_cuda(torch::Tensor const& output_grads_,
                       torch::Tensor const& softmax_results_,
                       float scale_factor) {
  auto output_grads = output_grads_.contiguous();
  auto softmax_results = softmax_results_.contiguous();

  // output grads is a 4d tensor with dimensions [batches, attn_heads, seq_len,
  // seq_len]
  const int batches = output_grads.size(0);
  const int attn_heads = output_grads.size(1);
  const int query_seq_len = output_grads.size(2);
  const int key_seq_len = output_grads.size(3);

  void* output_grads_ptr = static_cast<void*>(output_grads.data_ptr());

  // Softmax Grad
  DISPATCH_HALF_AND_BFLOAT(
      output_grads_.scalar_type(), "dispatch_scaled_masked_softmax_backward",
      dispatch_scaled_masked_softmax_backward<scalar_t, scalar_t, float>(
          reinterpret_cast<scalar_t*>(output_grads_ptr),
          reinterpret_cast<scalar_t*>(output_grads_ptr),
          reinterpret_cast<scalar_t const*>(softmax_results.data_ptr()),
          scale_factor, query_seq_len, key_seq_len, batches, attn_heads););

  // backward pass is completely in-place
  return output_grads;
}
}  // namespace scaled_masked_softmax
}  // namespace fused_softmax
}  // namespace multihead_attn

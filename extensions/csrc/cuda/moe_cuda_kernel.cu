#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <torch/extension.h>

#include <hipcub/hipcub.hpp>

#include "block_reduce.h"

template <typename T, int block_size, int pack_size>
__device__ void moe_dpch_one_fwd(T *src_row, T *dst_row, const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T pack[pack_size];
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(src_row + idx, pack);
    BlockStore(ts_store).Store(dst_row + idx, pack);
  }
}

template <typename T, int block_size, int pack_size>
__device__ void moe_dpch_one_bwd(T *src_row, T *dst_row, const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T pack[pack_size];
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(dst_row + idx, pack);
    BlockStore(ts_store).Store(src_row + idx, pack);
  }
}

template <typename T, int block_size, int pack_size>
__device__ void moe_dpch_two_fwd(T *src_row, T *dst_row1, T *dst_row2,
                                 const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T pack[pack_size];
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(src_row + idx, pack);
    BlockStore(ts_store).Store(dst_row1 + idx, pack);
    BlockStore(ts_store).Store(dst_row2 + idx, pack);
  }
}

template <typename T, int block_size, int pack_size>
__device__ void moe_dpch_two_bwd(T *src_row, T *dst_row1, T *dst_row2,
                                 const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T pack1[pack_size], pack2[pack_size];
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(dst_row1 + idx, pack1);
    BlockLoad(ts_load).Load(dst_row2 + idx, pack2);

#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      pack1[i] += pack2[i];
    }

    BlockStore(ts_store).Store(src_row + idx, pack1);
  }
}

template <typename T, int block_size, int pack_size>
__device__ void moe_cb_one_fwd(T *src_row, T *dst_row, const T weight,
                               const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T pack[pack_size];
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(src_row + idx, pack);

#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      pack[i] *= weight;
    }

    BlockStore(ts_store).Store(dst_row + idx, pack);
  }
}

template <typename T, int block_size, int pack_size>
__device__ void moe_cb_one_bwd(T *src_row, T *dst_row, T *tks_row,
                               T *weight_grad, const T weight, const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T grad[pack_size], tokens[pack_size];
  float thread_sum = 0;
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(dst_row + idx, grad);
    BlockLoad(ts_load).Load(tks_row + idx, tokens);

#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      thread_sum += grad[i] * tokens[i];
      grad[i] *= weight;
    }

    BlockStore(ts_store).Store(src_row + idx, grad);
  }

  blockReduce<ReduceType::kSum, 1>(&thread_sum);

  if (threadIdx.x == 0) *weight_grad = static_cast<T>(thread_sum);
}

template <typename T, int block_size, int pack_size>
__device__ void moe_cb_two_fwd(T *src_row1, T *src_row2, T *dst_row,
                               const T weight1, const T weight2,
                               const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T pack1[pack_size], pack2[pack_size];
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(src_row1 + idx, pack1);
    BlockLoad(ts_load).Load(src_row2 + idx, pack2);

#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      pack1[i] = pack1[i] * weight1 + pack2[i] * weight2;
    }

    BlockStore(ts_store).Store(dst_row + idx, pack1);
  }
}

template <typename T, int block_size, int pack_size>
__device__ void moe_cb_two_bwd(T *src_row1, T *src_row2, T *dst_row,
                               T *tks_row1, T *tks_row2, T *weight_grad1,
                               T *weight_grad2, const T weight1,
                               const T weight2, const int cols) {
  assert(cols % pack_size == 0);
  const int bpack_size = block_size * pack_size;

  typedef hipcub::BlockLoad<T, block_size, pack_size, hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;

  typedef hipcub::BlockStore<T, block_size, pack_size, hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  int tps = threadIdx.x * pack_size;
  T grad[pack_size], tokens1[pack_size], tokens2[pack_size], sgrad1[pack_size],
      sgrad2[pack_size];
  float thread_sum[2] = {0, 0};
  for (int idx = 0; idx + tps < cols; idx += bpack_size) {
    BlockLoad(ts_load).Load(dst_row + idx, grad);
    BlockLoad(ts_load).Load(tks_row1 + idx, tokens1);
    BlockLoad(ts_load).Load(tks_row2 + idx, tokens2);

#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      thread_sum[0] += grad[i] * tokens1[i];
      thread_sum[1] += grad[i] * tokens2[i];
      sgrad1[i] = weight1 * grad[i];
      sgrad2[i] = weight2 * grad[i];
    }

    BlockStore(ts_store).Store(src_row1 + idx, sgrad1);
    BlockStore(ts_store).Store(src_row2 + idx, sgrad2);
  }

  blockReduce<ReduceType::kSum, 2>(thread_sum);

  if (threadIdx.x == 0)
    *weight_grad1 = static_cast<T>(thread_sum[0]);
  else if (threadIdx.x == 1)
    *weight_grad2 = static_cast<T>(thread_sum[1]);
}

// DISPATCH KERNELS --------------------------------

template <typename T, int block_size, int pack_size>
__device__ void moe_dpch_fwd_selector(T *src_row, T *dst_row1, T *dst_row2,
                                      const int cols, const int indicator1,
                                      const int indicator2) {
  if (indicator1 != 0 && indicator2 != 0)
    moe_dpch_two_fwd<T, block_size, pack_size>(src_row, dst_row1, dst_row2,
                                               cols);
  else if (indicator1 != 0)
    moe_dpch_one_fwd<T, block_size, pack_size>(src_row, dst_row1, cols);
  else if (indicator2 != 0)
    moe_dpch_one_fwd<T, block_size, pack_size>(src_row, dst_row2, cols);
  else
    return;
}

template <typename T, int block_size, int pack_size>
__device__ void moe_dpch_bwd_selector(T *src_row, T *dst_row1, T *dst_row2,
                                      const int cols, const int indicator1,
                                      const int indicator2) {
  if (indicator1 != 0 && indicator2 != 0)
    moe_dpch_two_bwd<T, block_size, pack_size>(src_row, dst_row1, dst_row2,
                                               cols);
  else if (indicator1 != 0)
    moe_dpch_one_bwd<T, block_size, pack_size>(src_row, dst_row1, cols);
  else if (indicator2 != 0)
    moe_dpch_one_bwd<T, block_size, pack_size>(src_row, dst_row2, cols);
  else
    return;
}

template <typename T, int block_size, int pack_size>
__global__ void moe_dpch_fwd_kernel(T *batch_tokens, T *expert_input,
                                    int *mask1, int *mask2, int *dest1,
                                    int *dest2, const int h) {
  int row = blockIdx.x;
  int indicator2 = mask2 == nullptr ? 0 : mask2[row];
  moe_dpch_fwd_selector<T, block_size, pack_size>(
      batch_tokens + (row * h), expert_input + (dest1[row] * h),
      expert_input + (dest2[row] * h), h, mask1[row], indicator2);
}

template <typename T, int block_size, int pack_size>
__global__ void moe_dpch_bwd_kernel(T *tokens_grad, T *expert_grad, int *mask1,
                                    int *mask2, int *dest1, int *dest2,
                                    const int h) {
  int row = blockIdx.x;
  int indicator2 = mask2 == nullptr ? 0 : mask2[row];
  moe_dpch_bwd_selector<T, block_size, pack_size>(
      tokens_grad + (row * h), expert_grad + (dest1[row] * h),
      expert_grad + (dest2[row] * h), h, mask1[row], indicator2);
}

// COMBINE KERNELS --------------------------------

template <typename T, int block_size, int pack_size>
__device__ void moe_cb_fwd_selector(T *src_row1, T *src_row2, T *dst_row,
                                    const int cols, const T weight1,
                                    const T weight2, const int indicator1,
                                    const int indicator2) {
  if (indicator1 != 0 && indicator2 != 0)
    moe_cb_two_fwd<T, block_size, pack_size>(src_row1, src_row2, dst_row,
                                             weight1, weight2, cols);
  else if (indicator1 != 0)
    moe_cb_one_fwd<T, block_size, pack_size>(src_row1, dst_row, weight1, cols);
  else if (indicator2 != 0)
    moe_cb_one_fwd<T, block_size, pack_size>(src_row2, dst_row, weight2, cols);
  else
    return;
}

template <typename T, int block_size, int pack_size>
__device__ void moe_cb_bwd_selector(T *src_row1, T *src_row2, T *dst_row,
                                    const int cols, T *tks_row1, T *tks_row2,
                                    T *wt_grad1, T *wt_grad2, const T weight1,
                                    const T weight2, const int indicator1,
                                    const int indicator2) {
  if (indicator1 != 0 && indicator2 != 0)
    moe_cb_two_bwd<T, block_size, pack_size>(src_row1, src_row2, dst_row,
                                             tks_row1, tks_row2, wt_grad1,
                                             wt_grad2, weight1, weight2, cols);
  else if (indicator1 != 0)
    moe_cb_one_bwd<T, block_size, pack_size>(src_row1, dst_row, tks_row1,
                                             wt_grad1, weight1, cols);
  else if (indicator2 != 0)
    moe_cb_one_bwd<T, block_size, pack_size>(src_row2, dst_row, tks_row2,
                                             wt_grad2, weight2, cols);
  else
    return;
}

template <typename T, int block_size, int pack_size>
__global__ void moe_cb_fwd_kernel(T *expert_tokens, T *combine_tokens,
                                  T *logits, int *mask1, int *mask2, int *dest1,
                                  int *dest2, const int e, const int c,
                                  const int h) {
  int row = blockIdx.x, eid1 = dest1[row] / c, eid2 = dest2[row] / c;
  int indicator2 = mask2 == nullptr ? 0 : mask2[row];
  T *row_log = logits + (row * e);
  moe_cb_fwd_selector<T, block_size, pack_size>(
      expert_tokens + (dest1[row] * h), expert_tokens + (dest2[row] * h),
      combine_tokens + (row * h), h, row_log[eid1], row_log[eid2], mask1[row],
      indicator2);
}

template <typename T, int block_size, int pack_size>
__global__ void moe_cb_bwd_kernel(T *tokens_grad, T *expert_grad, T *tks,
                                  T *logits, T *logits_grad, int *mask1,
                                  int *mask2, int *dest1, int *dest2,
                                  const int e, const int c, const int h) {
  int row = blockIdx.x, eid1 = dest1[row] / c, eid2 = dest2[row] / c;
  int indicator2 = mask2 == nullptr ? 0 : mask2[row];
  T *row_log = logits + (row * e), *row_grad = logits_grad + (row * e);
  moe_cb_bwd_selector<T, block_size, pack_size>(
      expert_grad + (dest1[row] * h), expert_grad + (dest2[row] * h),
      tokens_grad + (row * h), h, tks + (dest1[row] * h),
      tks + (dest2[row] * h), row_grad + eid1, row_grad + eid2, row_log[eid1],
      row_log[eid2], mask1[row], indicator2);
}

// CUMSUM KERNEL --------------------------------

template <int block_size, int pack_size>
__global__ void cumsum_kernel(int *inputs, int *outputs, const int s,
                              const int e) {
  assert(s % pack_size == 0);
  constexpr int bpack_size = block_size * pack_size;
  int tid = threadIdx.x, bid = blockIdx.x, tps = tid * pack_size, last_sum = -1;
  __shared__ int temp[block_size + 1];
  int pack[pack_size];

  for (int idx = 0; idx < s; idx += bpack_size) {
    int offset = 1;

    if (idx + tps < s) {
      temp[tid] = inputs[tps * e + bid];
#pragma unroll
      for (int i = 1; i < pack_size; ++i) {
        pack[i] = inputs[(tps + i) * e + bid];
      }
#pragma unroll
      for (int i = 1; i < pack_size; ++i) {
        temp[tid] += pack[i];
      }
    }

    for (int i = block_size >> 1; i > 0; i >>= 1) {
      __syncthreads();
      if (tid < i) {
        int j = offset * (2 * tid + 1) - 1;
        temp[j + offset] += temp[j];
      }
      offset <<= 1;
    }

    if (tid == 0) {
      temp[block_size] = temp[block_size - 1];
      temp[block_size - 1] = 0;
    }

    for (int i = 1; i < block_size; i <<= 1) {
      offset >>= 1;
      __syncthreads();
      if (tid < i) {
        int j = offset * (2 * tid + 1) - 1, k = j + offset, ts = temp[j];
        temp[j] = temp[k];
        temp[k] += ts;
      }
    }
    __syncthreads();

    if (tid == 0) temp[0] = temp[block_size];
    __syncthreads();

    if (idx + tps < s) {
      temp[tid + 1] += last_sum;
#pragma unroll
      for (int i = pack_size - 1; i > 0; --i) {
        outputs[(tps + i) * e + bid] = temp[tid + 1];
        temp[tid + 1] -= pack[i];
      }
      outputs[tps * e + bid] = temp[tid + 1];
    }
    __syncthreads();

    last_sum += temp[0];
    inputs += bpack_size * e;
    outputs += bpack_size * e;
  }
}

// LAUNCH FUNCTIONS --------------------------------

template <typename T>
void moe_dpch_fwd_launch(T *batch_tokens, T *expert_input, int *mask1,
                         int *mask2, int *dest1, int *dest2, const int s,
                         const int h) {
  if (h < 256)
    moe_dpch_fwd_kernel<T, 32, 4>
        <<<s, 32>>>(batch_tokens, expert_input, mask1, mask2, dest1, dest2, h);
  else if (h < 512)
    moe_dpch_fwd_kernel<T, 32, 8>
        <<<s, 32>>>(batch_tokens, expert_input, mask1, mask2, dest1, dest2, h);
  else if (h < 1024)
    moe_dpch_fwd_kernel<T, 32, 16>
        <<<s, 32>>>(batch_tokens, expert_input, mask1, mask2, dest1, dest2, h);
  else if (h < 2048)
    moe_dpch_fwd_kernel<T, 64, 16>
        <<<s, 64>>>(batch_tokens, expert_input, mask1, mask2, dest1, dest2, h);
  else
    moe_dpch_fwd_kernel<T, 128, 16>
        <<<s, 128>>>(batch_tokens, expert_input, mask1, mask2, dest1, dest2, h);
}

template <typename T>
void moe_dpch_bwd_launch(T *tokens_grad, T *expert_grad, int *mask1, int *mask2,
                         int *dest1, int *dest2, const int s, const int h) {
  if (h < 256)
    moe_dpch_bwd_kernel<T, 32, 4>
        <<<s, 32>>>(tokens_grad, expert_grad, mask1, mask2, dest1, dest2, h);
  else if (h < 512)
    moe_dpch_bwd_kernel<T, 32, 8>
        <<<s, 32>>>(tokens_grad, expert_grad, mask1, mask2, dest1, dest2, h);
  else if (h < 1024)
    moe_dpch_bwd_kernel<T, 32, 16>
        <<<s, 32>>>(tokens_grad, expert_grad, mask1, mask2, dest1, dest2, h);
  else if (h < 2048)
    moe_dpch_bwd_kernel<T, 64, 16>
        <<<s, 64>>>(tokens_grad, expert_grad, mask1, mask2, dest1, dest2, h);
  else
    moe_dpch_bwd_kernel<T, 128, 16>
        <<<s, 128>>>(tokens_grad, expert_grad, mask1, mask2, dest1, dest2, h);
}

template <typename T>
void moe_cb_fwd_launch(T *expert_tokens, T *combine_tokens, T *logits,
                       int *mask1, int *mask2, int *dest1, int *dest2,
                       const int s, const int e, const int c, const int h) {
  if (h < 256)
    moe_cb_fwd_kernel<T, 32, 4><<<s, 32>>>(expert_tokens, combine_tokens,
                                           logits, mask1, mask2, dest1, dest2,
                                           e, c, h);
  else if (h < 512)
    moe_cb_fwd_kernel<T, 32, 8><<<s, 32>>>(expert_tokens, combine_tokens,
                                           logits, mask1, mask2, dest1, dest2,
                                           e, c, h);
  else if (h < 1024)
    moe_cb_fwd_kernel<T, 32, 16><<<s, 32>>>(expert_tokens, combine_tokens,
                                            logits, mask1, mask2, dest1, dest2,
                                            e, c, h);
  else if (h < 2048)
    moe_cb_fwd_kernel<T, 64, 16><<<s, 64>>>(expert_tokens, combine_tokens,
                                            logits, mask1, mask2, dest1, dest2,
                                            e, c, h);
  else
    moe_cb_fwd_kernel<T, 128, 16><<<s, 128>>>(expert_tokens, combine_tokens,
                                              logits, mask1, mask2, dest1,
                                              dest2, e, c, h);
}

template <typename T>
void moe_cb_bwd_launch(T *tokens_grad, T *expert_grad, T *tks, T *logits,
                       T *logits_grad, int *mask1, int *mask2, int *dest1,
                       int *dest2, const int s, const int e, const int c,
                       const int h) {
  if (h < 256)
    moe_cb_bwd_kernel<T, 32, 4><<<s, 32>>>(tokens_grad, expert_grad, tks,
                                           logits, logits_grad, mask1, mask2,
                                           dest1, dest2, e, c, h);
  else  // if (h < 512)
    moe_cb_bwd_kernel<T, 64, 4><<<s, 64>>>(tokens_grad, expert_grad, tks,
                                           logits, logits_grad, mask1, mask2,
                                           dest1, dest2, e, c, h);
  // else if (h < 1024)
  //     moe_cb_bwd_kernel<T, 128, 4><<<s, 128>>>
  //         (tokens_grad, expert_grad, tks, logits, logits_grad, mask1, mask2,
  //         dest1, dest2, e, c, h);
  // else
  //     moe_cb_bwd_kernel<T, 256, 4><<<s, 256>>>
  //         (tokens_grad, expert_grad, tks, logits, logits_grad, mask1, mask2,
  //         dest1, dest2, e, c, h);
}

void cumsum_launch(int *inputs, int *outputs, const int s, const int e) {
  if (s <= 256)
    cumsum_kernel<256, 1><<<e, 256>>>(inputs, outputs, s, e);
  else if (s <= 512)
    cumsum_kernel<512, 1><<<e, 512>>>(inputs, outputs, s, e);
  else if (s <= 1024)
    cumsum_kernel<1024, 1><<<e, 1024>>>(inputs, outputs, s, e);
  else if (s <= 2048)
    cumsum_kernel<1024, 2><<<e, 1024>>>(inputs, outputs, s, e);
  else
    cumsum_kernel<1024, 4><<<e, 1024>>>(inputs, outputs, s, e);
}

// API FUNCTIONS --------------------------------

#define DISPATCH_FLOAT_AND_HALF(TYPE, NAME, ...)                       \
  switch (TYPE) {                                                      \
    case at::ScalarType::Float: {                                      \
      using scalar_t = float;                                          \
      __VA_ARGS__;                                                     \
      break;                                                           \
    }                                                                  \
    case at::ScalarType::Half: {                                       \
      using scalar_t = at::Half;                                       \
      __VA_ARGS__;                                                     \
      break;                                                           \
    }                                                                  \
    default:                                                           \
      AT_ERROR(#NAME, " not implemented yet for specific data type."); \
  }

torch::Tensor moe_dispatch_cuda_forward(int s, int ec, int h,
                                        torch::Tensor batch_tokens,
                                        torch::Tensor mask,
                                        torch::Tensor dest_idx) {
  assert(h % 16 == 0);
  auto res = torch::zeros(
      {ec, h},
      torch::dtype(batch_tokens.dtype()).device(batch_tokens.device()));
  auto k = mask.size(0);

  DISPATCH_FLOAT_AND_HALF(
      batch_tokens.scalar_type(), "moe dispatch forward",
      moe_dpch_fwd_launch<scalar_t>(
          batch_tokens.data<scalar_t>(), res.data<scalar_t>(),
          mask[0].data<int>(), k == 1 ? nullptr : mask[1].data<int>(),
          dest_idx[0].data<int>(),
          k == 1 ? dest_idx[0].data<int>() : dest_idx[1].data<int>(), s, h));

  return res;
}

torch::Tensor moe_dispatch_cuda_backward(int s, int ec, int h,
                                         torch::Tensor expert_grad,
                                         torch::Tensor mask,
                                         torch::Tensor dest_idx) {
  assert(h % 16 == 0);
  auto res = torch::zeros(
      {s, h}, torch::dtype(expert_grad.dtype()).device(expert_grad.device()));
  auto k = mask.size(0);

  DISPATCH_FLOAT_AND_HALF(
      expert_grad.scalar_type(), "moe dispatch backward",
      moe_dpch_bwd_launch<scalar_t>(
          res.data<scalar_t>(), expert_grad.data<scalar_t>(),
          mask[0].data<int>(), k == 1 ? nullptr : mask[1].data<int>(),
          dest_idx[0].data<int>(),
          k == 1 ? dest_idx[0].data<int>() : dest_idx[1].data<int>(), s, h));

  return res;
}

torch::Tensor moe_combine_cuda_forward(int s, int e, int c, int h,
                                       torch::Tensor expert_tokens,
                                       torch::Tensor logits, torch::Tensor mask,
                                       torch::Tensor dest_idx) {
  assert(h % 16 == 0);
  assert(expert_tokens.dtype() == logits.dtype());

  auto res = torch::zeros(
      {s, h},
      torch::dtype(expert_tokens.dtype()).device(expert_tokens.device()));
  auto k = mask.size(0);

  DISPATCH_FLOAT_AND_HALF(
      expert_tokens.scalar_type(), "moe combine forward",
      moe_cb_fwd_launch<scalar_t>(
          expert_tokens.data<scalar_t>(), res.data<scalar_t>(),
          logits.data<scalar_t>(), mask[0].data<int>(),
          k == 1 ? nullptr : mask[1].data<int>(), dest_idx[0].data<int>(),
          k == 1 ? dest_idx[0].data<int>() : dest_idx[1].data<int>(), s, e, c,
          h));

  return res;
}

std::vector<torch::Tensor> moe_combine_cuda_backward(
    int s, int e, int c, int h, torch::Tensor tokens_grad,
    torch::Tensor expert_tokens, torch::Tensor logits, torch::Tensor mask,
    torch::Tensor dest_idx) {
  assert(h % 16 == 0);
  assert(tokens_grad.dtype() == expert_tokens.dtype());
  assert(expert_tokens.dtype() == logits.dtype());

  auto egrad = torch::zeros(
           {e * c, h},
           torch::dtype(tokens_grad.dtype()).device(tokens_grad.device())),
       wgrad = torch::zeros(
           {s, e}, torch::dtype(logits.dtype()).device(logits.device()));
  auto k = mask.size(0);

  DISPATCH_FLOAT_AND_HALF(
      tokens_grad.scalar_type(), "moe combine backward",
      moe_cb_bwd_launch<scalar_t>(
          tokens_grad.data<scalar_t>(), egrad.data<scalar_t>(),
          expert_tokens.data<scalar_t>(), logits.data<scalar_t>(),
          wgrad.data<scalar_t>(), mask[0].data<int>(),
          k == 1 ? nullptr : mask[1].data<int>(), dest_idx[0].data<int>(),
          k == 1 ? dest_idx[0].data<int>() : dest_idx[1].data<int>(), s, e, c,
          h));

  return {egrad, wgrad};
}

torch::Tensor cumsum_sub_one_in_dim0(torch::Tensor mask) {
  assert(mask.dim() == 2);
  assert(mask.dtype() == torch::kInt32);

  const int s = mask.size(0), e = mask.size(1);
  auto res =
      torch::empty({s, e}, torch::dtype(torch::kInt32).device(mask.device()));
  cumsum_launch(mask.data<int>(), res.data<int>(), s, e);

  return res;
}

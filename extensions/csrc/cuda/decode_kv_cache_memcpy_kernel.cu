#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "../common/vector_copy_utils.h"
#include "../common/micros.h"

template<typename scalar_t, int VecSize>
__global__ void decode_kv_cache_memcpy_kernel(
    const scalar_t* __restrict__ key,
    const scalar_t* __restrict__ value,
    scalar_t* __restrict__ key_cache,
    scalar_t* __restrict__ value_cache,
    const int* __restrict__ sequence_lengths,
    const int* __restrict__ block_tables,
    const int head_num,
    const int head_dim,
    const int block_size,
    const int64_t key_stride,
    const int64_t value_stride,
    const int block_table_stride
)
{
    const int seq_id = blockIdx.x;
    const int seq_len = sequence_lengths[seq_id] - 1;
    const int block_offset = seq_len % block_size;
    const int block_id = block_tables[seq_id * block_table_stride + seq_len / block_size];
    const int hidden_size = head_num * head_dim;

    if ( block_id < 0 ) {
        return ;
    }

    for (int i = threadIdx.x * VecSize; i < hidden_size; i += blockDim.x * VecSize) {
        const int head_id = i / head_dim;
        const int head_offset = i % head_dim;
        const int64_t key_src_id = seq_id * key_stride + i;
        const int64_t value_src_id = seq_id * value_stride + i;
        const int64_t target_id = block_id * hidden_size * block_size
                                      + head_id * block_size * head_dim
                                      + block_offset * head_dim + head_offset;

        copy_vector<scalar_t, VecSize>(key_cache + target_id, key + key_src_id);
        copy_vector<scalar_t, VecSize>(value_cache + target_id, value + value_src_id);
    }

}

template<typename scalar_t>
void apply_decode_kv_cache_memcpy(
    at::Tensor& key,                 // [num_tokens, head_num, head_dim]
    at::Tensor& value,               // [num_tokens, head_num, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& block_tables)        // [batch_size, max_seq_len]
{
    int num_tokens = key.size(0);
    int head_num = key.size(1);
    int head_dim = key.size(2);
    int block_size = key_cache.size(2);

    int64_t key_stride = key.stride(0);
    int64_t value_stride = value.stride(0);
    int block_table_stride = block_tables.stride(0);

    int vec_size = get_vec_size<scalar_t>(key);

    if (head_dim % vec_size != 0) {
        // Disable vectorized loading optimization when head_dim is not divisible by VecSize.
        vec_size = 1;
    }

    int thread_nums = head_num * head_dim / vec_size;

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 grid(num_tokens);
    dim3 block(std::min(thread_nums, 512));

    switch (vec_size) {
        case 1:
            decode_kv_cache_memcpy_kernel<scalar_t, 1><<<grid, block, 0, stream>>>(
                key.data_ptr<scalar_t>(),
                value.data_ptr<scalar_t>(),
                key_cache.data_ptr<scalar_t>(),
                value_cache.data_ptr<scalar_t>(),
                sequence_lengths.data_ptr<int>(),
                block_tables.data_ptr<int>(),
                head_num,
                head_dim,
                block_size,
                key_stride,
                value_stride,
                block_table_stride
            );
            break;
        case 2:
            decode_kv_cache_memcpy_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
                key.data_ptr<scalar_t>(),
                value.data_ptr<scalar_t>(),
                key_cache.data_ptr<scalar_t>(),
                value_cache.data_ptr<scalar_t>(),
                sequence_lengths.data_ptr<int>(),
                block_tables.data_ptr<int>(),
                head_num,
                head_dim,
                block_size,
                key_stride,
                value_stride,
                block_table_stride
            );
            break;
        case 4:
            decode_kv_cache_memcpy_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
                key.data_ptr<scalar_t>(),
                value.data_ptr<scalar_t>(),
                key_cache.data_ptr<scalar_t>(),
                value_cache.data_ptr<scalar_t>(),
                sequence_lengths.data_ptr<int>(),
                block_tables.data_ptr<int>(),
                head_num,
                head_dim,
                block_size,
                key_stride,
                value_stride,
                block_table_stride
            );
            break;
        default:
            AT_ERROR("Unsupported vectorized size ", vec_size);
            break;
    }

    AT_CUDA_CHECK(hipGetLastError());

}

void decode_kv_cache_memcpy(
    at::Tensor& key,                 // [num_tokens, head_num, head_dim]
    at::Tensor& value,               // [num_tokens, head_num, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& block_tables)        // [batch_size, max_seq_len]
{
    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        key.scalar_type(),
        "decode_kv_cache_memcpy",
        apply_decode_kv_cache_memcpy<scalar_t>(
            key,
            value,
            key_cache,
            value_cache,
            sequence_lengths,
            block_tables
        );)
}

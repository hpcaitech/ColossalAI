#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "utils/vec_copy.h"
#include "../common/micros.h"

template<typename scalar_t, bool Aligned, int VecSize>
__global__ void decode_kv_cache_memcpy_kernel(
    const scalar_t* __restrict__ key,
    const scalar_t* __restrict__ value,
    scalar_t* __restrict__ key_cache,
    scalar_t* __restrict__ value_cache,
    const int* __restrict__ sequence_lengths,
    const int* __restrict__ block_tables,
    const int head_num,
    const int head_dim,
    const int block_size,
    const int64_t key_stride,
    const int64_t value_stride,
    const int block_table_stride
)
{
    const int seq_id = blockIdx.x;
    const int seq_len = sequence_lengths[seq_id] - 1;
    const int block_offset = seq_len % block_size;
    const int block_id = block_tables[seq_id * block_table_stride + seq_len / block_size];
    const int hidden_size = head_num * head_dim;

    if ( block_id < 0 ) {
        return ;
    }

    int i = threadIdx.x * VecSize;

    for (; i <= (hidden_size - VecSize); i += blockDim.x * VecSize) {
        const int head_id = i / head_dim;
        const int head_offset = i % head_dim;
        const int64_t key_src_id = seq_id * key_stride + i;
        const int64_t value_src_id = seq_id * value_stride + i;
        const int64_t target_id = block_id * hidden_size * block_size
                                      + head_id * block_size * head_dim
                                      + block_offset * head_dim + head_offset;

        copy_vector<scalar_t, VecSize>(key_cache + target_id, key + key_src_id);
        copy_vector<scalar_t, VecSize>(value_cache + target_id, value + value_src_id);
    }

    if (!Aligned) {
        for (; i < hidden_size; ++i ) {
            const int head_id = i / head_dim;
            const int head_offset = i % head_dim;
            const int64_t key_src_id = seq_id * key_stride + i;
            const int64_t value_src_id = seq_id * value_stride + i;
            const int64_t target_id = block_id * hidden_size * block_size
                                        + head_id * block_size * head_dim
                                        + block_offset * head_dim + head_offset;

            key_cache[target_id] = key[key_src_id];
            value_cache[target_id] = value[value_src_id];
        }
    }

}

template<typename scalar_t>
void apply_decode_kv_cache_memcpy(
    at::Tensor& key,                 // [num_tokens, head_num, head_dim]
    at::Tensor& value,               // [num_tokens, head_num, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& block_tables)        // [batch_size, max_seq_len]
{
    int num_tokens = key.size(0);
    int head_num = key.size(1);
    int head_dim = key.size(2);
    int block_size = key_cache.size(2);

    int64_t key_stride = key.stride(0);
    int64_t value_stride = value.stride(0);
    int block_table_stride = block_tables.stride(0);

    int vec_size = get_vec_size<scalar_t>(key);

    bool aligned = true;
    if (head_dim % vec_size != 0) {
        aligned = false;
    }

    int thread_nums = head_num * head_dim / vec_size;
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 grid(num_tokens);
    dim3 block(std::min(thread_nums, 512));

#define DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, __vec_size)                                    \
    do {                                                                                                \
        decode_kv_cache_memcpy_kernel<scalar_t, __aligned, __vec_size><<<grid, block, 0, stream>>>(     \
                key.data_ptr<scalar_t>(),                                                               \
                value.data_ptr<scalar_t>(),                                                             \
                key_cache.data_ptr<scalar_t>(),                                                         \
                value_cache.data_ptr<scalar_t>(),                                                       \
                sequence_lengths.data_ptr<int>(),                                                       \
                block_tables.data_ptr<int>(),                                                           \
                head_num,                                                                               \
                head_dim,                                                                               \
                block_size,                                                                             \
                key_stride,                                                                             \
                value_stride,                                                                           \
                block_table_stride                                                                      \
            );                                                                                          \
    } while(0)

#define DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH_VEC_SIZE_CASE(__aligned, __vec_size)                      \
    do {                                                                                                \
        switch (__vec_size) {                                                                           \
            case 1:                                                                                     \
                DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, 1);                                    \
                break;                                                                                  \
            case 2:                                                                                     \
                DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, 2);                                    \
                break;                                                                                  \
            case 4:                                                                                     \
                DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, 4);                                    \
                break;                                                                                  \
            default:                                                                                    \
                AT_ERROR("Unsupported vectorized size ", __vec_size);                                   \
                break;                                                                                  \
        }                                                                                               \
    } while(0)

    if (aligned) {
        DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH_VEC_SIZE_CASE(true, vec_size);
    }
    else {
        DECODE_KV_CACHE_MEMCOPY_KERNEL_LAUNCH_VEC_SIZE_CASE(false, vec_size);
    }

    AT_CUDA_CHECK(hipGetLastError());

}

void decode_kv_cache_memcpy(
    at::Tensor& key,                 // [num_tokens, head_num, head_dim]
    at::Tensor& value,               // [num_tokens, head_num, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& block_tables)        // [batch_size, max_seq_len]
{
    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        key.scalar_type(),
        "decode_kv_cache_memcpy",
        apply_decode_kv_cache_memcpy<scalar_t>(
            key,
            value,
            key_cache,
            value_cache,
            sequence_lengths,
            block_tables
        );)
}

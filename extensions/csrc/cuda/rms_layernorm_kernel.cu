#include "hip/hip_runtime.h"
/*This code from VLLM:
 *     https://github.com/vllm-project/vllm/
 *     with minor changes. */

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <stdio.h>


#include "block_reduce.h"
#include "type_shim.h"

template<typename scalar_t>
__global__ void rms_layernorm_kernel(
  scalar_t* __restrict__ out,             // [..., hidden_size]
  const scalar_t* __restrict__ input,     // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;
  /*
   * since the open-sourced LLM's hidden dimensions mainly range from
   * 4096 (LLAMA-7B) to 8192 (LLAMA-65B), we thus set the supported
   * hidden dimension limit to 8192, and each thread's capacity
   * for caching input tensors to 8 (8192 = 8 * 1024) which
   * will cause problems for extremely large models, such as
   * Megatron-Turing NLG 530B with hidden dimensions up to 20480
   */
  float x_local[8];

  for (int idx = threadIdx.x, cnt = 0; idx < hidden_size; idx += blockDim.x, cnt++) {
    x_local[cnt] = (float) input[blockIdx.x * hidden_size + idx];
    variance += x_local[cnt] * x_local[cnt];
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x, cnt = 0; idx < hidden_size; idx += blockDim.x, cnt++) {
    out[blockIdx.x * hidden_size + idx] = ((scalar_t) (x_local[cnt] * s_variance)) * weight[idx];
  }
}

template<typename scalar_t>
__global__ void fused_add_rms_layernorm_kernel(
  scalar_t* __restrict__ input,           // [..., hidden_size]
  scalar_t* __restrict__ residual,        // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;
  float x_local[8];

  for (int idx = threadIdx.x, cnt = 0; idx < hidden_size; idx += blockDim.x, cnt++) {
    x_local[cnt] = (float) input[blockIdx.x * hidden_size + idx];
    x_local[cnt] += (float) residual[blockIdx.x * hidden_size + idx];
    variance += x_local[cnt] * x_local[cnt];
    residual[blockIdx.x * hidden_size + idx] = (scalar_t) x_local[cnt];
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x, cnt = 0; idx < hidden_size; idx += blockDim.x, cnt++) {
    input[blockIdx.x * hidden_size + idx] = ((scalar_t) (x_local[cnt] * s_variance)) * weight[idx];
  }
}

void rms_layernorm(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& weight,   // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
    input.scalar_type(),
    "rms_layernorm_kernel",
    rms_layernorm_kernel<scalar_t><<<grid, block, 0, stream>>>(
      out.data_ptr<scalar_t>(),
      input.data_ptr<scalar_t>(),
      weight.data_ptr<scalar_t>(),
      epsilon,
      num_tokens,
      hidden_size);)
}

void fused_add_rms_layernorm(
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& residual, // [..., hidden_size]
  torch::Tensor& weight,   // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
    input.scalar_type(),
    "fused_add_rms_layernorm_kernel",
    fused_add_rms_layernorm_kernel<scalar_t><<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      residual.data_ptr<scalar_t>(),
      weight.data_ptr<scalar_t>(),
      epsilon,
      num_tokens,
      hidden_size);)
}

#include "hip/hip_runtime.h"

#include "../common/micros.h"
#include "rotary_emb_and_cache_utils.h"
#include "stdio.h"

template<typename scalar_t, int VecSize>
__global__ void rotary_embedding_and_cache_copy_kernel(
    scalar_t* __restrict__ query,
    scalar_t* __restrict__ key,
    scalar_t* __restrict__ value,
    const scalar_t* __restrict__ cos,
    const scalar_t* __restrict__ sin,
    scalar_t* __restrict__ key_cache,
    scalar_t* __restrict__ value_cache,
    const int* __restrict__ sequence_lengths,
    const int* __restrict__ block_tables,
    const int64_t query_stride,
    const int64_t key_stride,
    const int64_t value_stride,
    const int64_t half_shard_element_num,
    const int cos_stride,
    const int sin_stride,
    const int block_table_stride,
    const int head_num,
    const int head_dim,
    const int kv_head_num,
    const int block_size
) {

    const int token_id = blockIdx.x;
    const int half_head_dim = head_dim / 2;
    const int shard_block_size = VecSize * 32;

    extern __shared__ char shard_ptr[];

    scalar_t *cos_ptr = (scalar_t*)shard_ptr;
    scalar_t *sin_ptr = cos_ptr + half_shard_element_num;

    // apply cos_sin memcopy
    cos_sin_memory_access<scalar_t, VecSize>(cos, sin, cos_ptr, sin_ptr, token_id, shard_block_size, cos_stride, sin_stride, half_head_dim);
    __syncthreads();

    //compute query
    apply_emb_rotary_compute<scalar_t, VecSize>(query, cos_ptr, sin_ptr, query_stride, token_id, shard_block_size, half_head_dim, head_num, head_dim);

    //compute key and copy kv
    apply_k_rotary_emb_compute<scalar_t, VecSize>(key, value, key_cache, value_cache, cos_ptr, sin_ptr, sequence_lengths, block_tables, key_stride, value_stride, token_id, block_table_stride, head_num, head_dim, kv_head_num, block_size, half_head_dim, shard_block_size);
}

template<typename scalar_t, int VecSize>
__global__ void rotary_embedding_kernel(
    scalar_t* __restrict__ query,
    scalar_t* __restrict__ key,
    const scalar_t* __restrict__ cos,
    const scalar_t* __restrict__ sin,
    const int64_t query_stride,
    const int64_t key_stride,
    const int64_t half_shard_element_num,
    const int cos_stride,
    const int sin_stride,
    const int head_num,
    const int head_dim,
    const int kv_head_num
) {
    const int token_id = blockIdx.x;
    const int half_head_dim = head_dim / 2;
    const int shard_block_size = VecSize * 32;

    extern __shared__ char shard_ptr[];

    scalar_t *cos_ptr = (scalar_t*)shard_ptr;
    scalar_t *sin_ptr = cos_ptr + half_shard_element_num;

    // apply cos_sin memcopy
    cos_sin_memory_access<scalar_t, VecSize>(cos, sin, cos_ptr, sin_ptr, token_id, shard_block_size, cos_stride, sin_stride, half_head_dim);
    __syncthreads();

    //compute query
    apply_emb_rotary_compute<scalar_t, VecSize>(query, cos_ptr, sin_ptr, query_stride, token_id, shard_block_size, half_head_dim, head_num, head_dim);

    //compute key
    apply_emb_rotary_compute<scalar_t, VecSize>(key, cos_ptr, sin_ptr, key_stride, token_id, shard_block_size, half_head_dim, kv_head_num, head_dim);
}

template<typename scalar_t>
void apply_rotary_embedding_and_cache_copy(
    at::Tensor& query,               // [num_tokens, head_num, head_dim]
    at::Tensor& key,                 // [num_tokens, kv_head_num, head_dim]
    at::Tensor& value,               // [num_tokens, kv_head_num, head_dim]
    at::Tensor& cos,                 // [num_tokens, head_dim]
    at::Tensor& sin,                 // [num_tokens, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& block_tables)        // [batch_size, max_seq_len]
{
    int num_tokens = query.size(0);
    int head_num = query.size(1);
    int head_dim = query.size(2);
    int kv_head_num = key.size(1);
    int block_size = key_cache.size(2);

    int64_t query_stride = query.stride(0);
    int64_t key_stride = key.stride(0);
    int64_t value_stride = value.stride(0);
    int cos_stride = cos.stride(0);
    int sin_stride = sin.stride(0);
    int block_table_stride = block_tables.stride(0);

    int vec_size = get_vec_size<scalar_t>(query);

    if ((head_dim / 2) % vec_size != 0) {
        // Disable vectorized loading optimization when head_dim is not divisible by VecSize.
        vec_size = 1;
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int thread_nums = head_num * head_dim / vec_size / 2;
    const int shard_block_size = vec_size * 32 * 2;

    dim3 grid(num_tokens);
    dim3 block(std::min(thread_nums, 512));
    int64_t shard_element_num = ((head_dim + shard_block_size - 1) / shard_block_size) * shard_block_size ;

    switch (vec_size) {
        case 1:
            rotary_embedding_and_cache_copy_kernel<scalar_t, 1><<<grid, block, shard_element_num * sizeof(scalar_t), stream>>>(
                query.data_ptr<scalar_t>(),
                key.data_ptr<scalar_t>(),
                value.data_ptr<scalar_t>(),
                cos.data_ptr<scalar_t>(),
                sin.data_ptr<scalar_t>(),
                key_cache.data_ptr<scalar_t>(),
                value_cache.data_ptr<scalar_t>(),
                sequence_lengths.data_ptr<int>(),
                block_tables.data_ptr<int>(),
                query_stride,
                key_stride,
                value_stride,
                shard_element_num / 2,
                cos_stride,
                sin_stride,
                block_table_stride,
                head_num,
                head_dim,
                kv_head_num,
                block_size
            );
            break;
        case 2:
            rotary_embedding_and_cache_copy_kernel<scalar_t, 2><<<grid, block, shard_element_num * sizeof(scalar_t), stream>>>(
                query.data_ptr<scalar_t>(),
                key.data_ptr<scalar_t>(),
                value.data_ptr<scalar_t>(),
                cos.data_ptr<scalar_t>(),
                sin.data_ptr<scalar_t>(),
                key_cache.data_ptr<scalar_t>(),
                value_cache.data_ptr<scalar_t>(),
                sequence_lengths.data_ptr<int>(),
                block_tables.data_ptr<int>(),
                query_stride,
                key_stride,
                value_stride,
                shard_element_num / 2,
                cos_stride,
                sin_stride,
                block_table_stride,
                head_num,
                head_dim,
                kv_head_num,
                block_size
            );
            break;
        case 4:
            rotary_embedding_and_cache_copy_kernel<scalar_t, 4><<<grid, block, shard_element_num * sizeof(scalar_t), stream>>>(
                query.data_ptr<scalar_t>(),
                key.data_ptr<scalar_t>(),
                value.data_ptr<scalar_t>(),
                cos.data_ptr<scalar_t>(),
                sin.data_ptr<scalar_t>(),
                key_cache.data_ptr<scalar_t>(),
                value_cache.data_ptr<scalar_t>(),
                sequence_lengths.data_ptr<int>(),
                block_tables.data_ptr<int>(),
                query_stride,
                key_stride,
                value_stride,
                shard_element_num / 2,
                cos_stride,
                sin_stride,
                block_table_stride,
                head_num,
                head_dim,
                kv_head_num,
                block_size
            );
            break;
        default:
            AT_ERROR("Unsupported vectorized size ", vec_size);
            break;
    }

    AT_CUDA_CHECK(hipGetLastError());
}

template<typename scalar_t>
void apply_rotary_embedding(
    at::Tensor& query,   // [total_tokens, head_num, head_dim]
    at::Tensor& key,     // [total_tokens, kv_head_num, head_dim]
    at::Tensor& cos,     // [total_tokens, head_dim]
    at::Tensor& sin      // [total_tokens, head_dim]
){
    int num_tokens = query.size(0);
    int head_num = query.size(1);
    int head_dim = query.size(2);
    int kv_head_num = key.size(1);

    int query_stride = query.stride(0);
    int key_stride = key.stride(0);
    int cos_stride = cos.stride(0);
    int sin_stride = sin.stride(0);

    int vec_size = get_vec_size<scalar_t>(query);

    if ((head_dim / 2) % vec_size != 0) {
        // Disable vectorized loading optimization when head_dim is not divisible by VecSize.
        vec_size = 1;
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int thread_nums = head_num * head_dim / vec_size / 2;
    const int shard_block_size = vec_size * 32 * 2;

    dim3 grid(num_tokens);
    dim3 block(std::min(thread_nums, 512));
    int64_t shard_element_num = ((head_dim + shard_block_size - 1) / shard_block_size) * shard_block_size ;

    switch (vec_size) {
        case 1:
            rotary_embedding_kernel<scalar_t, 1><<<grid, block, shard_element_num * sizeof(scalar_t), stream>>>(
                    query.data_ptr<scalar_t>(),
                    key.data_ptr<scalar_t>(),
                    cos.data_ptr<scalar_t>(),
                    sin.data_ptr<scalar_t>(),
                    query_stride,
                    key_stride,
                    shard_element_num / 2,
                    cos_stride,
                    sin_stride,
                    head_num,
                    head_dim,
                    kv_head_num
                );
            break;
        case 2:
            rotary_embedding_kernel<scalar_t, 2><<<grid, block, shard_element_num * sizeof(scalar_t), stream>>>(
                    query.data_ptr<scalar_t>(),
                    key.data_ptr<scalar_t>(),
                    cos.data_ptr<scalar_t>(),
                    sin.data_ptr<scalar_t>(),
                    query_stride,
                    key_stride,
                    shard_element_num / 2,
                    cos_stride,
                    sin_stride,
                    head_num,
                    head_dim,
                    kv_head_num
                );
            break;
        case 4:
            rotary_embedding_kernel<scalar_t, 4><<<grid, block, shard_element_num * sizeof(scalar_t), stream>>>(
                    query.data_ptr<scalar_t>(),
                    key.data_ptr<scalar_t>(),
                    cos.data_ptr<scalar_t>(),
                    sin.data_ptr<scalar_t>(),
                    query_stride,
                    key_stride,
                    shard_element_num / 2,
                    cos_stride,
                    sin_stride,
                    head_num,
                    head_dim,
                    kv_head_num
                );
            break;
        default:
            AT_ERROR("Unsupported vectorized size ", vec_size);
            break;
    }
    AT_CUDA_CHECK(hipGetLastError());
}

void rotary_embedding_and_cache_copy(
    at::Tensor& query,               // [num_tokens, head_num, head_dim]
    at::Tensor& key,                 // [num_tokens, kv_head_num, head_dim]
    at::Tensor& value,               // [num_tokens, kv_head_num, head_dim]
    at::Tensor& cos,                 // [num_tokens, head_dim]
    at::Tensor& sin,                 // [num_tokens, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& block_tables)        // [batch_size, max_seq_len]
{
    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        query.scalar_type(),
        "rotary_embedding_and_cache_copy",
        apply_rotary_embedding_and_cache_copy<scalar_t>(
            query,
            key,
            value,
            cos,
            sin,
            key_cache,
            value_cache,
            sequence_lengths,
            block_tables
        );)
}

void rotary_embedding(
    at::Tensor& query,   // [total_tokens, head_num, head_dim]
    at::Tensor& key,     // [total_tokens, kv_head_num, head_dim]
    at::Tensor& cos,     // [total_tokens, head_dim]
    at::Tensor& sin      // [total_tokens, head_dim]
){
    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        query.scalar_type(),
        "rotary_embedding",
        apply_rotary_embedding<scalar_t>(
            query,
            key,
            cos,
            sin
        );)
}

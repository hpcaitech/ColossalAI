#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "utils/vec_copy.h"
#include "../common/micros.h"

template<typename scalar_t, bool Aligned, int VecSize>
__global__ void context_kv_cache_memcpy_kernel(
    const scalar_t* __restrict__ key,
    const scalar_t* __restrict__ value,
    scalar_t* __restrict__ key_cache,
    scalar_t* __restrict__ value_cache,
    const int* __restrict__ sequence_lengths,
    const int* __restrict__ cu_seqlens,
    const int* __restrict__ block_tables,
    const int head_num,
    const int head_dim,
    const int block_size,
    const int batch_size,
    const int block_table_stride,
    const int64_t key_stride,
    const int64_t value_stride
)
{
    const int seq_token_id = blockIdx.x;
    const int seq_id = blockIdx.y;
    const int block_id = block_tables[seq_id * block_table_stride + seq_token_id / block_size];

    if ( block_id < 0 || seq_token_id > sequence_lengths[seq_id] - 1) {
        return ;
    }

    const int block_offset = seq_token_id % block_size;
    const int hidden_size = head_num * head_dim;
    const int total_token_id = cu_seqlens[seq_id] + seq_token_id;
    int head_id;
    int head_offset;
    int64_t key_src_id;
    int64_t value_src_id;
    int64_t target_id;

    int i = threadIdx.x * VecSize;

    for (; i <= (hidden_size - VecSize); i += blockDim.x * VecSize) {
        head_id = i / head_dim;
        head_offset = i % head_dim;
        key_src_id = total_token_id * key_stride + i;
        value_src_id = total_token_id * value_stride + i;
        target_id = block_id * hidden_size * block_size
                                      + head_id * block_size * head_dim
                                      + block_offset * head_dim + head_offset;

        copy_vector<scalar_t, VecSize>(key_cache + target_id, key + key_src_id);
        copy_vector<scalar_t, VecSize>(value_cache + target_id, value + value_src_id);
    }

    // tail process
    if (!Aligned) {
        for (; i < hidden_size; ++i ) {
            head_id = i / head_dim;
            head_offset = i % head_dim;
            key_src_id = total_token_id * key_stride + i;
            value_src_id = total_token_id * value_stride + i;
            target_id = block_id * hidden_size * block_size
                                        + head_id * block_size * head_dim
                                        + block_offset * head_dim + head_offset;

            key_cache[target_id] =  key[key_src_id];
            value_cache[target_id] = value[value_src_id];
        }
    }

}

template<typename scalar_t>
void apply_context_kv_cache_memcpy(
    at::Tensor& key,                 // [num_tokens, head_num, head_dim]
    at::Tensor& value,               // [num_tokens, head_num, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& cu_seqlens,          // [batch_size + 1]
    at::Tensor& block_tables,        // [batch_size, max_seq_len]
    int max_seq_len_in_batch)
{
    int num_tokens = key.size(0);
    int head_num = key.size(1);
    int head_dim = key.size(2);
    int block_size = key_cache.size(2);
    int batch_size = block_tables.size(0);

    int64_t key_stride = key.stride(0);
    int64_t value_stride = value.stride(0);
    int block_table_stride = block_tables.stride(0);

    int vec_size = get_vec_size<scalar_t>(key);

    bool aligned = true;
    if (head_dim % vec_size != 0) {
        aligned = false;
    }

    int thread_nums = head_num * head_dim / vec_size;
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 grid(max_seq_len_in_batch, batch_size);
    dim3 block(std::min(thread_nums, 512));

#define CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, __vec_size)                                   \
    do {                                                                                                \
        context_kv_cache_memcpy_kernel<scalar_t, __aligned, __vec_size><<<grid, block, 0, stream>>>(    \
                key.data_ptr<scalar_t>(),                                                               \
                value.data_ptr<scalar_t>(),                                                             \
                key_cache.data_ptr<scalar_t>(),                                                         \
                value_cache.data_ptr<scalar_t>(),                                                       \
                sequence_lengths.data_ptr<int>(),                                                       \
                cu_seqlens.data_ptr<int>(),                                                             \
                block_tables.data_ptr<int>(),                                                           \
                head_num,                                                                               \
                head_dim,                                                                               \
                block_size,                                                                             \
                batch_size,                                                                             \
                block_table_stride,                                                                     \
                key_stride,                                                                             \
                value_stride                                                                            \
            );                                                                                          \
    } while(0)

#define CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH_VEC_SIZE_CASE(__aligned)                                 \
    do {                                                                                                \
        switch (vec_size) {                                                                             \
            case 1:                                                                                     \
                CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, 1);                                   \
                break;                                                                                  \
            case 2:                                                                                     \
                CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, 2);                                   \
                break;                                                                                  \
            case 4:                                                                                     \
                CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH(__aligned, 4);                                   \
                break;                                                                                  \
            default:                                                                                    \
                AT_ERROR("Unsupported vectorized size ", vec_size);                                     \
                break;                                                                                  \
        }                                                                                               \
    } while(0)


    if (aligned) {
        CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH_VEC_SIZE_CASE(true);
    }
    else {
        CONTEXT_KV_CACHE_MEMCOPY_KERNEL_LAUNCH_VEC_SIZE_CASE(false);
    }

    AT_CUDA_CHECK(hipGetLastError());

}

void context_kv_cache_memcpy(
    at::Tensor& key,                 // [num_tokens, head_num, head_dim]
    at::Tensor& value,               // [num_tokens, head_num, head_dim]
    at::Tensor& key_cache,           // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& value_cache,         // [num_blocks, head_num, block_size, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    at::Tensor& cu_seqlens,          // [batch_size + 1]
    at::Tensor& block_tables,        // [batch_size, max_seq_len]
    int max_seq_len_in_batch)
{
    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        key.scalar_type(),
        "context_kv_cache_memcpy",
        apply_context_kv_cache_memcpy<scalar_t>(
            key,
            value,
            key_cache,
            value_cache,
            sequence_lengths,
            cu_seqlens,
            block_tables,
            max_seq_len_in_batch
        );)
}

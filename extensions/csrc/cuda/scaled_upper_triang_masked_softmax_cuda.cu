/*This code from NVIDIA Megatron:
 *     with minor changes. */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "scaled_upper_triang_masked_softmax.h"
#include "type_shim.h"

namespace multihead_attn {
namespace fused_softmax {
namespace scaled_upper_triang_masked_softmax {

torch::Tensor fwd_cuda(torch::Tensor const& input, float scale_factor) {
  // input is a 3d tensor with dimensions [attn_batches, seq_len, seq_len]
  const int attn_batches = input.size(0);
  const int seq_len = input.size(1);
  TORCH_INTERNAL_ASSERT(seq_len <= 2048);

  // Output
  auto act_options = input.options().requires_grad(false);
  torch::Tensor softmax_results =
      torch::empty({attn_batches, seq_len, seq_len}, act_options);

  // Softmax Intermediate Result Ptr
  void* input_ptr = static_cast<void*>(input.data_ptr());
  void* softmax_results_ptr = static_cast<void*>(softmax_results.data_ptr());

  DISPATCH_HALF_AND_BFLOAT(
      input.scalar_type(),
      "dispatch_scaled_upper_triang_masked_softmax_forward",
      dispatch_scaled_upper_triang_masked_softmax_forward<scalar_t, scalar_t,
                                                          float>(
          reinterpret_cast<scalar_t*>(softmax_results_ptr),
          reinterpret_cast<const scalar_t*>(input_ptr), scale_factor, seq_len,
          seq_len, attn_batches););
  return softmax_results;
}

torch::Tensor bwd_cuda(torch::Tensor const& output_grads_,
                       torch::Tensor const& softmax_results_,
                       float scale_factor) {
  auto output_grads = output_grads_.contiguous();
  auto softmax_results = softmax_results_.contiguous();

  // output grads is a 3d tensor with dimensions [attn_batches, seq_len,
  // seq_len]
  const int attn_batches = output_grads.size(0);
  const int seq_len = output_grads.size(1);
  TORCH_INTERNAL_ASSERT(output_grads.size(1) == output_grads.size(2));

  void* output_grads_ptr = static_cast<void*>(output_grads.data_ptr());

  // Softmax Grad
  DISPATCH_HALF_AND_BFLOAT(
      output_grads_.scalar_type(),
      "dispatch_scaled_upper_triang_masked_softmax_backward",
      dispatch_scaled_upper_triang_masked_softmax_backward<scalar_t, scalar_t,
                                                           float>(
          reinterpret_cast<scalar_t*>(output_grads_ptr),
          reinterpret_cast<scalar_t*>(output_grads_ptr),
          reinterpret_cast<scalar_t const*>(softmax_results.data_ptr()),
          scale_factor, seq_len, seq_len, attn_batches););

  // backward pass is completely in-place
  return output_grads;
}
}  // namespace scaled_upper_triang_masked_softmax
}  // namespace fused_softmax
}  // namespace multihead_attn

#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <stdio.h>

#include "type_shim.h"
#include "include/mp_type_traits.h"

template<typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  using MT = typename infer::dtype::MPTypeTrait<T>::Type;
  return static_cast<T>((static_cast<MT>(x)) / (static_cast<MT>(1.0f) + expf(static_cast<MT>(-x))));
}

template<typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void act_and_mul_kernel(
  const scalar_t* __restrict__ ins_data,
  scalar_t* __restrict__ outs_data,
  const int64_t numel) {
  using MT = typename infer::dtype::MPTypeTrait<scalar_t>::Type;

  int64_t idx = static_cast<int64_t>(threadIdx.x) + static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x);
  const int64_t grid_size = blockDim.x * gridDim.x;
  if(idx > numel) {
    return;
  }

  for(int64_t i = idx; i < numel; i += grid_size) {
    scalar_t x = ins_data[i];
    scalar_t y = ins_data[i+numel];
    outs_data[i] = static_cast<scalar_t>(static_cast<MT>(ACT_FN(x)) * static_cast<MT>(y));
  }
}

// Note(LiuYang):This func is designed for calculation mode like
// silu(x[:half_1stdim]) * (x[half_1stdim:])
torch::Tensor silu_and_mul(const torch::Tensor& ins)
{
    auto ins_shape = ins.sizes().vec();

    ins_shape[0] = ins_shape[0]/2;
    auto outs = torch::zeros(ins_shape,ins.options());
    auto outs_shape = ins.sizes().vec();

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // Note(Liuyang): numel of ins must be divisible by 2
    int64_t numel = ((torch::numel(ins)) >> 1);

    // TODO(LiuYang): Maybe we need to implement a function to get launch config
    dim3 grid((numel+255)/256);
    dim3 block(256);

    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        ins.scalar_type(),
        "silu_and_mul",
        act_and_mul_kernel<scalar_t,silu_kernel<scalar_t>><<<grid, block, 0, stream>>>(
            ins.data_ptr<scalar_t>(),
            outs.data_ptr<scalar_t>(),
            numel
        );)

    AT_CUDA_CHECK(hipGetLastError());
    return outs;
}

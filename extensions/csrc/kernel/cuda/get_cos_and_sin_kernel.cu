#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "utils/vec_copy.h"
#include "common/micros.h"

using colossalAI::cuda::utils::copy;
using colossalAI::cuda::utils::get_vec_size;


template <typename scalar_t, bool Aligned, int VecSize>
__device__ void apply_cos_and_sin_memcopy(
    scalar_t* __restrict__ cos,
    scalar_t* __restrict__ sin,
    const scalar_t* __restrict__ cos_cache_ptr,
    const scalar_t* __restrict__ sin_cache_ptr,
    const int* __restrict__ sequence_lengths,
    const int head_dim,
    const int dest_offset_id,
    const int src_offset_id
 ) {

    int begin_id = threadIdx.x * VecSize;

    for (; begin_id <= head_dim - VecSize; begin_id += blockDim.x){
        copy<scalar_t, VecSize>(cos_cache_ptr + src_offset_id + begin_id, cos + dest_offset_id + begin_id);
        copy<scalar_t, VecSize>(sin_cache_ptr + src_offset_id + begin_id, sin + dest_offset_id + begin_id);
    }

    if (!Aligned) {
        for (; begin_id < head_dim; ++begin_id ) {
            cos[dest_offset_id + begin_id] = cos_cache_ptr[src_offset_id + begin_id];
            sin[dest_offset_id + begin_id] = sin_cache_ptr[src_offset_id + begin_id];
        }
    }
}

template <typename scalar_t, bool Aligned, int VecSize>
__global__ void apply_get_context_cos_and_sin_kernel(
    scalar_t* __restrict__ cos,
    scalar_t* __restrict__ sin,
    const scalar_t* __restrict__ cos_cache_ptr,
    const scalar_t* __restrict__ sin_cache_ptr,
    const int* __restrict__ sequence_lengths,
    const int* __restrict__ cumsum_lengths,
    const int batch_size,
    const int head_dim
) {
    int token_id = blockIdx.x;
    if ( token_id >= sequence_lengths[blockIdx.y] ) {
        return ;
    }

    int src_offset_id = token_id * head_dim;
    int dest_offset_id = src_offset_id;

    if (blockIdx.y > 0) {
        dest_offset_id += cumsum_lengths[blockIdx.y - 1] * head_dim;
    }

    apply_cos_and_sin_memcopy<scalar_t, Aligned, VecSize>(
        cos,
        sin,
        cos_cache_ptr,
        sin_cache_ptr,
        sequence_lengths,
        head_dim,
        dest_offset_id,
        src_offset_id
    );

}

template <typename scalar_t, bool Aligned, int VecSize>
__global__ void apply_get_decode_cos_and_sin_kernel(
    scalar_t* __restrict__ cos,
    scalar_t* __restrict__ sin,
    const scalar_t* __restrict__ cos_cache_ptr,
    const scalar_t* __restrict__ sin_cache_ptr,
    const int* __restrict__ sequence_lengths,
    const int batch_size,
    const int head_dim
) {
    int src_offset_id = ( sequence_lengths[blockIdx.y] - 1 ) * head_dim;
    int dest_offset_id = blockIdx.y * head_dim;

    apply_cos_and_sin_memcopy<scalar_t, Aligned, VecSize>(
        cos,
        sin,
        cos_cache_ptr,
        sin_cache_ptr,
        sequence_lengths,
        head_dim,
        dest_offset_id,
        src_offset_id
    );
}

template<typename scalar_t>
void apply_get_cos_and_sin(
    at::Tensor& cos_cache,           // [max_rotary_position, head_dim]
    at::Tensor& sin_cache,           // [max_rotary_position, head_dim]
    at::Tensor& cos,                 // [num_tokens, head_dim]
    at::Tensor& sin,                 // [num_tokens, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    int max_seq_len_in_batch,
    bool is_prompts
) {
    int token_num = cos.size(0);
    int head_dim = cos.size(1);
    int batch_size = sequence_lengths.size(0);

    at::Tensor cumsum_lengths;

    int vec_size = get_vec_size<scalar_t>(cos);

    bool aligned = true;
    if (head_dim % vec_size != 0) {
        aligned = false;
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    int block_size_y;
    int block_size_x;

    if (is_prompts) {
        block_size_y = batch_size;
        block_size_x = max_seq_len_in_batch;
        // TODO: The cumsum operation can be fused into get_cos_and_sin kernel later on.
        cumsum_lengths = torch::cumsum(sequence_lengths, 0, torch::kInt32);
    }
    else{
        block_size_y = batch_size;
        block_size_x = 1;
    }

    int thread_nums = (head_dim + vec_size - 1) / vec_size;

    dim3 grid(block_size_x, block_size_y);
    dim3 block(std::min(thread_nums, 512));

#define GET_COS_AND_SIN_KERNEL_LAUNCH(__aligned, __vec_size)                                                        \
    do {                                                                                                            \
        if (is_prompts){                                                                                            \
            apply_get_context_cos_and_sin_kernel<scalar_t, __aligned, __vec_size><<<grid, block, 0, stream>>>(      \
                cos.data_ptr<scalar_t>(),                                                                           \
                sin.data_ptr<scalar_t>(),                                                                           \
                cos_cache.data_ptr<scalar_t>(),                                                                     \
                sin_cache.data_ptr<scalar_t>(),                                                                     \
                sequence_lengths.data_ptr<int>(),                                                                   \
                cumsum_lengths.data_ptr<int>(),                                                                     \
                batch_size,                                                                                         \
                head_dim                                                                                            \
            );                                                                                                      \
        }                                                                                                           \
        else {                                                                                                      \
            apply_get_decode_cos_and_sin_kernel<scalar_t, __aligned, __vec_size><<<grid, block, 0, stream>>>(       \
                cos.data_ptr<scalar_t>(),                                                                           \
                sin.data_ptr<scalar_t>(),                                                                           \
                cos_cache.data_ptr<scalar_t>(),                                                                     \
                sin_cache.data_ptr<scalar_t>(),                                                                     \
                sequence_lengths.data_ptr<int>(),                                                                   \
                batch_size,                                                                                         \
                head_dim                                                                                            \
            );                                                                                                      \
        }                                                                                                           \
    } while(0)

#define GET_COS_AND_SIN_KERNEL_LAUNCH_VEC_SIZE_CASE(__aligned)                                          \
    do {                                                                                                \
        switch (vec_size) {                                                                             \
            case 1:                                                                                     \
                GET_COS_AND_SIN_KERNEL_LAUNCH(__aligned, 1);                                            \
                break;                                                                                  \
            case 2:                                                                                     \
                GET_COS_AND_SIN_KERNEL_LAUNCH(__aligned, 2);                                            \
                break;                                                                                  \
            case 4:                                                                                     \
                GET_COS_AND_SIN_KERNEL_LAUNCH(__aligned, 4);                                            \
                break;                                                                                  \
            default:                                                                                    \
                AT_ERROR("Unsupported vectorized size ", vec_size);                                     \
                break;                                                                                  \
        }                                                                                               \
    } while(0)

    if (aligned) {
        GET_COS_AND_SIN_KERNEL_LAUNCH_VEC_SIZE_CASE(true);
    }
    else {
        GET_COS_AND_SIN_KERNEL_LAUNCH_VEC_SIZE_CASE(false);
    }

    AT_CUDA_CHECK(hipGetLastError());
}

void get_cos_and_sin(
    at::Tensor& cos_cache,           // [max_rotary_position, head_dim]
    at::Tensor& sin_cache,           // [max_rotary_position, head_dim]
    at::Tensor& cos,                 // [num_tokens, head_dim]
    at::Tensor& sin,                 // [num_tokens, head_dim]
    at::Tensor& sequence_lengths,    // [batch_size]
    int max_seq_len_in_batch,
    bool is_prompts
) {
    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        cos.scalar_type(),
        "get_cos_and_sin",
        apply_get_cos_and_sin<scalar_t>(
            cos_cache,
            sin_cache,
            cos,
            sin,
            sequence_lengths,
            max_seq_len_in_batch,
            is_prompts
        );)
}

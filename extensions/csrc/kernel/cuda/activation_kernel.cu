#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <stdio.h>

#include "common/micros.h"
#include "common/mp_type_traits.h"

using colossalAI::common::MPTypeTrait;

template<typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  using MT = typename MPTypeTrait<T>::Type;
  return static_cast<T>((static_cast<MT>(x)) / (static_cast<MT>(1.0f) + expf(static_cast<MT>(-x))));
}

template<typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void act_and_mul_kernel(
  const scalar_t* __restrict__ ins_data,
  scalar_t* __restrict__ outs_data,
  const int64_t numel) {
  using MT = typename MPTypeTrait<scalar_t>::Type;

  int64_t idx = static_cast<int64_t>(threadIdx.x) + static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x);
  const int64_t grid_size = blockDim.x * gridDim.x;
  if(idx > numel) {
    return;
  }

  for(int64_t i = idx; i < numel; i += grid_size) {
    scalar_t x = ins_data[i];
    scalar_t y = ins_data[i+numel];
    outs_data[i] = static_cast<scalar_t>(static_cast<MT>(ACT_FN(x)) * static_cast<MT>(y));
  }
}

// Note(LiuYang):This func is designed for calculation mode like
// silu(x[:half_1stdim]) * (x[half_1stdim:])
torch::Tensor silu_and_mul(const torch::Tensor& ins)
{
    // Note(LiuYang): According to torch doc, vec() may cost a lot, but I did't find a better api
    // to manipulate ins_shape which is IntArrayRef
    auto ins_shape = ins.sizes().vec();

    ins_shape[0] = ins_shape[0]/2;
    if (ins_shape[0] == 1) {
      ins_shape.erase(ins_shape.begin());
    }
    auto outs = torch::zeros(ins_shape,ins.options());

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // Note(Liuyang): numel of ins must be divisible by 2
    int64_t numel = ((torch::numel(ins)) >> 1);

    // Note(LiuYang): For better performance for special case of which input is [2, 64, 11008], now
    // I comment this part code，because it also cost a little time to calculate a better config
    // colossalAI::cuda::utils::NVGPUDevInfo dev_info(0);
    // auto config = colossalAI::cuda::utils::GetGPULaunchConfig1D(dev_info,numel,1);
    // dim3 grid = config.grid;
    // dim3 block = config.block;

    dim3 grid((numel+255)/256);
    dim3 block(256);

    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        ins.scalar_type(),
        "silu_and_mul",
        act_and_mul_kernel<scalar_t,silu_kernel<scalar_t>><<<grid, block, 0, stream>>>(
            ins.data_ptr<scalar_t>(),
            outs.data_ptr<scalar_t>(),
            numel
        );)

    AT_CUDA_CHECK(hipGetLastError());
    return outs;
}

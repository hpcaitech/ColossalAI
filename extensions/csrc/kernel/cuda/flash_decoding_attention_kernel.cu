#include "hip/hip_runtime.h"
/*This code adapted from vllm:
 *     https://github.com/vllm-project/vllm/blob/main/csrc/attention/attention_kernels.cu
 */

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include "common/micros.h"
#include "funcs/cast_functor.h"
#include "funcs/ternary_functor.h"
#include "funcs/binary_functor.h"
#include "common/vec_type_traits.h"
#include "attention/attention_utils.h"

#define WARP_SIZE 32
#define PARTITION_SIZE 512
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))
// 2^n => 2^n, 2^n-d => 2^(n-1)
#define ROUND_DOWN_HIGHEST_POWER_OF_TWO(x) (nextHighestPowerOf2((x - (x + 1) / 2 + 1)))

// a bit magic, you can ask chatgpt for help
// 2^n => 2^n, 2^n-d => 2^n
constexpr unsigned int nextHighestPowerOf2(unsigned int v) {
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

template <typename T>
inline __device__ void zero(T& dst) {
  constexpr int WORDS = sizeof(T) / 4;
  union {
    T raw;
    uint32_t words[WORDS];
  } tmp;

#pragma unroll
  for (int ii = 0; ii < WORDS; ii++) {
    tmp.words[ii] = 0u;
  }
  dst = tmp.raw;
}

using colossalAI::funcs::BinaryOpType;
using colossalAI::funcs::CastFunctor;
using colossalAI::funcs::TernaryOpFunctor;
using colossalAI::funcs::TernaryOpType;
using colossalAI::common::VecTypeTrait;
using colossalAI::common::FloatVecTypeTrait;
using namespace colossalAI::cuda::attention;

template<typename scalar_t, typename KVecT, int VEC_SIZE, int Q_SHARED_SIZE, int NUM_VECS_PER_THREAD, int NUM_THREADS_PER_X, int NUM_ROWS_PER_ROUNDS, int NUM_VECS_PER_TOKEN, int x>
__device__ void data_load(
  const float4* q_ptr,
  float4* q_shared,
  scalar_t* q_shared_ptr,
  KVecT* q_vecs,            // query cached at register for qk_dot, should be constructed with reference to key cache's layout
  const int* block_table,
  int* block_table_shared,
  const int lane,
  const int max_num_blocks_per_seq
) {

  #pragma unroll
  for (int idx = threadIdx.x; idx < Q_SHARED_SIZE; idx += blockDim.x) {
    q_shared[idx] = q_ptr[idx];
  }

  #pragma unroll
  for (int idx = threadIdx.x; idx < max_num_blocks_per_seq; idx += blockDim.x) {
    block_table_shared[idx] = block_table[idx];
  }

  __syncthreads();

  // each warp access a whole block

  #pragma unroll
  for (int idx = lane, i = 0; idx < NUM_ROWS_PER_ROUNDS * NUM_VECS_PER_TOKEN; idx += WARP_SIZE, i += 1) {
    const int offset0 = idx / NUM_THREADS_PER_X / NUM_ROWS_PER_ROUNDS;
    const int offset1 = idx % NUM_THREADS_PER_X;
    q_vecs[i] = *reinterpret_cast<KVecT*>(q_shared_ptr + offset0 * x + offset1 * VEC_SIZE);
  }
}

template<typename scalar_t, typename cache_t, typename KVecT, typename KQuantVecT, int NUM_WARPS, int NUM_VECS_PER_THREAD, int BLOCK_SIZE, int NUM_ROWS_PER_ROUNDS, int NUM_VECS_PER_TOKEN, int NUM_THREADS_PER_X, int x, int VEC_SIZE>
__device__ void qk_gemv(
  const cache_t* __restrict__ k_cache,
  const KVecT (&q_vecs)[NUM_VECS_PER_THREAD], // Qk_dot needs NUM_VECS_PER_THREAD to do loop unrolling
  float* logits,                              // shared memory to cache Qk_dot results
  int* block_table_shared,
  const float alibi_slope,
  const int context_len,
  float &qk_max,
  const float scale,
  const int kv_head_idx,
  const int warp_idx,
  const int lane,
  const int thread_group_offset,
  const int start_block_idx,
  const int end_block_idx,
  const int start_token_idx,
  const int kv_block_stride,
  const int kv_head_stride) {

  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    const int64_t physical_block_number = static_cast<int64_t>(block_table_shared[block_idx]);

    KVecT k_vecs[NUM_VECS_PER_THREAD];

    #pragma unroll
    for (int i = 0; i < BLOCK_SIZE; i += NUM_ROWS_PER_ROUNDS) {
      const cache_t* k_ptr = k_cache + physical_block_number * kv_block_stride
                                     + kv_head_idx * kv_head_stride
                                     + i * x;
      #pragma unroll
      for (int idx = lane, j = 0; idx < NUM_ROWS_PER_ROUNDS * NUM_VECS_PER_TOKEN; idx += WARP_SIZE, j += 1) {
        const int offset0 = idx / NUM_THREADS_PER_X / NUM_ROWS_PER_ROUNDS;
        const int offset1 = (idx / NUM_THREADS_PER_X) % NUM_ROWS_PER_ROUNDS;
        const int offset2 = idx % NUM_THREADS_PER_X;
        k_vecs[j] = CastFunctor<KQuantVecT, KVecT>()(*reinterpret_cast<const KQuantVecT*>(k_ptr + offset0 * BLOCK_SIZE * x + offset1 * x + offset2 * VEC_SIZE));
      }

      float qk = scale * Qk_dot<scalar_t, NUM_ROWS_PER_ROUNDS * NUM_THREADS_PER_X, NUM_THREADS_PER_X>::dot(q_vecs, k_vecs);

      if (thread_group_offset == 0 && lane < NUM_ROWS_PER_ROUNDS * NUM_THREADS_PER_X) {
        const int token_idx = block_idx * BLOCK_SIZE + i * NUM_ROWS_PER_ROUNDS + lane / NUM_THREADS_PER_X;
        qk += (alibi_slope != 0) ? alibi_slope * (token_idx - context_len + 1) : 0;
        const bool mask = token_idx >= context_len;
        logits[token_idx - start_token_idx] = mask ? 0.f : qk;
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }
}

template<int NUM_THREADS, int NUM_WARPS, int NUM_ROWS_PER_ROUNDS, int NUM_THREADS_PER_X>
__device__ void softmax(
  float* red_shared_mem,
  float* logits,
  float &qk_max,
  float &exp_sum,
  int num_tokens) {
  // there exists a __syncthreads within this function
  qk_max = block_max<NUM_WARPS, NUM_ROWS_PER_ROUNDS * NUM_THREADS_PER_X, NUM_THREADS_PER_X>(red_shared_mem, qk_max);

  // Get the sum of the exp values.
  for (int i = threadIdx.x; i < num_tokens; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }

  exp_sum = block_sum<NUM_WARPS>(&red_shared_mem[NUM_WARPS], exp_sum);
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = threadIdx.x; i < num_tokens; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();
}

template<typename scalar_t, typename cache_t, typename FloatVecT, typename VVecT, typename VQuantVecT, int NUM_WARPS, int NUM_ROUNDS_PER_TOKEN, int NUM_THREADS_PER_TOKEN, int BLOCK_SIZE, int VEC_SIZE, int NUM_VECS_PER_TOKEN, int WARP_STRIDE>
__device__ void sv_gemv(
  const cache_t* __restrict__ v_cache,
  int* block_table_shared,
  float* out_shared_mem,      // shared memory to cache sv_gemv results
  float* logits,
  FloatVecT* accs,            // registers for accumulation
  const int lane,
  const int warp_idx,
  const int kv_head_idx,
  const int start_block_idx,
  const int end_block_idx,
  const int context_len,
  const int start_token_idx,
  const int kv_block_stride,
  const int kv_head_stride) {

  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    zero(accs[i]);
  }

  VVecT zero_value;
  zero(zero_value);
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    const int64_t physical_block_number = static_cast<int64_t>(block_table_shared[block_idx]);
    scalar_t logit;

    #pragma unroll
    for (int idx = lane; idx < BLOCK_SIZE * NUM_VECS_PER_TOKEN; idx += WARP_STRIDE) {
      const int token_idx = block_idx * BLOCK_SIZE + idx / NUM_VECS_PER_TOKEN;
      const cache_t* v_ptr = v_cache + physical_block_number * kv_block_stride
                                     + kv_head_idx * kv_head_stride
                                     + idx * VEC_SIZE;

      VVecT v_vecs[NUM_ROUNDS_PER_TOKEN];

      #pragma unroll
      for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
        v_vecs[i] = CastFunctor<VQuantVecT, VVecT>()(*((reinterpret_cast<const VQuantVecT*>(v_ptr) + i * WARP_SIZE)));
      }

      if (token_idx >= context_len) {
        #pragma unroll
        for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
          v_vecs[i] = zero_value;
        }
      }

      logit = CastFunctor<float, scalar_t>()(logits[token_idx - start_token_idx]);
      #pragma unroll
      for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
        accs[i] = TernaryOpFunctor<scalar_t, VVecT, FloatVecT, TernaryOpType::kFma>()(logit, v_vecs[i], accs[i]);
      }
    }
  }

  // must insert a sync since both logits and out_shared_mem occupy the same buffer space
  __syncthreads();

  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    block_sum<FloatVecT, NUM_WARPS, NUM_THREADS_PER_TOKEN, VEC_SIZE>(out_shared_mem, accs[i]);
  }
}

// We only support head size of { 64, 128, 256 }
// models like Phi-2, whose head size is 80, is not supported right now
template<typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE, int NUM_THREADS>
__global__ void flash_decoding_attention_kernel_v1(
  scalar_t* __restrict__ out,                 // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ q,             // [num_tokens, num_heads, head_size]
  const cache_t* __restrict__ k_cache,        // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  const cache_t* __restrict__ v_cache,        // [num_blocks, num_kv_heads, block_size, head_size]
  const int* __restrict__ context_lens,       // [num_tokens]
  const int* __restrict__ block_tables,       // [num_tokens, max_num_blocks_per_seq]
  const float* __restrict__ alibi_slopes,     // [num_heads]
  const int max_seq_len,
  const int num_kv_heads,
  const float scale,
  const int max_num_blocks_per_seq,
  const int q_stride,                         // num_heads * head_size
  const int kv_block_stride,
  const int kv_head_stride) {
  const int seq_idx = blockIdx.y;
  const int head_idx = blockIdx.x;
  const int thread_idx = threadIdx.x;
  const int lane = thread_idx % WARP_SIZE;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  constexpr int x = sizeof(float4) / sizeof(scalar_t);
  constexpr int Q_SHARED_SIZE = HEAD_SIZE / x;
  // here thread_group does not determine the number of threads responsible for a key
  // but only the VEC_SIZE of each thread
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int VEC_SIZE = MIN(ROUND_DOWN_HIGHEST_POWER_OF_TWO((HEAD_SIZE / THREAD_GROUP_SIZE)), x);
  constexpr int NUM_VECS_PER_TOKEN = HEAD_SIZE / VEC_SIZE;
  constexpr int NUM_THREADS_PER_TOKEN = MIN(NUM_VECS_PER_TOKEN, WARP_SIZE);
  constexpr int NUM_ROUNDS_PER_TOKEN = NUM_VECS_PER_TOKEN / NUM_THREADS_PER_TOKEN;
  constexpr int WARP_STRIDE = WARP_SIZE * NUM_ROUNDS_PER_TOKEN;
  constexpr int NUM_THREADS_PER_X = x / VEC_SIZE;
  constexpr int NUM_ROWS_PER_ROUNDS = MIN(WARP_SIZE / NUM_THREADS_PER_X, BLOCK_SIZE);
  constexpr int NUM_VECS_PER_THREAD = NUM_ROWS_PER_ROUNDS * NUM_VECS_PER_TOKEN / WARP_SIZE;

  using KVecT = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using VVecT = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using KQuantVecT = typename VecTypeTrait<cache_t, VEC_SIZE>::Type;
  using VQuantVecT = typename VecTypeTrait<cache_t, VEC_SIZE>::Type;
  using LVecT = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using FloatVecT = typename FloatVecTypeTrait<LVecT>::Type;

  const int context_len = context_lens[seq_idx];
  const float alibi_slope = alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];
  const int thread_group_offset = lane % NUM_THREADS_PER_X;
  const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  const int shared_memory_offset = DIVIDE_ROUND_UP(max_num_blocks_per_seq * sizeof(int), sizeof(float4)) * sizeof(float4);

  __shared__ float4 q_shared[Q_SHARED_SIZE];
  __shared__ float red_shared_mem[2 * NUM_WARPS];
  extern __shared__ char shared_mem[];
  int* block_table_shared = reinterpret_cast<int*>(shared_mem);
  float* logits = reinterpret_cast<float*>(shared_mem + shared_memory_offset);
  float* out_shared_mem = reinterpret_cast<float*>(shared_mem + shared_memory_offset);
  float qk_max = -FLT_MAX;
  float exp_sum = 0.f;

  const float4* q_ptr = reinterpret_cast<const float4*>(q + seq_idx * q_stride + head_idx * HEAD_SIZE);
  scalar_t* q_shared_ptr = reinterpret_cast<scalar_t*>(q_shared);
  KVecT q_vecs[NUM_VECS_PER_THREAD];

  // 1. load query and block_table from global memory to shared memory
  data_load<scalar_t, KVecT, VEC_SIZE, Q_SHARED_SIZE, NUM_VECS_PER_THREAD, NUM_THREADS_PER_X, NUM_ROWS_PER_ROUNDS, NUM_VECS_PER_TOKEN, x>(q_ptr, q_shared, q_shared_ptr, q_vecs, block_table, block_table_shared, lane, max_num_blocks_per_seq);

  // 2. compute the dot product of query and key cache
  qk_gemv<scalar_t, cache_t, KVecT, KQuantVecT, NUM_WARPS, NUM_VECS_PER_THREAD, BLOCK_SIZE, NUM_ROWS_PER_ROUNDS, NUM_VECS_PER_TOKEN, NUM_THREADS_PER_X, x, VEC_SIZE>(k_cache, q_vecs, logits, block_table_shared, alibi_slope, context_len, qk_max, scale, kv_head_idx, warp_idx, lane, thread_group_offset, 0, num_context_blocks, 0, kv_block_stride, kv_head_stride);

  // 3. compute the softmax
  softmax<NUM_THREADS, NUM_WARPS, NUM_ROWS_PER_ROUNDS, NUM_THREADS_PER_X>(red_shared_mem, logits, qk_max, exp_sum, context_len);

  FloatVecT accs[NUM_ROUNDS_PER_TOKEN];

  // 4. compute the dot product of softmax tensor and value cache
  sv_gemv<scalar_t, cache_t, FloatVecT, VVecT, VQuantVecT, NUM_WARPS, NUM_ROUNDS_PER_TOKEN, NUM_THREADS_PER_TOKEN, BLOCK_SIZE, VEC_SIZE, NUM_VECS_PER_TOKEN, WARP_STRIDE>(v_cache, block_table_shared, out_shared_mem, logits, accs, lane, warp_idx, kv_head_idx, 0, num_context_blocks, context_len, 0, kv_block_stride, kv_head_stride);

  // 5. write back to global memory
  scalar_t* out_ptr = out + seq_idx * q_stride + head_idx * HEAD_SIZE;
  LVecT out_reg;
  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    if (thread_idx < NUM_THREADS_PER_TOKEN) {
      out_reg = CastFunctor<FloatVecT, LVecT>()(accs[i]);
      (reinterpret_cast<LVecT*>(out_ptr))[thread_idx + i * NUM_THREADS_PER_TOKEN] = out_reg;
    }
  }
}

#define LAUNCH_FLASH_DECODING_ATTENTION_V1(HEAD_SIZE)                                            \
  hipFuncSetAttribute(                                                                          \
    ((void*)flash_decoding_attention_kernel_v1<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>), \
    hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);                               \
  flash_decoding_attention_kernel_v1<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>             \
                       <<<grid, block, shared_mem_size, stream>>>(                               \
    reinterpret_cast<T*>(out.data_ptr()),                                                        \
    reinterpret_cast<T*>(query.data_ptr()),                                                      \
    reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),                                            \
    reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),                                          \
    context_lens.data_ptr<int>(),                                                                \
    block_tables.data_ptr<int>(),                                                                \
    alibi_slopes_ptr,                                                                            \
    max_context_len,                                                                             \
    num_kv_heads,                                                                                \
    scale,                                                                                       \
    max_num_blocks_per_seq,                                                                      \
    q_stride,                                                                                    \
    kv_block_stride,                                                                             \
    kv_head_stride);

template<
  typename T,
  typename CACHE_T,
  int BLOCK_SIZE,
  int NUM_THREADS = 128>
void flash_decoding_attention_v1_launcher(
  torch::Tensor& out,              // [num_tokens, num_heads, head_size]
  torch::Tensor& query,            // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,        // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,      // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& context_lens,     // [num_tokens]
  torch::Tensor& block_tables,     // [num_tokens, max_num_blocks_per_seq]
  int max_context_len,
  float scale,
  const c10::optional<torch::Tensor>& alibi_slopes) {
  int num_tokens = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int q_stride = query.stride(0);

  int max_num_blocks_per_seq = block_tables.size(1);

  int num_kv_heads = key_cache.size(1);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  const int VEC_SIZE = MIN(ROUND_DOWN_HIGHEST_POWER_OF_TWO((head_size / THREAD_GROUP_SIZE)), sizeof(float4) / sizeof(T));
  const int NUM_VECS_PER_TOKEN = head_size / VEC_SIZE;
  const int NUM_THREADS_PER_TOKEN = MIN(NUM_VECS_PER_TOKEN, WARP_SIZE);

  int padded_max_context_len = DIVIDE_ROUND_UP(max_context_len, BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_context_len * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * NUM_THREADS_PER_TOKEN * VEC_SIZE * sizeof(float);
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size) + DIVIDE_ROUND_UP(max_num_blocks_per_seq * sizeof(int), sizeof(float4)) * sizeof(float4);

  const float* alibi_slopes_ptr = alibi_slopes ?
    reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
    : nullptr;

  dim3 grid(num_heads, num_tokens, 1);
  dim3 block(NUM_THREADS);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model.
    case 64:
      LAUNCH_FLASH_DECODING_ATTENTION_V1(64);
      break;
    case 128:
      LAUNCH_FLASH_DECODING_ATTENTION_V1(128);
      break;
    case 256:
      LAUNCH_FLASH_DECODING_ATTENTION_V1(256);
      break;
    default:
      AT_ERROR("head size must be 64, 128, 256");
      break;
  }
}

#define CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE)                             \
  flash_decoding_attention_v1_launcher<T, CACHE_T, BLOCK_SIZE>(              \
    out,                                                                     \
    query,                                                                   \
    key_cache,                                                               \
    value_cache,                                                             \
    context_lens,                                                            \
    block_tables,                                                            \
    max_context_len,                                                         \
    scale,                                                                   \
    alibi_slopes);


template<typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE, int NUM_THREADS>
__global__ void flash_decoding_attention_kernel_v2(
  scalar_t* __restrict__ out,                 // [num_tokens, num_heads, max_num_partitions, head_size]
  float* __restrict__ exp_sums,               // [num_tokens, num_heads, max_num_partitions]
  float* __restrict__ max_logits,             // [num_tokens, num_heads, max_num_partitions]
  const scalar_t* __restrict__ q,             // [num_tokens, num_heads, head_size]
  const cache_t* __restrict__ k_cache,        // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  const cache_t* __restrict__ v_cache,        // [num_blocks, num_kv_heads, block_size, head_size]
  const int* __restrict__ context_lens,       // [num_tokens]
  const int* __restrict__ block_tables,       // [num_tokens, max_num_blocks_per_seq]
  const float* __restrict__ alibi_slopes,     // [num_heads]
  const int max_seq_len,
  const int num_kv_heads,
  const float scale,
  const int max_num_blocks_per_seq,
  const int q_stride,                         // num_heads * head_size
  const int tmp_stride,                       // num_heads * max_num_partitions
  const int kv_block_stride,
  const int kv_head_stride) {
  const int partition_idx = blockIdx.z;
  const int seq_idx = blockIdx.y;
  const int head_idx = blockIdx.x;
  const int thread_idx = threadIdx.x;
  const int lane = thread_idx % WARP_SIZE;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int max_num_partitions = gridDim.z;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  constexpr int x = sizeof(float4) / sizeof(scalar_t);
  constexpr int Q_SHARED_SIZE = HEAD_SIZE / x;
  // here thread_group does not determine the number of threads responsible for a key
  // but only the VEC_SIZE of each thread
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int VEC_SIZE = MIN(ROUND_DOWN_HIGHEST_POWER_OF_TWO((HEAD_SIZE / THREAD_GROUP_SIZE)), x);
  constexpr int NUM_VECS_PER_TOKEN = HEAD_SIZE / VEC_SIZE;
  constexpr int NUM_THREADS_PER_TOKEN = MIN(NUM_VECS_PER_TOKEN, WARP_SIZE);
  constexpr int NUM_ROUNDS_PER_TOKEN = NUM_VECS_PER_TOKEN / NUM_THREADS_PER_TOKEN;
  constexpr int WARP_STRIDE = WARP_SIZE * NUM_ROUNDS_PER_TOKEN;
  constexpr int NUM_THREADS_PER_X = x / VEC_SIZE;
  constexpr int NUM_ROWS_PER_ROUNDS = MIN(WARP_SIZE / NUM_THREADS_PER_X, BLOCK_SIZE);
  constexpr int NUM_VECS_PER_THREAD = NUM_ROWS_PER_ROUNDS * NUM_VECS_PER_TOKEN / WARP_SIZE;
  constexpr int NUM_BLOCKS_PER_PARTITION = PARTITION_SIZE / BLOCK_SIZE;

  using KVecT = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using VVecT = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using KQuantVecT = typename VecTypeTrait<cache_t, VEC_SIZE>::Type;
  using VQuantVecT = typename VecTypeTrait<cache_t, VEC_SIZE>::Type;
  using LVecT = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using FloatVecT = typename FloatVecTypeTrait<LVecT>::Type;

  const int context_len = context_lens[seq_idx];

  if (partition_idx * PARTITION_SIZE >= context_len) {
    return;
  }

  const float alibi_slope = alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];
  const int thread_group_offset = lane % NUM_THREADS_PER_X;
  const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);

  // [start_block_idx, end_block_idx) is the range of blocks to process.
  const int start_block_idx = partition_idx * NUM_BLOCKS_PER_PARTITION;
  const int end_block_idx = MIN(start_block_idx + NUM_BLOCKS_PER_PARTITION, num_context_blocks);
  const int num_blocks = end_block_idx - start_block_idx;

  // [start_token_idx, end_token_idx) is the range of tokens to process.
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx = MIN(start_token_idx + num_blocks * BLOCK_SIZE, context_len);
  const int num_tokens = end_token_idx - start_token_idx;

  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  const int shared_memory_offset = DIVIDE_ROUND_UP(max_num_blocks_per_seq * sizeof(int), sizeof(float4)) * sizeof(float4);

  __shared__ float4 q_shared[Q_SHARED_SIZE];
  __shared__ float red_shared_mem[2 * NUM_WARPS];
  extern __shared__ char shared_mem[];
  int* block_table_shared = reinterpret_cast<int*>(shared_mem);
  float* logits = reinterpret_cast<float*>(shared_mem + shared_memory_offset);
  float* out_shared_mem = reinterpret_cast<float*>(shared_mem + shared_memory_offset);
  float qk_max = -FLT_MAX;
  float exp_sum = 0.f;

  const float4* q_ptr = reinterpret_cast<const float4*>(q + seq_idx * q_stride + head_idx * HEAD_SIZE);
  scalar_t* q_shared_ptr = reinterpret_cast<scalar_t*>(q_shared);
  KVecT q_vecs[NUM_VECS_PER_THREAD];

  // 1. load query and block_table from global memory to shared memory
  data_load<scalar_t, KVecT, VEC_SIZE, Q_SHARED_SIZE, NUM_VECS_PER_THREAD, NUM_THREADS_PER_X, NUM_ROWS_PER_ROUNDS, NUM_VECS_PER_TOKEN, x>(q_ptr, q_shared, q_shared_ptr, q_vecs, block_table, block_table_shared, lane, max_num_blocks_per_seq);

  // 2. compute the dot product of query and key cache
  qk_gemv<scalar_t, cache_t, KVecT, KQuantVecT, NUM_WARPS, NUM_VECS_PER_THREAD, BLOCK_SIZE, NUM_ROWS_PER_ROUNDS, NUM_VECS_PER_TOKEN, NUM_THREADS_PER_X, x, VEC_SIZE>(k_cache, q_vecs, logits, block_table_shared, alibi_slope, context_len, qk_max, scale, kv_head_idx, warp_idx, lane, thread_group_offset, start_block_idx, end_block_idx, start_token_idx, kv_block_stride, kv_head_stride);

  // 3. compute the softmax
  softmax<NUM_THREADS, NUM_WARPS, NUM_ROWS_PER_ROUNDS, NUM_THREADS_PER_X>(red_shared_mem, logits, qk_max, exp_sum, num_tokens);

  if (thread_idx == 0) {
    float* max_logits_ptr = max_logits + seq_idx * tmp_stride
                                       + head_idx * max_num_partitions
                                       + partition_idx;
    float* exp_sums_ptr = exp_sums + seq_idx * tmp_stride
                                   + head_idx * max_num_partitions
                                   + partition_idx;
    *max_logits_ptr = qk_max;
    *exp_sums_ptr = exp_sum;
  }

  FloatVecT accs[NUM_ROUNDS_PER_TOKEN];

  // 4. compute the dot product of softmax tensor and value cache
  sv_gemv<scalar_t, cache_t, FloatVecT, VVecT, VQuantVecT, NUM_WARPS, NUM_ROUNDS_PER_TOKEN, NUM_THREADS_PER_TOKEN, BLOCK_SIZE, VEC_SIZE, NUM_VECS_PER_TOKEN, WARP_STRIDE>(v_cache, block_table_shared, out_shared_mem, logits, accs, lane, warp_idx, kv_head_idx, start_block_idx, end_block_idx, context_len, start_token_idx, kv_block_stride, kv_head_stride);

  // 5. write back to global memory
  scalar_t* out_ptr = out + seq_idx * q_stride * max_num_partitions
                          + head_idx * HEAD_SIZE * max_num_partitions
                          + partition_idx * HEAD_SIZE;
  LVecT out_reg;
  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    if (thread_idx < NUM_THREADS_PER_TOKEN) {
      out_reg = CastFunctor<FloatVecT, LVecT>()(accs[i]);
      (reinterpret_cast<LVecT*>(out_ptr))[thread_idx + i * NUM_THREADS_PER_TOKEN] = out_reg;
    }
  }
}

template<typename scalar_t, int HEAD_SIZE, int NUM_THREADS>
__global__ void flash_decoding_reduce_kernel(
  scalar_t* __restrict__ out,                 // [num_tokens, num_heads, head_size]
  float* __restrict__ exp_sums,               // [num_tokens, num_heads, max_num_partitions]
  float* __restrict__ max_logits,             // [num_tokens, num_heads, max_num_partitions]
  scalar_t* __restrict__ tmp_out,             // [num_tokens, num_heads, max_num_partitions, head_size]
  const int* __restrict__ context_lens,       // [num_tokens]
  const int out_stride,
  const int tmp_stride,
  const int max_num_partitions) {
  const int seq_idx = blockIdx.y;
  const int head_idx = blockIdx.x;

  const int context_len = context_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(context_len, PARTITION_SIZE);

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;

  extern __shared__ char shared_mem[];
  __shared__ float red_smem[2 * NUM_WARPS];
  float* shared_max_logits = reinterpret_cast<float*>(shared_mem);
  const float* max_logits_ptr = max_logits + seq_idx * tmp_stride
                                           + head_idx * max_num_partitions;

  float max_logit = -FLT_MAX;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    const float tmp_max_logit = max_logits_ptr[i];
    shared_max_logits[i] = tmp_max_logit;
    max_logit = fmaxf(max_logit, tmp_max_logit);
  }

  __syncthreads();

  max_logit = block_max<NUM_WARPS, WARP_SIZE, 1>(red_smem, max_logit);

  float* shared_exp_sums = reinterpret_cast<float*>(shared_mem + num_partitions * sizeof(float));
  const float* exp_sums_ptr = exp_sums + seq_idx * tmp_stride
                                       + head_idx * max_num_partitions;

  float global_exp_sum = 0.f;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    float tmp_max_logit = shared_max_logits[i];
    float rescaled_exp_sum = exp_sums_ptr[i] * expf(tmp_max_logit - max_logit);
    global_exp_sum += rescaled_exp_sum;
    shared_exp_sums[i] = rescaled_exp_sum;
  }

  __syncthreads();

  global_exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], global_exp_sum);
  const float inv_global_exp_sum = __fdividef(1.f, global_exp_sum + 1e-6f);

  const scalar_t* tmp_out_ptr = tmp_out + seq_idx * out_stride * max_num_partitions
                                        + head_idx * max_num_partitions * HEAD_SIZE;
  scalar_t* out_ptr = out + seq_idx * out_stride + head_idx * HEAD_SIZE;

  #pragma unroll
  for (int i = threadIdx.x; i < HEAD_SIZE; i += NUM_THREADS) {
    float acc = 0.f;
    for (int j = 0; j < num_partitions; j++) {
      acc += CastFunctor<scalar_t, float>()(tmp_out_ptr[j * HEAD_SIZE + i]) * shared_exp_sums[j] * inv_global_exp_sum;
    }
    out_ptr[i] = CastFunctor<float, scalar_t>()(acc);
  }
}


#define LAUNCH_FLASH_DECODING_ATTENTION_V2(HEAD_SIZE)                                            \
  hipFuncSetAttribute(                                                                          \
    ((void*)flash_decoding_attention_kernel_v2<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>), \
    hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);                               \
  flash_decoding_attention_kernel_v2<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>             \
                       <<<grid, block, shared_mem_size, stream>>>(                               \
    reinterpret_cast<T*>(tmp_out.data_ptr()),                                                    \
    reinterpret_cast<float*>(exp_sums.data_ptr()),                                               \
    reinterpret_cast<float*>(max_logits.data_ptr()),                                             \
    reinterpret_cast<T*>(query.data_ptr()),                                                      \
    reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),                                            \
    reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),                                          \
    reinterpret_cast<int*>(context_lens.data_ptr()),                                             \
    reinterpret_cast<int*>(block_tables.data_ptr()),                                             \
    alibi_slopes_ptr,                                                                            \
    max_context_len,                                                                             \
    num_kv_heads,                                                                                \
    scale,                                                                                       \
    max_num_blocks_per_seq,                                                                      \
    q_stride,                                                                                    \
    tmp_stride,                                                                                  \
    kv_block_stride,                                                                             \
    kv_head_stride);                                                                             \
  hipFuncSetAttribute(                                                                          \
    ((void*)flash_decoding_reduce_kernel<T, HEAD_SIZE, NUM_THREADS>),                            \
    hipFuncAttributeMaxDynamicSharedMemorySize, reduce_shared_mem_size);                        \
  flash_decoding_reduce_kernel<T, HEAD_SIZE, NUM_THREADS>                                        \
                       <<<reduce_grid, block, reduce_shared_mem_size, stream>>>(                 \
    reinterpret_cast<T*>(out.data_ptr()),                                                        \
    reinterpret_cast<float*>(exp_sums.data_ptr()),                                               \
    reinterpret_cast<float*>(max_logits.data_ptr()),                                             \
    reinterpret_cast<T*>(tmp_out.data_ptr()),                                                    \
    reinterpret_cast<int*>(context_lens.data_ptr()),                                             \
    q_stride,                                                                                    \
    tmp_stride,                                                                                  \
    max_num_partitions);


template<
  typename T,
  typename CACHE_T,
  int BLOCK_SIZE,
  int NUM_THREADS = 128>
void flash_decoding_attention_v2_launcher(
  torch::Tensor& out,              // [num_tokens, num_heads, head_size]
  torch::Tensor& exp_sums,         // [num_tokens, num_heads, max_num_partitions]
  torch::Tensor& max_logits,       // [num_tokens, num_heads, max_num_partitions]
  torch::Tensor& tmp_out,          // [num_tokens, num_heads, max_num_partitions, head_size]
  torch::Tensor& query,            // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,        // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,      // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& context_lens,     // [num_tokens]
  torch::Tensor& block_tables,     // [num_tokens, max_num_blocks_per_seq]
  int max_context_len,
  float scale,
  const c10::optional<torch::Tensor>& alibi_slopes) {
  int num_tokens = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int q_stride = query.stride(0);
  int tmp_stride = exp_sums.stride(0);

  int max_num_blocks_per_seq = block_tables.size(1);

  int num_kv_heads = key_cache.size(1);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  const int VEC_SIZE = MIN(ROUND_DOWN_HIGHEST_POWER_OF_TWO((head_size / THREAD_GROUP_SIZE)), sizeof(float4) / sizeof(T));
  const int NUM_VECS_PER_TOKEN = head_size / VEC_SIZE;
  const int NUM_THREADS_PER_TOKEN = MIN(NUM_VECS_PER_TOKEN, WARP_SIZE);

  int max_num_partitions = DIVIDE_ROUND_UP(max_context_len, PARTITION_SIZE);
  int logits_size = PARTITION_SIZE * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * NUM_THREADS_PER_TOKEN * VEC_SIZE * sizeof(float);
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size) + DIVIDE_ROUND_UP(max_num_blocks_per_seq * sizeof(int), sizeof(float4)) * sizeof(float4);

  const float* alibi_slopes_ptr = alibi_slopes ?
    reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
    : nullptr;

  dim3 grid(num_heads, num_tokens, max_num_partitions);
  dim3 block(NUM_THREADS);

  dim3 reduce_grid(num_heads, num_tokens);
  int reduce_shared_mem_size = 2 * max_num_partitions * sizeof(float);

  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model.
    case 64:
      LAUNCH_FLASH_DECODING_ATTENTION_V2(64);
      break;
    case 128:
      LAUNCH_FLASH_DECODING_ATTENTION_V2(128);
      break;
    case 256:
      LAUNCH_FLASH_DECODING_ATTENTION_V2(256);
      break;
    default:
      AT_ERROR("head size must be 64, 128, 256");
      break;
  }
}

#define CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE)                 \
  flash_decoding_attention_v2_launcher<T, CACHE_T, BLOCK_SIZE>(  \
    out,                                                         \
    exp_sums,                                                    \
    max_logits,                                                  \
    tmp_out,                                                     \
    query,                                                       \
    key_cache,                                                   \
    value_cache,                                                 \
    context_lens,                                                \
    block_tables,                                                \
    max_context_len,                                             \
    scale,                                                       \
    alibi_slopes);

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_LAUNCHER_BLOCK_SIZE(Version, T, CACHE_T)                 \
  switch (block_size) {                                               \
    case 8:                                                           \
      CALL_##Version##_LAUNCHER(T, CACHE_T, 8);                       \
      break;                                                          \
    case 16:                                                          \
      CALL_##Version##_LAUNCHER(T, CACHE_T, 16);                      \
      break;                                                          \
    case 32:                                                          \
      CALL_##Version##_LAUNCHER(T, CACHE_T, 32);                      \
      break;                                                          \
    default:                                                          \
      AT_ERROR("block size must be 8, 16, 32");                       \
      break;                                                          \
  }

#define CALL_LAUNCHER_DTYPE(Version)                                            \
  if(key_cache.scalar_type() == at::ScalarType::Byte)                           \
  {                                                                             \
    switch (query.scalar_type()) {                                              \
      case at::ScalarType::Float:                                               \
        CALL_LAUNCHER_BLOCK_SIZE(Version, float, uint8_t);                      \
        break;                                                                  \
      case at::ScalarType::Half:                                                \
        CALL_LAUNCHER_BLOCK_SIZE(Version, half, uint8_t);                       \
        break;                                                                  \
      case at::ScalarType::BFloat16:                                            \
        CALL_LAUNCHER_BLOCK_SIZE(Version, __hip_bfloat16, uint8_t);              \
        break;                                                                  \
    }                                                                           \
  }                                                                             \
  else                                                                          \
  {                                                                             \
    switch (query.scalar_type()) {                                              \
      case at::ScalarType::Float:                                               \
        CALL_LAUNCHER_BLOCK_SIZE(Version, float, float);                        \
        break;                                                                  \
      case at::ScalarType::Half:                                                \
        CALL_LAUNCHER_BLOCK_SIZE(Version, half, half);                          \
        break;                                                                  \
      case at::ScalarType::BFloat16:                                            \
        CALL_LAUNCHER_BLOCK_SIZE(Version, __hip_bfloat16, __hip_bfloat16);        \
        break;                                                                  \
    }                                                                           \
  }

void flash_decoding_attention(
  torch::Tensor& out,             // [num_tokens, num_heads, head_size]
  torch::Tensor& query,           // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,       // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,     // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& context_lens,    // [num_tokens]
  torch::Tensor& block_tables,    // [num_tokens, max_num_blocks_per_seq]
  int block_size,
  int max_context_len,
  torch::Tensor& tmp_out,         // [num_tokens, num_heads, max_num_partitions, head_size]
  torch::Tensor& exp_sums,        // [num_tokens, num_heads, max_num_partitions]
  torch::Tensor& max_logits,      // [num_tokens, num_heads, max_num_partitions]
  const c10::optional<torch::Tensor>& alibi_slopes,
  float scale) {

  int num_tokens = query.size(0);
  int num_heads = query.size(1);

  int max_num_partitions = DIVIDE_ROUND_UP(max_context_len, PARTITION_SIZE);
  // TODO(luoxiang): Need to be tuned
  bool use_v1 = max_context_len <= 8192 && (max_num_partitions == 1 || num_tokens * num_heads > 512);

  if (use_v1) {
    CALL_LAUNCHER_DTYPE(V1);
  } else {
    CALL_LAUNCHER_DTYPE(V2);
  }
}


#undef LAUNCH_FLASH_DECODING_ATTENTION_V1
#undef CALL_LAUNCHER
#undef CALL_LAUNCHER_BLOCK_SIZE
#undef CALL_LAUNCHER_DTYPE

#include "hip/hip_runtime.h"
/*This code adapted from vllm:
 *     https://github.com/vllm-project/vllm/blob/main/csrc/attention/attention_kernels.cu
 *     with different kvcache layout. */

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <stdio.h>

#include "common/micros.h"
#include "funcs/cast_functor.h"
#include "funcs/ternary_functor.h"
#include "funcs/binary_functor.h"
#include "common/vec_type_traits.h"
#include "attention/attention_utils.h"

#define WARP_SIZE 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))
// 2^n => 2^n, 2^n-d => 2^(n-1)
#define ROUND_DOWN_HIGHEST_POWER_OF_TWO(x) (nextHighestPowerOf2((x - (x + 1) / 2 + 1)))

// a bit magic, you can ask chatgpt for help
// 2^n => 2^n, 2^n-d => 2^n
constexpr unsigned int nextHighestPowerOf2(unsigned int v) {
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

using colossalAI::funcs::BinaryOpType;
using colossalAI::funcs::CastFunctor;
using colossalAI::funcs::TernaryOpFunctor;
using colossalAI::funcs::TernaryOpType;
using colossalAI::funcs::zero;
using colossalAI::common::VecTypeTrait;
using colossalAI::common::FloatVecTypeTrait;
using namespace colossalAI::cuda::attention;


// We only support head size of { 64, 128, 256 }
// models like Phi-2, whose head size is 80, is not supported right now
template<typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE, int NUM_THREADS>
__global__ void flash_decoding_attention_kernel(
  scalar_t* __restrict__ out,                 // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ q,             // [num_tokens, num_heads, head_size]
  const cache_t* __restrict__ k_cache,        // [num_blocks, num_kv_heads, block_size, head_size]
  const cache_t* __restrict__ v_cache,        // [num_blocks, num_kv_heads, block_size, head_size]
  const int* __restrict__ context_lens,       // [num_tokens]
  const int* __restrict__ block_tables,       // [num_tokens, max_num_blocks_per_seq]
  const int max_seq_len,
  const int num_kv_heads,
  const float scale,
  const int max_num_blocks_per_seq,
  const int q_stride,                         // num_heads * head_size
  const int kv_block_stride,
  const int kv_head_stride) {
  const int seq_idx = blockIdx.y;
  const int head_idx = blockIdx.x;
  const int thread_idx = threadIdx.x;
  const int lane = thread_idx % WARP_SIZE;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  constexpr int Q_SHARED_SIZE = (HEAD_SIZE * sizeof(scalar_t)) / sizeof(float4);
  // here thread_group does not determine the number of threads responsible for a key
  // but only the VEC_SIZE of each thread
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int VEC_SIZE = MIN(ROUND_DOWN_HIGHEST_POWER_OF_TWO((HEAD_SIZE / THREAD_GROUP_SIZE)), sizeof(float4) / sizeof(scalar_t));
  constexpr int NUM_VECS_PER_TOKEN = HEAD_SIZE / VEC_SIZE;
  constexpr int NUM_THREADS_PER_TOKEN = MIN(NUM_VECS_PER_TOKEN, WARP_SIZE);
  constexpr int NUM_ROUNDS_PER_TOKEN = NUM_VECS_PER_TOKEN / NUM_THREADS_PER_TOKEN;
  constexpr int WARP_STRIDE = WARP_SIZE * NUM_ROUNDS_PER_TOKEN;

  using K_vec = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using V_vec = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using L_vec = typename VecTypeTrait<scalar_t, VEC_SIZE>::Type;
  using Float_vec = typename FloatVecTypeTrait<L_vec>::Type;

  const int context_len = context_lens[seq_idx];
  const int thread_group_offset = thread_idx % NUM_THREADS_PER_TOKEN;
  const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;

  __shared__ float4 q_shared[Q_SHARED_SIZE];
  __shared__ float red_shared_mem[2 * NUM_WARPS];
  extern __shared__ char shared_mem[];
  float* logits = reinterpret_cast<float*>(shared_mem);
  float* out_shared_mem = reinterpret_cast<float*>(shared_mem);
  float qk_max = -FLT_MAX;

  const float4* q_ptr = reinterpret_cast<const float4*>(q + seq_idx * q_stride + head_idx * HEAD_SIZE);
  #pragma unroll
  for (int idx = thread_idx; idx < Q_SHARED_SIZE; idx += blockDim.x) {
    q_shared[idx] = q_ptr[idx];
  }
  __syncthreads();

  scalar_t* q_shared_ptr = reinterpret_cast<scalar_t*>(q_shared);
  // each warp access a whole block
  for (int block_idx = warp_idx; block_idx < num_context_blocks; block_idx += NUM_WARPS) {
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);
    #pragma unroll
    for (int idx = lane; idx < BLOCK_SIZE * NUM_VECS_PER_TOKEN; idx += WARP_STRIDE) {
      const int token_idx = block_idx * BLOCK_SIZE + idx / NUM_VECS_PER_TOKEN;
      const cache_t* k_ptr = k_cache + physical_block_number * kv_block_stride
                                     + kv_head_idx * kv_head_stride
                                     + idx * VEC_SIZE;

      K_vec k_vecs[NUM_ROUNDS_PER_TOKEN];
      K_vec q_vecs[NUM_ROUNDS_PER_TOKEN];

      // we must calculate at least one row of hidden vectors
      #pragma unroll
      for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
        k_vecs[i] = (reinterpret_cast<const K_vec*>(k_ptr))[i * WARP_SIZE];
        q_vecs[i] = (reinterpret_cast<K_vec*>(q_shared_ptr))[(idx + i * WARP_SIZE) % NUM_VECS_PER_TOKEN];
      }

      float qk = scale * Qk_dot<scalar_t, NUM_THREADS_PER_TOKEN>::dot(q_vecs, k_vecs);

      if (thread_group_offset == 0) {
        const bool mask = token_idx >= context_len;
        logits[token_idx] = mask ? 0.f : qk;
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }

  // there exists a __syncthreads within this function
  qk_max = block_max<NUM_WARPS, NUM_THREADS_PER_TOKEN>(red_shared_mem, qk_max);

  // Get the sum of the exp values.
  float exp_sum = 0.f;
  for (int i = thread_idx; i < context_len; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }

  exp_sum = block_sum<NUM_WARPS>(&red_shared_mem[NUM_WARPS], exp_sum);
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < context_len; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();

  Float_vec accs[NUM_ROUNDS_PER_TOKEN];
  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    zero(accs[i]);
  }

  V_vec zero_value;
  zero(zero_value);
  for (int block_idx = warp_idx; block_idx < num_context_blocks; block_idx += NUM_WARPS) {
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);
    scalar_t logit;

    #pragma unroll
    for (int idx = lane; idx < BLOCK_SIZE * NUM_VECS_PER_TOKEN; idx += WARP_STRIDE) {
      const int token_idx = block_idx * BLOCK_SIZE + idx / NUM_VECS_PER_TOKEN;
      const cache_t* v_ptr = v_cache + physical_block_number * kv_block_stride
                                     + kv_head_idx * kv_head_stride
                                     + idx * VEC_SIZE;

      V_vec v_vecs[NUM_ROUNDS_PER_TOKEN];

      #pragma unroll
      for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
        v_vecs[i] = (reinterpret_cast<const V_vec*>(v_ptr))[i * WARP_SIZE];
      }

      if (token_idx >= context_len) {
        #pragma unroll
        for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
          v_vecs[i] = zero_value;
        }
      }

      logit = CastFunctor<float, scalar_t>()(logits[token_idx]);
      #pragma unroll
      for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
        accs[i] = TernaryOpFunctor<scalar_t, V_vec, Float_vec, TernaryOpType::kFma>()(logit, v_vecs[i], accs[i]);
      }
    }
  }

  // must insert a sync since both logits and out_shared_mem occupy the same buffer space
  __syncthreads();

  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    block_sum<Float_vec, NUM_WARPS, NUM_THREADS_PER_TOKEN, VEC_SIZE>(out_shared_mem, accs[i]);
  }

  scalar_t* out_ptr = out + seq_idx * q_stride + head_idx * HEAD_SIZE;
  L_vec out_reg;
  #pragma unroll
  for (int i = 0; i < NUM_ROUNDS_PER_TOKEN; i++) {
    if (thread_idx < NUM_THREADS_PER_TOKEN) {
      out_reg = CastFunctor<Float_vec, L_vec>()(accs[i]);
      (reinterpret_cast<L_vec*>(out_ptr))[thread_idx + i * NUM_THREADS_PER_TOKEN] = out_reg;
    }
  }
}

#define LAUNCH_FLASH_DECODING_ATTENTION_V1(HEAD_SIZE)                                         \
  hipFuncSetAttribute(                                                                       \
    ((void*)flash_decoding_attention_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>), \
    hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);                            \
  flash_decoding_attention_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>             \
                       <<<grid, block, shared_mem_size, stream>>>(                            \
    reinterpret_cast<T*>(out.data_ptr()),                                                     \
    reinterpret_cast<T*>(query.data_ptr()),                                                   \
    reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),                                         \
    reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),                                       \
    context_lens.data_ptr<int>(),                                                             \
    block_tables.data_ptr<int>(),                                                             \
    max_context_len,                                                                          \
    num_kv_heads,                                                                             \
    scale,                                                                                    \
    max_num_blocks_per_seq,                                                                   \
    q_stride,                                                                                 \
    kv_block_stride,                                                                          \
    kv_head_stride);

template<
  typename T,
  typename CACHE_T,
  int BLOCK_SIZE,
  int NUM_THREADS = 128>
void flash_decoding_attention_v1_launcher(
  torch::Tensor& out,              // [num_tokens, num_heads, head_size]
  torch::Tensor& query,            // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,        // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& value_cache,      // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& context_lens,     // [num_tokens]
  torch::Tensor& block_tables,     // [num_tokens, max_num_blocks_per_seq]
  int max_context_len,
  float scale) {
  int num_tokens = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int num_kv_heads = key_cache.size(1);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  const int VEC_SIZE = MIN(ROUND_DOWN_HIGHEST_POWER_OF_TWO((head_size / THREAD_GROUP_SIZE)), sizeof(float4) / sizeof(T));
  const int NUM_VECS_PER_TOKEN = head_size / VEC_SIZE;
  const int NUM_THREADS_PER_TOKEN = MIN(NUM_VECS_PER_TOKEN, WARP_SIZE);

  int padded_max_context_len = DIVIDE_ROUND_UP(max_context_len, BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_context_len * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * NUM_THREADS_PER_TOKEN * VEC_SIZE * sizeof(float);
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size);

  dim3 grid(num_heads, num_tokens, 1);
  dim3 block(NUM_THREADS);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model.
    case 64:
      LAUNCH_FLASH_DECODING_ATTENTION_V1(64);
      break;
    case 128:
      LAUNCH_FLASH_DECODING_ATTENTION_V1(128);
      break;
    case 256:
      LAUNCH_FLASH_DECODING_ATTENTION_V1(256);
      break;
    default:
      AT_ERROR("head size must be 64, 128, 256");
      break;
  }
}

#define CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE)                             \
  flash_decoding_attention_v1_launcher<T, CACHE_T, BLOCK_SIZE>(              \
    out,                                                                     \
    query,                                                                   \
    key_cache,                                                               \
    value_cache,                                                             \
    context_lens,                                                            \
    block_tables,                                                            \
    max_context_len,                                                         \
    scale);

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V1_LAUNCHER_BLOCK_SIZE(T, CACHE_T)                       \
  switch (block_size) {                                               \
    case 8:                                                           \
      CALL_V1_LAUNCHER(T, CACHE_T, 8);                                \
      break;                                                          \
    case 16:                                                          \
      CALL_V1_LAUNCHER(T, CACHE_T, 16);                               \
      break;                                                          \
    case 32:                                                          \
      CALL_V1_LAUNCHER(T, CACHE_T, 32);                               \
      break;                                                          \
    default:                                                          \
      AT_ERROR("block size must be 8, 16, 32");                       \
      break;                                                          \
  }

void flash_decoding_attention(
  torch::Tensor& out,             // [num_tokens, num_heads, head_size]
  torch::Tensor& query,           // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,       // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& value_cache,     // [num_blocks, num_kv_heads, block_size, head_size]
  torch::Tensor& context_lens,    // [num_tokens]
  torch::Tensor& block_tables,    // [num_tokens, max_num_blocks_per_seq]
  int block_size,
  int max_context_len,
  torch::Tensor& tmp_out,         // [num_tokens, num_heads, max_num_partitions, head_size]
  torch::Tensor& tmp_out_lse,     // [num_tokens, num_heads, max_num_partitions]
  float scale) {
  switch (query.scalar_type()) {
    case at::ScalarType::Float:
      CALL_V1_LAUNCHER_BLOCK_SIZE(float, float);
      break;
    case at::ScalarType::Half:
      CALL_V1_LAUNCHER_BLOCK_SIZE(half, half);
      break;
    case at::ScalarType::BFloat16:
      CALL_V1_LAUNCHER_BLOCK_SIZE(__hip_bfloat16, __hip_bfloat16);
      break;
    default:
      AT_ERROR("Unsupported data type: ", toString(query.scalar_type()));
  }
}


#undef LAUNCH_FLASH_DECODING_ATTENTION_V1
#undef CALL_V1_LAUNCHER
#undef CALL_V1_LAUNCHER_BLOCK_SIZE

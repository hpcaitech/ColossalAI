/* coding=utf-8
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "scaled_masked_softmax.h"
#include "type_shim.h"



int get_batch_per_block_cuda(int query_seq_len, int key_seq_len, int batches, int attn_heads){
    return get_batch_per_block(query_seq_len, key_seq_len, batches, attn_heads);
}


torch::Tensor fwd_cuda(
    torch::Tensor const& input,
    torch::Tensor const& mask,
    float scale_factor)
{
  // input is a 4d tensor with dimensions [batches, attn_heads, seq_len, seq_len]
  const int batches = input.size(0);
  const int pad_batches = mask.size(0);
  const int attn_heads = input.size(1);
  const int query_seq_len = input.size(2);
  const int key_seq_len = input.size(3);
  TORCH_INTERNAL_ASSERT(key_seq_len <= 8192);
  TORCH_INTERNAL_ASSERT(query_seq_len > 1);
  TORCH_INTERNAL_ASSERT(pad_batches == 1 || pad_batches == batches);
  TORCH_INTERNAL_ASSERT(mask.size(1) == 1);
  TORCH_INTERNAL_ASSERT(mask.size(2) == query_seq_len);
  TORCH_INTERNAL_ASSERT(mask.size(3) == key_seq_len);

  // Output 
  auto act_options = input.options().requires_grad(false);
  torch::Tensor softmax_results = 
      torch::empty({batches, attn_heads, query_seq_len, key_seq_len}, act_options);

  // Softmax Intermediate Result Ptr
  void* input_ptr = static_cast<void*>(input.data_ptr());
  void* mask_ptr = static_cast<void*>(mask.data_ptr());
  void* softmax_results_ptr = static_cast<void*>(softmax_results.data_ptr());

  DISPATCH_HALF_AND_BFLOAT(
      input.scalar_type(),
      "dispatch_scaled_masked_softmax_forward",
      dispatch_scaled_masked_softmax_forward<scalar_t, scalar_t, float>(
          reinterpret_cast<scalar_t*>(softmax_results_ptr),
          reinterpret_cast<const scalar_t*>(input_ptr),
          reinterpret_cast<const uint8_t*>(mask_ptr),
          scale_factor,
          query_seq_len,
          key_seq_len,
          batches,
          attn_heads,
          pad_batches
      );
  );
  return softmax_results;
}

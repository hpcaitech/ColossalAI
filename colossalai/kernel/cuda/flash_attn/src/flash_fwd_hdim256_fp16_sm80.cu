// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<> void run_mha_fwd_<cutlass::half_t, 256>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim256<cutlass::half_t>(params, stream);
}

// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::half_t, 32>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::half_t;
//     BOOL_SWITCH(params.p_dropout < 1.f, Is_dropout, [&] {
//         run_flash_fwd<Flash_fwd_kernel_traits<32, 128, 128, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         // For dropout there might be a lot of register spilling?
//         // These two are very slow due to register spilling
//         // run_flash_fwd<Flash_fwd_kernel_traits<32, 256, 128, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<32, 128, 256, 4, false, elem_type>>(params, stream);
//         // This one is slightly slower
//         // run_flash_fwd<Flash_fwd_kernel_traits<32, 256, 64, 4, false, elem_type>>(params, stream);
//     });
// }
template<>
void run_mha_fwd_<cutlass::half_t, 32>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim32<cutlass::half_t>(params, stream);
}
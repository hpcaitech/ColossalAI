// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::half_t, 96>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::half_t;
//     BOOL_SWITCH(params.p_dropout < 1.f, Is_dropout, [&] {
//         run_flash_fwd<Flash_fwd_kernel_traits<96, 128, 64, 4, true, false, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<96, 128, 64, 4, true, true, elem_type>, Is_dropout>(params, stream);
//         // This 3rd one is good for H100, and A100, A6000
//         run_flash_fwd<Flash_fwd_kernel_traits<96, 128, 64, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<96, 128, 64, 4, false, true, elem_type>, Is_dropout>(params, stream);
//         // These two are always slower
//         // run_flash_fwd<Flash_fwd_kernel_traits<96, 128, 128, 4, true, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<96, 64, 128, 4, true, elem_type>>(params, stream);
//     });
// }
template<> void run_mha_fwd_<cutlass::half_t, 96>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim96<cutlass::half_t>(params, stream);
}
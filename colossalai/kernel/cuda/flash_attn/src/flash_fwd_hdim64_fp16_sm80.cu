// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::half_t, 64>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::half_t;
//     if (params.p_dropout == 1.f) {
//         // Using 8 warps is 18% slower for seqlen=2k, 2 warps is 5% slower
//         // Using block size (64 x 256) is 27% slower for seqlen=2k
//         // Using block size (256 x 64) is 85% slower for seqlen=2k, because of register spilling
//         run_flash_fwd<Flash_fwd_kernel_traits<64, 128, 128, 4, false, false, elem_type>, false>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<64, 128, 64, 4, true, false, elem_type>, false>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<64, 128, 64, 4, true, true, elem_type>, false>(params, stream);
//     } else {
//         run_flash_fwd<Flash_fwd_kernel_traits<64, 128, 64, 4, false, false, elem_type>, true>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<64, 128, 64, 4, true, true, elem_type>, true>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<64, 128, 64, 4, true, false, elem_type>, true>(params, stream);
//     }
// }
template<>
void run_mha_fwd_<cutlass::half_t, 64>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim64<cutlass::half_t>(params, stream);
}
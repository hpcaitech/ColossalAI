// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<> void run_mha_fwd_<cutlass::bfloat16_t, 256>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim256<cutlass::bfloat16_t>(params, stream);
}

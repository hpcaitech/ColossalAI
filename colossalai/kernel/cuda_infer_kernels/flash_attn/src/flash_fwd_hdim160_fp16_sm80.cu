// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::half_t, 160>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::half_t;
//     BOOL_SWITCH(params.p_dropout < 1.f, Is_dropout, [&] {
//         run_flash_fwd<Flash_fwd_kernel_traits<160, 128, 32, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<160, 128, 32, 4, false, true, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<160, 128, 64, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<160, 64, 64, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<160, 128, 64, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<160, 64, 128, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<160, 64, 64, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<160, 128, 64, 8, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<160, 128, 128, 8, false, elem_type>>(params, stream);
//         // For A6000, no-causal, 1st is fastest. causal, 4th is fastest.
//         // For A100, H100, 1st is fastest.
//     });
// }
template<>
void run_mha_fwd_<cutlass::half_t, 160>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim160<cutlass::half_t>(params, stream);
}
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<> void run_mha_fwd_<cutlass::half_t, 224>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim224<cutlass::half_t>(params, stream);
}

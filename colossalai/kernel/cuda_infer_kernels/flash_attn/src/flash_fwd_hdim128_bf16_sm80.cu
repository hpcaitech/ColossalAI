// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::bfloat16_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::bfloat16_t;
//     if (params.p_dropout == 1.f) {
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 64, 4, false, false, elem_type>, false>(params, stream);
//     } else {
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 32, 4, false, false, elem_type>, true>(params, stream);
//     }
// }
template<>
void run_mha_fwd_<cutlass::bfloat16_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128<cutlass::bfloat16_t>(params, stream);
}
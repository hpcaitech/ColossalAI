// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::half_t, 192>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::half_t;
//     BOOL_SWITCH(params.p_dropout < 1.f, Is_dropout, [&] {
//         run_flash_fwd<Flash_fwd_kernel_traits<192, 64, 64, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<192, 128, 32, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<192, 64, 32, 4, false, false, elem_type>, Is_dropout>(params, stream);
//         // This one is slightly faster for causal?
//         // run_flash_fwd<Flash_fwd_kernel_traits<192, 128, 64, 8, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<192, 128, 32, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<192, 128, 64, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<192, 64, 128, 4, false, elem_type>>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<192, 128, 128, 8, false, elem_type>>(params, stream);
//     });
//     // For A100 H100, 1st is faster with dropout, 3rd is faster without dropout
//     // For A6000, 1st is faster when causal, 3rd is faster when not causal
// }
template<>
void run_mha_fwd_<cutlass::half_t, 192>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim192<cutlass::half_t>(params, stream);
}
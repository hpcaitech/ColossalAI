#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>

void dense_layer_fp32_kernel(const float *in, const float *weight, float *out, const int M,
                                 const int K, const int N, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const float alpha = 1.0f, beta = 0.0f;
  hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weight,
                                HIP_R_32F, N, in, HIP_R_32F, K, &beta, out, HIP_R_32F, N,
                                HIP_R_32F, static_cast<hipblasGemmAlgo_t>(cublasAlgo));
}

void dense_layer_fp16_kernel(const __half *in, const __half *weight, __half *out, const int M,
                                 const int K, const int N, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const __half alpha = (__half)1.0f, beta = (__half)0.0f;
  hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weight,
                                HIP_R_16F, N, in, HIP_R_16F, K, &beta, out, HIP_R_16F, N,
                                HIP_R_16F, static_cast<hipblasGemmAlgo_t>(cublasAlgo));
}


void cublas_Gemm_Strided_Batched_FP16_Kernel(const __half *A, const __half *B, __half *out, const int M,
                                 const int K, const int N, const int batch_count,
                                 hipblasOperation_t trans_A, hipblasOperation_t trans_B,
                                 __half alpha, __half beta, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;
  

  hipblasGemmStridedBatchedEx(
      cublas_handle, trans_B, trans_A, N, M, K, &alpha, B, HIP_R_16F, ldb, K * N, A, HIP_R_16F,
      lda, M * K, &beta, out, HIP_R_16F, N, M * N, batch_count, HIP_R_16F,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo));
}

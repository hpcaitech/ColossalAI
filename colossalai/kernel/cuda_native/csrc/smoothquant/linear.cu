// modified from https://github.com/Guangxuan-Xiao/torch-int/blob/main/torch_int/kernels/linear.cu

#include "linear.h"
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/half.h>

#include <cutlass/gemm/device/gemm.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/epilogue/thread/linear_combination_silu.h>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <torch/torch.h>
torch::Tensor linear_silu_a8_w8_bfp32_ofp32(torch::Tensor input,  // INT8
                                       torch::Tensor weight, // INT8
                                       torch::Tensor bias,   // FP32
                                       float alpha,          // FP32
                                       float beta            // FP32
) {
  auto M = input.size(0);
  auto N = weight.size(0);
  auto K = input.size(1);

  using ElementOutput = float;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

#if CUDA_ARCH  >= 800
  using EpilogueOp = cutlass::epilogue::thread::LinearCombinationSilu<
      ElementOutput, // <- data type of output matrix
      128 / cutlass::sizeof_bits<
                ElementOutput>::value, // <- this is the number of elements per
                                       // vectorized memory access. For half
                                       // precision, it's 8 elements. This
                                       // becomes the vector width of math
                                       // instructions in epilogue too
      ElementAccumulator,              // <- data type of accumulator
      ElementComputeEpilogue // <- data type for alpha in linear combination
                             // function
      >;
  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
	  EpilogueOp,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3>;
#elif CUDA_ARCH  >= 750
  using EpilogueOp = cutlass::epilogue::thread::LinearCombinationSilu<
      ElementOutput, // <- data type of output matrix
      128 / cutlass::sizeof_bits<
                ElementOutput>::value, // <- this is the number of elements per
                                       // vectorized memory access. For half
                                       // precision, it's 8 elements. This
                                       // becomes the vector width of math
                                       // instructions in epilogue too
      ElementAccumulator,              // <- data type of accumulator
      ElementComputeEpilogue // <- data type for alpha in linear combination
                             // function
      >;

  using DefaultGemmCfg = cutlass::gemm::device::DefaultGemmConfiguration<
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,
      ElementInputA, ElementInputB, ElementOutput, ElementAccumulator>;
  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,
      DefaultGemmCfg::ThreadblockShape, DefaultGemmCfg::WarpShape,
      DefaultGemmCfg::InstructionShape,
	EpilogueOp>;
#elif CUDA_ARCH  >= 700
  #define USE_TORCH_SILU
  using DefaultGemmCfg = cutlass::gemm::device::DefaultGemmConfiguration<
      cutlass::arch::OpClassSimt, cutlass::arch::Sm70,
      ElementInputA, ElementInputB, ElementOutput, ElementAccumulator>;
  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassSimt, cutlass::arch::Sm70,
      DefaultGemmCfg::ThreadblockShape, DefaultGemmCfg::WarpShape,
      DefaultGemmCfg::InstructionShape,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 1, ElementAccumulator, ElementComputeEpilogue>>;
#else
  #error "Unsupported cuda arch"
#endif

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  auto device = input.device();
  // use the broadcasted bias as the output
  auto out = bias.to(device).view({1, -1}).repeat({M, 1});

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      input.data_ptr<ElementInputA>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      weight.data_ptr<ElementInputB>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      out.data_ptr<ElementOutput>(), LayoutOutput::packed(output_size));

  typename Gemm::Arguments arguments{
      problem_size, // <- problem size of matrix multiplication
      input_ref,    // <- reference to matrix A on device
      weight_ref,   // <- reference to matrix B on device
      out_ref,      // <- reference to matrix C on device
      out_ref,      // <- reference to matrix D on device
      {alpha, beta}, 1};
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }
#ifdef USE_TORCH_SILU
#undef USE_TORCH_SILU
  out = torch::silu(out);
#endif
  return out;
}

// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

// template<>
// void run_mha_fwd_<cutlass::half_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
//     using elem_type = cutlass::half_t;
//     if (params.p_dropout == 1.f) {
//         // Using 8 warps (128 x 128 and 256 x 64) is 28% slower for seqlen=2k
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 64, 4, false, false, elem_type>, false>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 64, 4, true, false, elem_type>, false>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 64, 4, false, true, elem_type>, false>(params, stream);
//         // run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 64, 4, true, true, elem_type>, false>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 32, 4, false, false, elem_type>, false>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 64, 64, 4, false, false, elem_type>, false>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 64, 128, 4, false, false, elem_type>, false>(params, stream);
//         // 1st ones are good for H100, A100
//         // 2nd one is good for A6000 bc we get slightly better occupancy
//     } else {
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 32, 4, false, false, elem_type>, true>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 32, 4, true, false, elem_type>, true>(params, stream);
//         run_flash_fwd<Flash_fwd_kernel_traits<128, 128, 32, 4, true, true, elem_type>, true>(params, stream);
//         // 1st one is good for H100, A100, A6000
//     }
// }

template<>
void run_mha_fwd_<cutlass::half_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128<cutlass::half_t>(params, stream);
}
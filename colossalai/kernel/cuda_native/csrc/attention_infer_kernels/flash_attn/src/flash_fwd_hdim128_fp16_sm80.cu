/******************************************************************************
 * The following codes are modified from the original FlashAttn library: https://github.com/Dao-AILab/flash-attention
 ******************************************************************************/

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"


template<>
void run_mha_fwd_<cutlass::half_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128<cutlass::half_t>(params, stream);
}
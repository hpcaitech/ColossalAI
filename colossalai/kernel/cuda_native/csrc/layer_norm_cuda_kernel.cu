#include "hip/hip_runtime.h"
/*This code from NVIDIA apex:
 *     https://github.com/NVIDIA/apex
 *     with minor changes. */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/DeviceUtils.cuh"
#include "type_shim.h"

template <typename U>
__device__ void cuWelfordOnlineSum(const U curr, U& mu, U& sigma2, U& count) {
  count = count + U(1);
  U delta = curr - mu;
  U lmean = mu + delta / count;
  mu = lmean;
  U delta2 = curr - lmean;
  sigma2 = sigma2 + delta * delta2;
}

template <typename U>
__device__ void cuChanOnlineSum(const U muB, const U sigma2B, const U countB,
                                U& mu, U& sigma2, U& count) {
  U delta = muB - mu;
  U nA = count;
  U nB = countB;
  count = count + countB;
  U nX = count;
  if (nX > U(0)) {
    nA = nA / nX;
    nB = nB / nX;
    mu = nA * mu + nB * muB;
    sigma2 = sigma2 + sigma2B + delta * delta * nA * nB * nX;
  } else {
    mu = U(0);
    sigma2 = U(0);
  }
}

template <typename T, typename U>
__device__ void cuWelfordMuSigma2(const T* __restrict__ vals, const int n1,
                                  const int n2, const int i1, U& mu, U& sigma2,
                                  U* buf) {
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  U count = U(0);
  mu = U(0);
  sigma2 = U(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const T* lvals = vals + i1 * n2;
    int l = 4 * thrx;
    for (; l + 3 < n2; l += 4 * numx) {
      for (int k = 0; k < 4; ++k) {
        U curr = static_cast<U>(lvals[l + k]);
        cuWelfordOnlineSum<U>(curr, mu, sigma2, count);
      }
    }
    for (; l < n2; ++l) {
      U curr = static_cast<U>(lvals[l]);
      cuWelfordOnlineSum<U>(curr, mu, sigma2, count);
    }
    // intra-warp reductions
    for (int l = 0; l <= 4; ++l) {
      int srcLaneB = (threadIdx.x + (1 << l)) & 31;
      U muB = WARP_SHFL(mu, srcLaneB);
      U countB = WARP_SHFL(count, srcLaneB);
      U sigma2B = WARP_SHFL(sigma2, srcLaneB);
      cuChanOnlineSum<U>(muB, sigma2B, countB, mu, sigma2, count);
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      U* ubuf = (U*)buf;
      U* ibuf = (U*)(ubuf + blockDim.y);
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset &&
            threadIdx.y < 2 * offset) {
          const int wrt_y = threadIdx.y - offset;
          ubuf[2 * wrt_y] = mu;
          ubuf[2 * wrt_y + 1] = sigma2;
          ibuf[wrt_y] = count;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          U muB = ubuf[2 * threadIdx.y];
          U sigma2B = ubuf[2 * threadIdx.y + 1];
          U countB = ibuf[threadIdx.y];
          cuChanOnlineSum<U>(muB, sigma2B, countB, mu, sigma2, count);
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        ubuf[0] = mu;
        ubuf[1] = sigma2;
      }
      __syncthreads();
      mu = ubuf[0];
      sigma2 = ubuf[1] / U(n2);
      // don't care about final value of count, we know count == n2
    } else {
      mu = WARP_SHFL(mu, 0);
      sigma2 = WARP_SHFL(sigma2 / U(n2), 0);
    }
  }
}

template <>
__device__ void cuWelfordMuSigma2(const at::Half* __restrict__ vals,
                                  const int n1, const int n2, const int i1,
                                  float& mu, float& sigma2, float* buf) {
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  float count = 0.0f;
  mu = float(0);
  sigma2 = float(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const at::Half* lvals = vals + i1 * n2;
    int l = 8 * thrx;
    if ((((size_t)lvals) & 3) != 0) {
      // 16 bit alignment
      // first thread consumes first point
      if (thrx == 0) {
        float curr = static_cast<float>(lvals[0]);
        cuWelfordOnlineSum(curr, mu, sigma2, count);
      }
      ++l;
    }
    // at this point, lvals[l] are 32 bit aligned for all threads.
    for (; l + 7 < n2; l += 8 * numx) {
      for (int k = 0; k < 8; k += 2) {
        float2 curr = __half22float2(*((__half2*)(lvals + l + k)));
        cuWelfordOnlineSum(curr.x, mu, sigma2, count);
        cuWelfordOnlineSum(curr.y, mu, sigma2, count);
      }
    }
    for (; l < n2; ++l) {
      float curr = static_cast<float>(lvals[l]);
      cuWelfordOnlineSum(curr, mu, sigma2, count);
    }
    // intra-warp reductions
    for (int l = 0; l <= 4; ++l) {
      int srcLaneB = (threadIdx.x + (1 << l)) & 31;
      float muB = WARP_SHFL(mu, srcLaneB);
      float countB = WARP_SHFL(count, srcLaneB);
      float sigma2B = WARP_SHFL(sigma2, srcLaneB);
      cuChanOnlineSum(muB, sigma2B, countB, mu, sigma2, count);
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      float* ubuf = (float*)buf;
      float* ibuf = (float*)(ubuf + blockDim.y);
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset &&
            threadIdx.y < 2 * offset) {
          const int wrt_y = threadIdx.y - offset;
          ubuf[2 * wrt_y] = mu;
          ubuf[2 * wrt_y + 1] = sigma2;
          ibuf[wrt_y] = count;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          float muB = ubuf[2 * threadIdx.y];
          float sigma2B = ubuf[2 * threadIdx.y + 1];
          float countB = ibuf[threadIdx.y];
          cuChanOnlineSum(muB, sigma2B, countB, mu, sigma2, count);
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        ubuf[0] = mu;
        ubuf[1] = sigma2;
      }
      __syncthreads();
      mu = ubuf[0];
      sigma2 = ubuf[1] / float(n2);
      // don't care about final value of count, we know count == n2
    } else {
      mu = WARP_SHFL(mu, 0);
      sigma2 = WARP_SHFL(sigma2 / float(n2), 0);
    }
  }
}

template <typename U>
U rsqrt(U v) {
  return U(1) / sqrt(v);
}
template <>
float rsqrt(float v) {
  return rsqrtf(v);
}
template <>
double rsqrt(double v) {
  return rsqrt(v);
}

namespace {
// This is the un-specialized struct.  Note that we prevent instantiation of
// this struct by putting an undefined symbol in the function body so it won't
// compile.
//  template <typename T>
//  struct SharedMemory
//  {
//      // Ensure that we won't compile any un-specialized types
//      __device__ T *getPointer()
//      {
//          extern __device__ void error(void);
//          error();
//          return NULL;
//      }
//  };
// https://github.com/NVIDIA/apex/issues/246
template <typename T>
struct SharedMemory;

template <>
struct SharedMemory<float> {
  __device__ float* getPointer() {
    extern __shared__ float s_float[];
    return s_float;
  }
};

}  // namespace

template <typename T, typename U, typename V>
__global__ void cuApplyLayerNorm(V* __restrict__ output_vals,
                                 U* __restrict__ mean, U* __restrict__ invvar,
                                 const T* __restrict__ vals, const int n1,
                                 const int n2, const U epsilon,
                                 const V* __restrict__ gamma,
                                 const V* __restrict__ beta) {
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
    SharedMemory<U> shared;
    U* buf = shared.getPointer();
    U mu, sigma2;
    cuWelfordMuSigma2(vals, n1, n2, i1, mu, sigma2, buf);
    const T* lvals = vals + i1 * n2;
    V* ovals = output_vals + i1 * n2;
    U c_invvar = rsqrt(sigma2 + epsilon);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL && beta != NULL) {
      for (int i = thrx; i < n2; i += numx) {
        U curr = static_cast<U>(lvals[i]);
        ovals[i] = gamma[i] * static_cast<V>(c_invvar * (curr - mu)) + beta[i];
      }
    } else {
      for (int i = thrx; i < n2; i += numx) {
        U curr = static_cast<U>(lvals[i]);
        ovals[i] = static_cast<V>(c_invvar * (curr - mu));
      }
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      mean[i1] = mu;
      invvar[i1] = c_invvar;
    }
  }
}

template <typename T, typename U, typename V>
__device__ void cuLoadWriteStridedInputs(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, U* warp_buf1, U* warp_buf2,
    const T* input, const V* dout, const int i1_end, const int n2,
    const U* __restrict__ mean, const U* __restrict__ invvar) {
  int i1 = i1_block + thr_load_row_off;
  if (i1 < i1_end) {
    U curr_mean = mean[i1];
    U curr_invvar = invvar[i1];
    for (int k = 0; k < blockDim.y; ++k) {
      int i2 = i2_off + k;
      int load_idx = i1 * n2 + i2;
      int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
      if (i2 < n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        warp_buf1[write_idx] = curr_dout;
        warp_buf2[write_idx] =
            curr_dout * (curr_input - curr_mean) * curr_invvar;
      } else {
        warp_buf1[write_idx] = U(0);
        warp_buf2[write_idx] = U(0);
      }
    }
  } else {
    for (int k = 0; k < blockDim.y; ++k) {
      int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
      warp_buf1[write_idx] = U(0);
      warp_buf2[write_idx] = U(0);
    }
  }
}

template <typename T, typename U, typename V>
__device__ void cuLoadAddStridedInputs(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, U* warp_buf1, U* warp_buf2,
    const T* input, const V* dout, const int i1_end, const int n2,
    const U* __restrict__ mean, const U* __restrict__ invvar) {
  int i1 = i1_block + thr_load_row_off;
  if (i1 < i1_end) {
    U curr_mean = mean[i1];
    U curr_invvar = invvar[i1];
    for (int k = 0; k < blockDim.y; ++k) {
      int i2 = i2_off + k;
      int load_idx = i1 * n2 + i2;
      int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
      if (i2 < n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        warp_buf1[write_idx] += curr_dout;
        warp_buf2[write_idx] +=
            curr_dout * (curr_input - curr_mean) * curr_invvar;
      }
    }
  }
}

template <typename T, typename U, typename V>
__global__ void cuComputePartGradGammaBeta(
    const V* __restrict__ dout, const T* __restrict__ input, const int n1,
    const int n2, const U* __restrict__ mean, const U* __restrict__ invvar,
    U epsilon, U* part_grad_gamma, U* part_grad_beta) {
  const int numsegs_n1 =
      (n1 + blockDim.y * blockDim.y - 1) / (blockDim.y * blockDim.y);
  const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
  const int i1_beg = blockIdx.y * segs_per_block * blockDim.y * blockDim.y;
  const int i1_beg_plus_one =
      (blockIdx.y + 1) * segs_per_block * blockDim.y * blockDim.y;
  const int i1_end = i1_beg_plus_one < n1 ? i1_beg_plus_one : n1;
  const int row_stride = blockDim.x + 1;
  const int thr_load_col_off = (threadIdx.x * blockDim.y) & (blockDim.x - 1);
  const int thr_load_row_off =
      (threadIdx.x * blockDim.y) / blockDim.x + threadIdx.y * blockDim.y;
  const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
  SharedMemory<U> shared;
  U* buf = shared.getPointer();  // buf has at least blockDim.x * blockDim.y *
                                 // blockDim.y + (blockDim.y -
                                 // 1)*(blockDim.x/blockDim.y) elements
  U* warp_buf1 = (U*)buf;
  U* warp_buf2 = warp_buf1 + blockDim.y * blockDim.y * row_stride;
  // compute partial sums from strided inputs
  // do this to increase number of loads in flight
  cuLoadWriteStridedInputs(i1_beg, thr_load_row_off, thr_load_col_off, i2_off,
                           row_stride, warp_buf1, warp_buf2, input, dout,
                           i1_end, n2, mean, invvar);
  for (int i1_block = i1_beg + blockDim.y * blockDim.y; i1_block < i1_end;
       i1_block += blockDim.y * blockDim.y) {
    cuLoadAddStridedInputs(i1_block, thr_load_row_off, thr_load_col_off, i2_off,
                           row_stride, warp_buf1, warp_buf2, input, dout,
                           i1_end, n2, mean, invvar);
  }
  __syncthreads();
  // inter-warp reductions
  // sum within each warp
  U acc1 = U(0);
  U acc2 = U(0);
  for (int k = 0; k < blockDim.y; ++k) {
    int row1 = threadIdx.y + k * blockDim.y;
    int idx1 = row1 * row_stride + threadIdx.x;
    acc1 += warp_buf1[idx1];
    acc2 += warp_buf2[idx1];
  }
  warp_buf1[threadIdx.y * row_stride + threadIdx.x] = acc1;
  warp_buf2[threadIdx.y * row_stride + threadIdx.x] = acc2;
  __syncthreads();
  // sum all warps
  for (int offset = blockDim.y / 2; offset > 1; offset /= 2) {
    if (threadIdx.y < offset) {
      int row1 = threadIdx.y;
      int row2 = threadIdx.y + offset;
      int idx1 = row1 * row_stride + threadIdx.x;
      int idx2 = row2 * row_stride + threadIdx.x;
      warp_buf1[idx1] += warp_buf1[idx2];
      warp_buf2[idx1] += warp_buf2[idx2];
    }
    __syncthreads();
  }
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.y == 0 && i2 < n2) {
    int row1 = threadIdx.y;
    int row2 = threadIdx.y + 1;
    int idx1 = row1 * row_stride + threadIdx.x;
    int idx2 = row2 * row_stride + threadIdx.x;
    part_grad_beta[blockIdx.y * n2 + i2] = warp_buf1[idx1] + warp_buf1[idx2];
    part_grad_gamma[blockIdx.y * n2 + i2] = warp_buf2[idx1] + warp_buf2[idx2];
  }
}

template <typename U, typename V>
__global__ void cuComputeGradGammaBeta(const U* part_grad_gamma,
                                       const U* part_grad_beta,
                                       const int part_size, const int n1,
                                       const int n2, V* grad_gamma,
                                       V* grad_beta) {
  // sum partial gradients for gamma and beta
  SharedMemory<U> shared;
  U* buf = shared.getPointer();
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (i2 < n2) {
    // each warp does sequential reductions until reduced part_size is num_warps
    int num_warp_reductions = part_size / blockDim.y;
    U sum_gamma = U(0);
    U sum_beta = U(0);
    const U* part_grad_gamma_ptr =
        part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
    const U* part_grad_beta_ptr =
        part_grad_beta + threadIdx.y * num_warp_reductions * n2 + i2;
    for (int warp_offset = 0; warp_offset < num_warp_reductions;
         ++warp_offset) {
      sum_gamma += part_grad_gamma_ptr[warp_offset * n2];
      sum_beta += part_grad_beta_ptr[warp_offset * n2];
    }
    // inter-warp reductions
    const int nbsize3 = blockDim.x * blockDim.y / 2;
    for (int offset = blockDim.y / 2; offset >= 1; offset /= 2) {
      // top half write to shared memory
      if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
        const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
        buf[write_idx] = sum_gamma;
        buf[write_idx + nbsize3] = sum_beta;
      }
      __syncthreads();
      // bottom half sums
      if (threadIdx.y < offset) {
        const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
        sum_gamma += buf[read_idx];
        sum_beta += buf[read_idx + nbsize3];
      }
      __syncthreads();
    }
    // write out fully summed gradients
    if (threadIdx.y == 0) {
      grad_gamma[i2] = sum_gamma;
      grad_beta[i2] = sum_beta;
    }
  }
}

template <typename T, typename U, typename V>
__global__ void cuComputeGradInput(const V* __restrict__ dout,
                                   const T* __restrict__ input, const int n1,
                                   const int n2, const U* __restrict__ mean,
                                   const U* __restrict__ invvar, U epsilon,
                                   const V* gamma, T* grad_input) {
  for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
    U sum_loss1 = U(0);
    U sum_loss2 = U(0);
    const U c_mean = mean[i1];
    const U c_invvar = invvar[i1];
    const T* k_input = input + i1 * n2;
    const V* k_dout = dout + i1 * n2;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL) {
      int l = 4 * thrx;
      for (; l + 3 < n2; l += 4 * numx) {
        for (int k = 0; k < 4; ++k) {
          const U c_h = static_cast<U>(k_input[l + k]);
          const U c_loss = static_cast<U>(k_dout[l + k]);
          sum_loss1 += c_loss * gamma[l + k];
          sum_loss2 += c_loss * gamma[l + k] * (c_h - c_mean) * c_invvar;
        }
      }
      for (; l < n2; ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        sum_loss1 += c_loss * gamma[l];
        sum_loss2 += c_loss * gamma[l] * (c_h - c_mean) * c_invvar;
      }
    } else {
      int l = 4 * thrx;
      for (; l + 3 < n2; l += 4 * numx) {
        for (int k = 0; k < 4; ++k) {
          const U c_h = static_cast<U>(k_input[l + k]);
          const U c_loss = static_cast<U>(k_dout[l + k]);
          sum_loss1 += c_loss;
          sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
        }
      }
      for (; l < n2; ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        sum_loss1 += c_loss;
        sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
      }
    }
    // intra-warp reductions
    for (int mask = blockDim.x / 2; mask > 0; mask /= 2) {
      sum_loss1 += WARP_SHFL_XOR(sum_loss1, mask);
      sum_loss2 += WARP_SHFL_XOR(sum_loss2, mask);
    }
    // inter-warp reductions
    if (blockDim.y > 1) {
      SharedMemory<U> shared;
      U* buf = shared.getPointer();
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
          const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[2 * wrt_i] = sum_loss1;
          buf[2 * wrt_i + 1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          sum_loss1 += buf[2 * read_i];
          sum_loss2 += buf[2 * read_i + 1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        buf[2 * threadIdx.x] = sum_loss1;
        buf[2 * threadIdx.x + 1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y != 0) {
        sum_loss1 = buf[2 * threadIdx.x];
        sum_loss2 = buf[2 * threadIdx.x + 1];
      }
    }
    // all threads now have the two sums over l
    U fH = (U)n2;
    U term1 = (U(1) / fH) * c_invvar;
    T* k_grad_input = grad_input + i1 * n2;
    if (gamma != NULL) {
      for (int l = thrx; l < n2; l += numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss * gamma[l];
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx; l < n2; l += numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss;
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
  }
}

template <typename T, typename U, typename V>
void HostApplyLayerNorm(V* output, U* mean, U* invvar, const T* input, int n1,
                        int n2, double epsilon, const V* gamma, const V* beta) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  const dim3 threads(32, 4, 1);
  const uint64_t maxGridY =
      at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
  const dim3 blocks(1, std::min((uint64_t)n1, maxGridY), 1);
  int nshared =
      threads.y > 1 ? threads.y * sizeof(U) + (threads.y / 2) * sizeof(U) : 0;
  cuApplyLayerNorm<<<blocks, threads, nshared, stream>>>(
      output, mean, invvar, input, n1, n2, U(epsilon), gamma, beta);
}

void cuda_layer_norm(at::Tensor* output, at::Tensor* mean, at::Tensor* invvar,
                     at::Tensor* input, int n1, int n2,
#ifdef VERSION_GE_1_1
                     at::IntArrayRef normalized_shape,
#else
                     at::IntList normalized_shape,
#endif
                     at::Tensor* gamma, at::Tensor* beta, double epsilon) {
  using namespace at;
  DISPATCH_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
      input->scalar_type(), output->scalar_type(), "cuda_layer_norm_kernel",
      HostApplyLayerNorm(output->DATA_PTR<scalar_t_out>(),
                         mean->DATA_PTR<float>(), invvar->DATA_PTR<float>(),
                         input->DATA_PTR<scalar_t_in>(), n1, n2, epsilon,
                         gamma != NULL ? gamma->DATA_PTR<scalar_t_out>() : NULL,
                         beta != NULL ? beta->DATA_PTR<scalar_t_out>() : NULL);)
}

template <typename T, typename U, typename V>
void HostLayerNormGradient(const V* dout, const U* mean, const U* invvar,
                           at::Tensor* input, int n1, int n2, const V* gamma,
                           const V* beta, double epsilon, T* grad_input,
                           V* grad_gamma, V* grad_beta) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();

  if (gamma != NULL && beta != NULL) {
    // compute grad_gamma(j) and grad_beta(j)
    const int part_size = 16;
    const dim3 threads2(32, 4, 1);
    const dim3 blocks2((n2 + threads2.x - 1) / threads2.x, part_size, 1);
    const int nshared2_a =
        2 * sizeof(U) * threads2.y * threads2.y * (threads2.x + 1);
    const int nshared2_b = threads2.x * threads2.y * sizeof(U);
    const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;
    at::Tensor part_grad_gamma = at::empty(
        {part_size, n2}, input->options().dtype(at::ScalarType::Float));
    at::Tensor part_grad_beta = at::empty_like(part_grad_gamma);
    cuComputePartGradGammaBeta<<<blocks2, threads2, nshared2, stream>>>(
        dout, input->DATA_PTR<T>(), n1, n2, mean, invvar, U(epsilon),
        part_grad_gamma.DATA_PTR<U>(), part_grad_beta.DATA_PTR<U>());

    const dim3 threads3(32, 8, 1);
    const dim3 blocks3((n2 + threads2.x - 1) / threads2.x, 1, 1);
    const int nshared3 = threads3.x * threads3.y * sizeof(U);
    cuComputeGradGammaBeta<<<blocks3, threads3, nshared3, stream>>>(
        part_grad_gamma.DATA_PTR<U>(), part_grad_beta.DATA_PTR<U>(), part_size,
        n1, n2, grad_gamma, grad_beta);
  }

  // compute grad_input
  const uint64_t maxGridY =
      at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
  const dim3 blocks1(1, std::min((uint64_t)n1, maxGridY), 1);
  const dim3 threads1(32, 4, 1);
  int nshared = threads1.y > 1 ? threads1.y * threads1.x * sizeof(U) : 0;
  cuComputeGradInput<<<blocks1, threads1, nshared, stream>>>(
      dout, input->DATA_PTR<T>(), n1, n2, mean, invvar, U(epsilon), gamma,
      grad_input);
}

void cuda_layer_norm_gradient(at::Tensor* dout, at::Tensor* mean,
                              at::Tensor* invvar, at::Tensor* input, int n1,
                              int n2,
#ifdef VERSION_GE_1_1
                              at::IntArrayRef normalized_shape,
#else
                              at::IntList normalized_shape,
#endif
                              at::Tensor* gamma, at::Tensor* beta,
                              double epsilon, at::Tensor* grad_input,
                              at::Tensor* grad_gamma, at::Tensor* grad_beta) {
  using namespace at;
  DISPATCH_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
      input->scalar_type(), gamma->scalar_type(),
      "cuda_layer_norm_gradient_kernel",
      HostLayerNormGradient(
          dout->DATA_PTR<scalar_t_out>(), mean->DATA_PTR<float>(),
          invvar->DATA_PTR<float>(), input, n1, n2,
          // TMJ pass NULL argument for gamma, beta, grad_gamma and grad_beta
          // if gamma Tensor is NULL on input.
          gamma != NULL ? gamma->DATA_PTR<scalar_t_out>() : NULL,
          gamma != NULL ? beta->DATA_PTR<scalar_t_out>() : NULL, epsilon,
          grad_input->DATA_PTR<scalar_t_in>(),
          gamma != NULL ? grad_gamma->DATA_PTR<scalar_t_out>() : NULL,
          gamma != NULL ? grad_beta->DATA_PTR<scalar_t_out>() : NULL);)
}

#include "hip/hip_runtime.h"
// Adapted from turboderp exllama: https://github.com/turboderp/exllama

#include "q4_matmul.cuh"
#include "column_remap.cuh"
#include "util.cuh"
#include "matrix.cuh"
#include "cu_compat.cuh"
#include "cuda_buffers.cuh"
#if defined(USE_ROCM)
#include "hip_compat.cuh"
#endif

const int THREADS_X = 32;       // Block size and thread count along columns in w and out
const int THREADS_Y = 1;        // Block size and thread count along rows in x and out

typedef void (*fp_q4_matmul_kernel)
(
    const half*,
    const uint32_t*,
    half*,
    const half*,
    const uint32_t*,
    const int,
    const int,
    const int,
    const int,
    const int,
    const uint32_t*,
    bool
);

template<bool use_half2, bool use_groupsize, bool use_x_map>
__global__ void q4_matmul_kernel
(
    const half* __restrict__ x,
    const uint32_t* __restrict__ w,
    half* __restrict__ out,
    const half* __restrict__ w_scales,
    const uint32_t* __restrict__ w_zeros,
    const int height,
    const int dim,
    const int width,
    const int groupsize,
    const int block_size_z,
    const uint32_t* __restrict__ x_map,
    bool no_zero
)
{
    // Start of block

    int x_column = block_size_z * blockIdx.z;
    int x_column_end = min(dim, block_size_z * (blockIdx.z + 1));

    int w_column = THREADS_X * blockIdx.x + threadIdx.x;
    int x_row = THREADS_Y * blockIdx.y + threadIdx.y;

    int iterations = (x_column_end - x_column) / 8;

    // Views

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_scales_(w_scales, dim / groupsize, width);
    MatrixView_q4_row w_zeros_(w_zeros, dim / groupsize, width);
    MatrixView_q4_column w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    // Zero output

    if (!no_zero && blockIdx.z == 0 && (threadIdx.x & 1) == 0)
    {
        *((uint32_t*) out_.item_ptr(x_row, w_column)) = 0;
        __syncthreads();
    }

    // Loop over part of x row (and w column)

    half2 acc = {};
    half acc_h = {};

    if constexpr (use_groupsize)
    {
        // For quant matrices where groupsize divides BLOCK_SIZE_Z we always start on a group boundary, so this
        // could be slightly faster

        for (int k = x_column, group = x_column / groupsize; k < x_column + iterations * 8; group++, k += groupsize)
        {
            if constexpr (use_half2)
            {
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map) acc = dot_product_8_x_map(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8, x_map);
                else                     acc = dot_product_8      (acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8);
            }
            else
            {
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map) acc_h = dot_product_8_x_map_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8, x_map);
                else                     acc_h = dot_product_8_h      (acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8);
            }
        }
    }
    else
    {
        // Otherwise assume groupsize is a multiple of 8, do 8 columns per iteration and trust the cache

        for (int k = x_column; k < x_column + iterations * 8; k += 8)
        {
            if constexpr (use_half2)
            {
                int group = k / groupsize;
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map) acc = dot_product_8_x_map(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1, x_map);
                else                     acc = dot_product_8      (acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1);
            }
            else
            {
                int group = k / groupsize;
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map) acc_h = dot_product_8_x_map_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1, x_map);
                else                     acc_h = dot_product_8_h      (acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1);
            }
        }
    }

    // Add to block result

    if constexpr (use_half2)
    {
        half result = __hadd(__low2half(acc), __high2half(acc));
        atomicAdd(out_.item_ptr(x_row, w_column), result);
    }
    else
    {
        atomicAdd(out_.item_ptr(x_row, w_column), acc_h);
    }
}

fp_q4_matmul_kernel q4_matmul_kernel_pick(ExLlamaTuning* tuningParams, int block_size_z, int groupsize, uint32_t* x_map)
{
    // <bool use_half2, bool use_groupsize, bool use_x_map>
    if (tuningParams->matmul_no_half2) {
        if (block_size_z % groupsize == 0) {
            if (x_map) return q4_matmul_kernel<false, true,  true >;
            else       return q4_matmul_kernel<false, true,  false>;
        } else {
            if (x_map) return q4_matmul_kernel<false, false, true >;
            else       return q4_matmul_kernel<false, false, false>;
        }
    } else {
        if (block_size_z % groupsize == 0)
        {
            if (x_map) return q4_matmul_kernel<true,  true,  true >;
            else       return q4_matmul_kernel<true,  true,  false>;
        } else {
            if (x_map) return q4_matmul_kernel<true,  false, true >;
            else       return q4_matmul_kernel<true,  false, false>;
        }
    }
};

// Compute y = x @ w

void q4_matmul_cuda
(
    ExLlamaTuning* tuningParams,
    const half* x,
    const int x_height,
    const Q4Matrix* w,
    half* out,
    bool no_zero,
    hipStream_t alt_stream
)
{
    int height = x_height;
    int dim = w->height;
    int width = w->width;

    hipSetDevice(w->device);

    uint32_t* x_map = w->cuda_x_map;
    const half* x_mapped = x;
    if (x_map && !tuningParams->matmul_fused_remap && !alt_stream)
    {
        CudaBuffers* buffers = get_buffers(w->device);
        column_remap_cuda(x, buffers->temp_state, x_height, dim, w->cuda_x_map);
        x_mapped = buffers->temp_state;
        x_map = NULL;
    }

    int block_size_z;
    if (w->width == 4096) block_size_z = 384;           // 7B
    else if (w->width == 11008) block_size_z = 256;
    else if (w->width == 5120) block_size_z = 384;      // 13B
    else if (w->width == 13824) block_size_z = 256;
    else if (w->width == 6656) block_size_z = 256;      // 33B
    else if (w->width == 17920) block_size_z = 128;
    else block_size_z = 256;

    //if (!no_zero) hipMemsetAsync(out, 0, x_height * w->width * sizeof(half));

    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (width + threads.x - 1) / threads.x,
        (height + threads.y - 1) / threads.y,
        (dim + block_size_z - 1) / block_size_z
    );

    fp_q4_matmul_kernel kernel = q4_matmul_kernel_pick(tuningParams, block_size_z, w->groupsize, x_map);

    kernel<<<blocks, threads, 0, alt_stream>>> (x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, x_map, no_zero);
}

void q4_matmul_recons_cuda
(
    ExLlamaTuning* tuningParams,
    const half* x,
    const int x_height,
    Q4Matrix* w,
    half* out,
    const hipblasHandle_t handle,
    bool no_zero
)
{
    int height = x_height;
    int dim = w->height;
    int width = w->width;

    hipSetDevice(w->device);
    CudaBuffers* buffers = get_buffers(w->device);

    const half* x_mapped = x;
    if (w->cuda_x_map)
    {
        TORCH_CHECK(buffers->temp_state_size >= x_height * dim, "temp_state buffer is too small");
        column_remap_cuda(x, buffers->temp_state, x_height, dim, w->cuda_x_map);
        x_mapped = buffers->temp_state;
    }

    w->reconstruct(buffers->temp_dq);

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700
    const float alpha = 1.0f;
    const float beta = no_zero ? 1.0f : 0.0f;
    cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, buffers->temp_dq, HIP_R_16F, width,
                  x_mapped, HIP_R_16F, dim, &beta, out, HIP_R_16F, width);
#else
    const half alpha = __float2half(1.0f);
    const half beta = no_zero ? __float2half(1.0f) : __float2half(0.0f);
    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, buffers->temp_dq, width, x_mapped, dim, &beta, out, width);
#endif
}

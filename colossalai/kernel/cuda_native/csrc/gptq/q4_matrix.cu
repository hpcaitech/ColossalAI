#include "hip/hip_runtime.h"
// Adapted from turboderp exllama: https://github.com/turboderp/exllama

#include "q4_matrix.cuh"
#include <vector>
#include "util.cuh"
#include "matrix.cuh"

using namespace std;

const int UNSHUF_BLOCKSIZE_X = 64;

const int RECONS_THREADS_X = 64;      // Block size and thread count along columns in out, each thread converts 1 column
const int RECONS_THREADS_Y = 1;       // Block size and thread count along rows in x and out, each thread converts 8 rows

vector<Q4Matrix*> g_q4_matrices;

void g_q4_keep_matrix(Q4Matrix* m)
{
    g_q4_matrices.push_back(m);
}

void g_q4_free_matrices()
{
    for (const auto& m : g_q4_matrices) delete m;
    g_q4_matrices.clear();
}

Q4Matrix::Q4Matrix
(
    const int _height,
    const int _width,
    const int _groups,

    uint32_t* _qweight,
    uint32_t* _qzeros,
    half* _scales,
    uint32_t* _g_idx,

    const int _device
) :
    height(_height),
    width(_width),
    groups(_groups),
    device(_device)
{
    hipSetDevice(device);

    cuda_qweight = _qweight;
    cuda_qzeros = _qzeros;
    cuda_scales = _scales;

    groupsize = height / groups;

    if (_g_idx) make_sequential(_g_idx);
}

Q4Matrix::~Q4Matrix()
{
}

// Make sequential

__global__ void make_sequential_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const uint32_t* __restrict__ x_map,
    const int w_height,
    const int w_width
)
{
    const uint64_t* w2 = (uint64_t*) w;
    uint64_t* w_new2 = (uint64_t*) w_new;
    int w2_stride = w_width >> 1;

    int w2_column = UNSHUF_BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    if (w2_column >= w2_stride) return;

    int w_new2_row = blockIdx.y;

    int x_map_idx = w_new2_row << 3;

    uint64_t dst = 0;

    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        int source_row = x_map[x_map_idx++];

        int w2_row = source_row >> 3;
        int w2_subrow = source_row & 0x07;
        int w2_row_shift = w2_subrow << 2;
        int wnew2_row_shift = i << 2;

        uint64_t src = w2[w2_row * w2_stride + w2_column];
        src >>= w2_row_shift;
        src &= 0x0000000f0000000f;
        src <<= wnew2_row_shift;
        dst |= src;
    }

    w_new2[w_new2_row * w2_stride + w2_column] = dst;
}

void Q4Matrix::make_sequential(const uint32_t* cpu_g_idx)
{
    uint32_t* cuda_new_qweight = NULL;
    hipMalloc(&cuda_new_qweight, height / 8 * width * sizeof(uint32_t));
    hipMalloc(&cuda_x_map, height * sizeof(uint32_t));  // TODO: Should probably be allocated in PyTorch

    uint32_t* cpu_g_idx_map = (uint32_t*) calloc(groups, sizeof(uint32_t));
    uint32_t* cpu_x_map = (uint32_t*) malloc(height * sizeof(uint32_t));
    uint32_t* cpu_x_map_inv = (uint32_t*) malloc(height * sizeof(uint32_t));

    // Group histogram

    for (int i = 0; i < height; i++) cpu_g_idx_map[cpu_g_idx[i]]++;

    // Group map

    for (int i = 0, acc = 0; i < groups; i++)
    {
        short tmp = cpu_g_idx_map[i];
        cpu_g_idx_map[i] = acc;
        acc += tmp;
    }

    // X map (inverse)

    for (int row = 0; row < height; row++)
    {
        uint32_t target_group = cpu_g_idx[row];
        uint32_t target_row = cpu_g_idx_map[target_group];
        cpu_g_idx_map[target_group]++;
        cpu_x_map_inv[row] = target_row;
    }

    // X map

    for (int row = 0; row < height; row++) cpu_x_map[cpu_x_map_inv[row]] = row;

    // Move to CUDA

    hipMemcpyAsync(cuda_x_map, cpu_x_map, height * sizeof(uint32_t), hipMemcpyHostToDevice);

    // Rearrange rows in w

    dim3 threads(UNSHUF_BLOCKSIZE_X, 1, 1);
    dim3 blocks
    (
        (width + UNSHUF_BLOCKSIZE_X * 2 - 1) / (UNSHUF_BLOCKSIZE_X * 2),
        height / 8,
        1
    );

    make_sequential_kernel<<<blocks, threads>>>(cuda_qweight, cuda_new_qweight, cuda_x_map, height / 8, width);

    // Replace qweights

    hipMemcpyAsync(cuda_qweight, cuda_new_qweight, height / 8 * width * sizeof(uint32_t), hipMemcpyDeviceToDevice);

    // Cleanup

    hipDeviceSynchronize();
    hipFree(cuda_new_qweight);
    free(cpu_g_idx_map);
    free(cpu_x_map);
    free(cpu_x_map_inv);
}

__global__ void reconstruct_kernel
(
    const uint32_t* __restrict__ w,
    half* __restrict__ out,  // (y)
    const half* __restrict__ w_scales,
    const uint32_t* __restrict__ w_zeros,
    const int height,
    const int width,
    const int groupsize
)
{
    // Start of block

    int column = RECONS_THREADS_X * blockIdx.x + threadIdx.x;
    int row = (RECONS_THREADS_Y * blockIdx.y + threadIdx.y) * 8;
    if (column >= width) return;
    
    // Views

    MatrixView_q4_column w_(w, height, width);
    MatrixView_half_rw out_(out, height, width);
    MatrixView_half w_scales_(w_scales, height / groupsize, width);
    MatrixView_q4_row w_zeros_(w_zeros, height / groupsize, width);

    // Groupsize version

    int group = row / groupsize;

    half w_scale = w_scales_.item(group, column);
    uint32_t w_zero = w_zeros_.item(group, column) + 1;

    uint32_t w_read = w_.item_uint32_t(row, column);
    half* out_ptr = out_.item_ptr(row, column);

    #pragma unroll
    for (int s = 0; s < 32; s += 4)
    {
        half w_item = __hmul(__int2half_rn((int)((w_read >> s) & 0x0f) - w_zero), w_scale);
        *out_ptr = w_item; out_ptr += out_.width;
    }
}

void Q4Matrix::reconstruct(half* out)
{
    dim3 threads(RECONS_THREADS_X, RECONS_THREADS_Y, 1);

    dim3 blocks
    (
        (width + threads.x - 1) / threads.x,
        (height / 8 + threads.y - 1) / threads.y,
        1
    );

    reconstruct_kernel<<<blocks, threads>>>(cuda_qweight, out, cuda_scales, cuda_qzeros, height / 8, width, groupsize);
}

#include "hip/hip_runtime.h"
// Adapted from turboderp exllama: https://github.com/turboderp/exllama

#include "column_remap.cuh"
#include "util.cuh"

const int SHUF_BLOCKSIZE_X = 256;
const int SHUF_BLOCKSIZE_Y = 16;

__global__ void column_remap_kernel
(
    const half* __restrict__ x,
    half* __restrict__ x_new,
    const int x_width,
    const int x_height,
    const uint32_t* x_map
)
{
    int x_column = SHUF_BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int x_row = SHUF_BLOCKSIZE_Y * blockIdx.y;
    if (x_column >= x_width) return;
    //if (x_row >= x_height) return;

    int x_stride = x_width;
    int x_idx = x_row * x_stride + x_column;

    int x_row_end = min(x_row + SHUF_BLOCKSIZE_Y, x_height);
    int x_idx_end = x_row_end * x_stride + x_column;

    int s_column = x_map[x_column];
    int s_idx = x_row * x_stride + s_column;

    while (x_idx < x_idx_end)
    {
        x_new[x_idx] = x[s_idx];
        x_idx += x_stride;
        s_idx += x_stride;
    }
}

// Remap columns in x to correspond to sequential group index before matmul
//
// perform x -> seq_x such that seq_x @ seq_w == x @ w

void column_remap_cuda
(
    const half* x,
    half* x_new,
    const int x_height,
    const int x_width,
    const uint32_t* x_map
)
{
    dim3 threads(SHUF_BLOCKSIZE_X, 1, 1);

    dim3 blocks
    (
        (x_width + SHUF_BLOCKSIZE_X - 1) / SHUF_BLOCKSIZE_X,
        (x_height + SHUF_BLOCKSIZE_Y - 1) / SHUF_BLOCKSIZE_Y,
        1
    );

    column_remap_kernel<<<blocks, threads>>>(x, x_new, x_width, x_height, x_map);
}

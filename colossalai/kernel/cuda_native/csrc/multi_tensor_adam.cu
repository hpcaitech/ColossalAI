#include "hip/hip_runtime.h"
// modified from
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_adam.cu
/* Copyright 2020 The Microsoft DeepSpeed Team
   Copyright NVIDIA/apex
   This file is adapted from fused adam in NVIDIA/apex, commit a109f85
   Licensed under the MIT License.
*/
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "multi_tensor_apply.cuh"
#include "type_shim.h"

#define BLOCK_SIZE 512
#define ILP 4

typedef enum {
  ADAM_MODE_0 = 0,  // L2 regularization mode
  ADAM_MODE_1 = 1   // Decoupled weight decay mode(AdamW)
} adamMode_t;

using MATH_T = float;

template <typename T_g, typename T_p>
struct AdamFunctor {
  __device__ __forceinline__ void operator()(
      int chunk_size, volatile int *noop_gmem, TensorListMetadata<4> &tl,
      const float beta1, const float beta2, const float beta1_correction,
      const float beta2_correction, const float epsilon, const float lr,
      adamMode_t mode, const float decay, const float div_scale) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T_g *g = (T_g *)tl.addresses[0][tensor_loc];
    g += chunk_idx * chunk_size;

    T_p *p = (T_p *)tl.addresses[1][tensor_loc];
    p += chunk_idx * chunk_size;

    T_p *m = (T_p *)tl.addresses[2][tensor_loc];
    m += chunk_idx * chunk_size;

    T_p *v = (T_p *)tl.addresses[3][tensor_loc];
    v += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (int i_start = 0; i_start < n && i_start < chunk_size;
         i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
          r_v[ii] = v[i];
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (div_scale > 0) r_g[ii] /= div_scale;

        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = r_p[ii];
          m[i] = r_m[ii];
          v[i] = r_v[ii];
        }
      }
    }
  }
};

void multi_tensor_adam_cuda(int chunk_size, at::Tensor noop_flag,
                            std::vector<std::vector<at::Tensor>> tensor_lists,
                            const float lr, const float beta1,
                            const float beta2, const float epsilon,
                            const int step, const int mode,
                            const int bias_correction, const float weight_decay,
                            const float div_scale) {
  using namespace at;

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  DISPATCH_FLOAT_AND_HALF_FOR_G_P(
      tensor_lists[0][0].scalar_type(), tensor_lists[1][0].scalar_type(), 0,
      "adam",
      multi_tensor_apply<4>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            AdamFunctor<g_scalar_t_0, p_scalar_t_0>(), beta1,
                            beta2, bias_correction1, bias_correction2, epsilon,
                            lr, (adamMode_t)mode, weight_decay, div_scale);)

  AT_CUDA_CHECK(hipGetLastError());
}

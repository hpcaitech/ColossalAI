#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "block_reduce.h"
#include "kernels.h"

namespace cg = cooperative_groups;
const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32

template <typename T>
__forceinline__ __device__ T add_eps(T x) {
  return fabsf(x) > LN_EPSILON ? x : (x < 0 ? -LN_EPSILON : LN_EPSILON);
}

/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size* seq_len, hidden_size], ln result.
vars: [batch_size* seq_len], variance per token
means: [batch_size* seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_square_sum +=
        val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 4.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  float4 *output_f4 = (float4 *)ln_res + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 vscale = __ldg((const float4 *)scale + idx);
    float4 vbias = __ldg((const float4 *)bias + idx);
    float4 val = inp_f4[idx];
    val.x = (val.x - s_mean) * s_var * vscale.x + vbias.x;
    val.y = (val.y - s_mean) * s_var * vscale.y + vbias.y;
    val.z = (val.z - s_mean) * s_var * vscale.z + vbias.z;
    val.w = (val.w - s_mean) * s_var * vscale.w + vbias.w;
    output_f4[idx] = val;
  }
}

template <>
__global__ void ker_layer_norm<__half>(__half *ln_res, __half *vars,
                                       __half *means, const __half *inp,
                                       const __half *scale, const __half *bias,
                                       int hidden_size) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val_f4 = inp_f4[idx];
    __half2 *val_h2 = (__half2 *)(&val_f4);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 val_f2 = __half22float2(val_h2[i]);
      l_sum += val_f2.x + val_f2.y;
      l_square_sum += val_f2.x * val_f2.x + val_f2.y * val_f2.y;
    }
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 8.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  float4 *output_f4 = (float4 *)ln_res + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    // load scale, bias, input
    float4 scale_f4 = __ldg((const float4 *)scale + idx);
    __half2 *scale_h2 = (__half2 *)(&scale_f4);
    float4 bias_f4 = __ldg((const float4 *)bias + idx);
    __half2 *bias_h2 = (__half2 *)(&bias_f4);
    float4 val_f4 = inp_f4[idx];
    __half2 *val_h2 = (__half2 *)(&val_f4);

#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 scale_f2 = __half22float2(scale_h2[i]);
      float2 bias_f2 = __half22float2(bias_h2[i]);
      float2 val_f2 = __half22float2(val_h2[i]);
      val_f2.x = (val_f2.x - s_mean) * s_var * scale_f2.x + bias_f2.x;
      val_f2.y = (val_f2.y - s_mean) * s_var * scale_f2.y + bias_f2.y;
      val_h2[i] = __float22half2_rn(val_f2);
    }
    output_f4[idx] = val_f4;
  }
}

// __global__ void ker_layer_norm_x2(__half *ln_res, __half *vars,
//                                        __half *means, const __half *inp,
//                                        const __half *scale, const __half
//                                        *bias, int hidden_size) {
//   // step 0. compute local sum
//   float l_sum = 0;
//   float l_square_sum = 0;
//   const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * 2 * hidden_size;
//   for (uint idx = 2 * threadIdx.x; idx < hidden_size * 2; idx += blockDim.x *
//   2) {
//     float4 val_f4 = inp_f4[idx];
//     float4 val_f4_1 = inp_f4[idx+1];
//     __half2 *val_h2 = (__half2 *)(&val_f4);
//     __half2 *val_h2_1 = (__half2 *)(&val_f4_1);
// #pragma unroll
//     for (int i = 0; i < 4; i++) {
//       float2 val_f2 = __half22float2(val_h2[i]);
//       float2 val_f2_1 = __half22float2(val_h2_1[i]);
//       l_sum += val_f2.x + val_f2.y + val_f2_1.x + val_f2_1.y;
//       l_square_sum += val_f2.x * val_f2.x + val_f2.y * val_f2.y + val_f2_1.x
//       * val_f2_1.x + val_f2_1.y * val_f2_1.y;
//     }
//   }

//   // step 1. compute reduce sum
//   float mean_dim = float(hidden_size) * 8.f * 2;
//   float reduce_val[2] = {l_sum, l_square_sum};
//   blockReduce<ReduceType::kSum, 2>(reduce_val);
//   __shared__ float s_mean, s_var;
//   if (threadIdx.x == 0) {
//     s_mean = reduce_val[0] / mean_dim;
//     if (means != nullptr) {
//       means[blockIdx.x] = s_mean;
//     }
//     s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
//     vars[blockIdx.x] = s_var;
//     s_var = rsqrtf(s_var);
//   }
//   __syncthreads();

//   // step 2. layer norm result
//   float4 *output_f4 = (float4 *)ln_res + blockIdx.x * hidden_size * 2;
//   for (uint idx = 2 * threadIdx.x; idx < hidden_size * 2; idx += blockDim.x *
//   2) {
//     // load scale, bias, input
//     float4 scale_f4 = __ldg((const float4 *)scale + idx);
//     __half2 *scale_h2 = (__half2 *)(&scale_f4);
//     float4 scale_f4_1 = __ldg((const float4 *)scale + idx + 1);
//     __half2 *scale_h2_1 = (__half2 *)(&scale_f4_1);
//     float4 bias_f4 = __ldg((const float4 *)bias + idx);
//     __half2 *bias_h2 = (__half2 *)(&bias_f4);
//     float4 bias_f4_1 = __ldg((const float4 *)bias + idx + 1);
//     __half2 *bias_h2_1 = (__half2 *)(&bias_f4_1);
//     float4 val_f4 = inp_f4[idx];
//     __half2 *val_h2 = (__half2 *)(&val_f4);
//     float4 val_f4_1 = inp_f4[idx+1];
//     __half2 *val_h2_1 = (__half2 *)(&val_f4_1);

// #pragma unroll
//     for (int i = 0; i < 4; i++) {
//       float2 scale_f2 = __half22float2(scale_h2[i]);
//       float2 scale_f2_1 = __half22float2(scale_h2_1[i]);
//       float2 bias_f2 = __half22float2(bias_h2[i]);
//       float2 bias_f2_1 = __half22float2(bias_h2_1[i]);
//       float2 val_f2 = __half22float2(val_h2[i]);
//       float2 val_f2_1 = __half22float2(val_h2_1[i]);
//       val_f2.x = (val_f2.x - s_mean) * s_var * scale_f2.x + bias_f2.x;
//       val_f2.y = (val_f2.y - s_mean) * s_var * scale_f2.y + bias_f2.y;
//       val_h2[i] = __float22half2_rn(val_f2);
//       val_f2_1.x = (val_f2_1.x - s_mean) * s_var * scale_f2_1.x +
//       bias_f2_1.x; val_f2_1.y = (val_f2_1.y - s_mean) * s_var * scale_f2_1.y
//       + bias_f2_1.y; val_h2_1[i] = __float22half2_rn(val_f2_1);
//     }
//     output_f4[idx] = val_f4;
//     output_f4[idx+1] = val_f4_1;
//   }
// }

// __global__ void ker_layer_norm_x4(__half *ln_res, __half *vars,
//                                        __half *means, const __half *inp,
//                                        const __half *scale, const __half
//                                        *bias, int hidden_size) {
//   // step 0. compute local sum
//   float l_sum = 0;
//   float l_square_sum = 0;
//   const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size * 4;
//   for (uint idx = 4 * threadIdx.x; idx < hidden_size * 4; idx += blockDim.x *
//   4) {
//     float4 val_f4 = inp_f4[idx];
//     float4 val_f4_1 = inp_f4[idx+1];
//     float4 val_f4_2 = inp_f4[idx+2];
//     float4 val_f4_3 = inp_f4[idx+3];
//     __half2 *val_h2 = (__half2 *)(&val_f4);
//     __half2 *val_h2_1 = (__half2 *)(&val_f4_1);
//     __half2 *val_h2_2 = (__half2 *)(&val_f4_2);
//     __half2 *val_h2_3 = (__half2 *)(&val_f4_3);
// #pragma unroll
//     for (int i = 0; i < 4; i++) {
//       float2 val_f2 = __half22float2(val_h2[i]);
//       float2 val_f2_1 = __half22float2(val_h2_1[i]);
//       float2 val_f2_2 = __half22float2(val_h2_2[i]);
//       float2 val_f2_3 = __half22float2(val_h2_3[i]);
//       l_sum += val_f2.x + val_f2.y + val_f2_1.x + val_f2_1.y + val_f2_2.x +
//       val_f2_2.y + val_f2_3.x + val_f2_3.y; l_square_sum += val_f2.x *
//       val_f2.x + val_f2.y * val_f2.y; l_square_sum += val_f2_1.x * val_f2_1.x
//       + val_f2_1.y * val_f2_1.y; l_square_sum += val_f2_2.x * val_f2_2.x +
//       val_f2_2.y * val_f2_2.y; l_square_sum += val_f2_3.x * val_f2_3.x +
//       val_f2_3.y * val_f2_3.y;
//     }
//   }

//   // step 1. compute reduce sum
//   float mean_dim = float(hidden_size) * 8.f * 4;
//   float reduce_val[2] = {l_sum, l_square_sum};
//   blockReduce<ReduceType::kSum, 2>(reduce_val);
//   __shared__ float s_mean, s_var;
//   if (threadIdx.x == 0) {
//     s_mean = reduce_val[0] / mean_dim;
//     if (means != nullptr) {
//       means[blockIdx.x] = s_mean;
//     }
//     s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
//     vars[blockIdx.x] = s_var;
//     s_var = rsqrtf(s_var);
//   }
//   __syncthreads();

//   // step 2. layer norm result
//   float4 *output_f4 = (float4 *)ln_res + blockIdx.x * hidden_size * 4;
//   for (uint idx = 4 * threadIdx.x; idx < hidden_size * 4; idx += blockDim.x *
//   4) {
//     // load scale, bias, input
//     float4 scale_f4 = __ldg((const float4 *)scale + idx);
//     __half2 *scale_h2 = (__half2 *)(&scale_f4);
//     float4 scale_f4_1 = __ldg((const float4 *)scale + idx + 1);
//     __half2 *scale_h2_1 = (__half2 *)(&scale_f4_1);
//     float4 scale_f4_2 = __ldg((const float4 *)scale + idx + 2);
//     __half2 *scale_h2_2 = (__half2 *)(&scale_f4_2);
//     float4 scale_f4_3 = __ldg((const float4 *)scale + idx + 3);
//     __half2 *scale_h2_3 = (__half2 *)(&scale_f4_3);
//     float4 bias_f4 = __ldg((const float4 *)bias + idx);
//     __half2 *bias_h2 = (__half2 *)(&bias_f4);
//     float4 bias_f4_1 = __ldg((const float4 *)bias + idx + 1);
//     __half2 *bias_h2_1 = (__half2 *)(&bias_f4_1);
//     float4 bias_f4_2 = __ldg((const float4 *)bias + idx + 2);
//     __half2 *bias_h2_2 = (__half2 *)(&bias_f4_2);
//     float4 bias_f4_3 = __ldg((const float4 *)bias + idx + 3);
//     __half2 *bias_h2_3 = (__half2 *)(&bias_f4_3);
//     float4 val_f4 = inp_f4[idx];
//     __half2 *val_h2 = (__half2 *)(&val_f4);
//     float4 val_f4_1 = inp_f4[idx+1];
//     __half2 *val_h2_1 = (__half2 *)(&val_f4_1);
//     float4 val_f4_2 = inp_f4[idx+2];
//     __half2 *val_h2_2 = (__half2 *)(&val_f4_2);
//     float4 val_f4_3 = inp_f4[idx+3];
//     __half2 *val_h2_3 = (__half2 *)(&val_f4_3);

// #pragma unroll
//     for (int i = 0; i < 4; i++) {
//       float2 scale_f2 = __half22float2(scale_h2[i]);
//       float2 scale_f2_1 = __half22float2(scale_h2_1[i]);
//       float2 scale_f2_2 = __half22float2(scale_h2_2[i]);
//       float2 scale_f2_3 = __half22float2(scale_h2_3[i]);
//       float2 bias_f2 = __half22float2(bias_h2[i]);
//       float2 bias_f2_1 = __half22float2(bias_h2_1[i]);
//       float2 bias_f2_2 = __half22float2(bias_h2_2[i]);
//       float2 bias_f2_3 = __half22float2(bias_h2_3[i]);
//       float2 val_f2 = __half22float2(val_h2[i]);
//       float2 val_f2_1 = __half22float2(val_h2_1[i]);
//       float2 val_f2_2 = __half22float2(val_h2_2[i]);
//       float2 val_f2_3 = __half22float2(val_h2_3[i]);
//       val_f2.x = (val_f2.x - s_mean) * s_var * scale_f2.x + bias_f2.x;
//       val_f2.y = (val_f2.y - s_mean) * s_var * scale_f2.y + bias_f2.y;
//       val_f2_1.x = (val_f2_1.x - s_mean) * s_var * scale_f2_1.x +
//       bias_f2_1.x; val_f2_1.y = (val_f2_1.y - s_mean) * s_var * scale_f2_1.y
//       + bias_f2_1.y; val_f2_2.x = (val_f2_2.x - s_mean) * s_var *
//       scale_f2_2.x + bias_f2_2.x; val_f2_2.y = (val_f2_2.y - s_mean) * s_var
//       * scale_f2_2.y + bias_f2_2.y; val_f2_3.x = (val_f2_3.x - s_mean) *
//       s_var * scale_f2_3.x + bias_f2_3.x; val_f2_3.y = (val_f2_3.y - s_mean)
//       * s_var * scale_f2_3.y + bias_f2_3.y; val_h2[i] =
//       __float22half2_rn(val_f2); val_h2_1[i] = __float22half2_rn(val_f2_1);
//       val_h2_2[i] = __float22half2_rn(val_f2_2);
//       val_h2_3[i] = __float22half2_rn(val_f2_3);
//     }
//     output_f4[idx] = val_f4;
//     output_f4[idx+1] = val_f4_1;
//     output_f4[idx+2] = val_f4_2;
//     output_f4[idx+3] = val_f4_3;
//   }
// }

template <>
void launch_layer_norm<float>(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim);
}

template <>
void launch_layer_norm<__half>(__half *ln_res, __half *vars, __half *means,
                               const __half *inp, const __half *scale,
                               const __half *bias, int batch_size,
                               int hidden_dim, hipStream_t stream) {
  if (hidden_dim % 8 != 0) {
    throw std::runtime_error("violate hidden_dim % 8 = 0");
  }
  hidden_dim >>= 3;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<__half><<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim);
  // if (hidden_dim % 8 != 0) {
  //   throw std::runtime_error("violate hidden_dim % 8 = 0");
  // }
  // hidden_dim >>= 3;

  // if (hidden_dim * 8 < 8192) {
  //   int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  //   dim3 grid_dim(batch_size);
  //   dim3 block_dim(nthread);
  //   ker_layer_norm<__half><<<grid_dim, block_dim, 0, stream>>>(
  //       ln_res, vars, means, inp, scale, bias, hidden_dim);
  // } else if (hidden_dim * 8 >= 8192 && hidden_dim * 8 <= 8192 * 2) {
  //   hidden_dim >>= 1;
  //   int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  //   dim3 grid_dim(batch_size);
  //   dim3 block_dim(nthread);
  //   ker_layer_norm_x2<<<grid_dim, block_dim, 0, stream>>>(
  //       ln_res, vars, means, inp, scale, bias, hidden_dim);
  // } else if (hidden_dim * 8 > 8192 * 2 && hidden_dim * 8 <= 8192 * 4) {
  //   hidden_dim >>= 2;
  //   int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  //   dim3 grid_dim(batch_size);
  //   dim3 block_dim(nthread);
  //   ker_layer_norm_x4<<<grid_dim, block_dim, 0, stream>>>(
  //       ln_res, vars, means, inp, scale, bias, hidden_dim);
  // } else {
  //   throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 32768");
  // }
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma


@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad, const T *inp_or_out,
                                        const T *gamma, const T *betta,
                                        const T *vars, const T *means, int rows,
                                        int width) {
  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = threadIdx.y * width + idx;
  int y_stride = width * TILE_DIM;

  // Loop across inp height
  float dbetta = 0;
  float dgamma = 0;
  float dout, val;
  if (idx < width) {
    if (means == nullptr) {
      float vbetta = (float)betta[idx];
      float vgamma = (float)gamma[idx];
      for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        dout = (float)out_grad[offset];
        // inp_or_out is output
        val = (float)inp_or_out[offset];
        dbetta += dout;
        dgamma += ((val - vbetta) / add_eps(vgamma) * dout);
        offset += y_stride;
      }
    } else {
      for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        dout = (float)out_grad[offset];
        // inp_or_out is input
        val = (float)inp_or_out[offset];
        dbetta += dout;
        dgamma += ((val - (float)means[r]) *
                   rsqrtf((float)vars[r] + LN_EPSILON) * dout);
        offset += y_stride;
      }
    }
  }

  // Sum the shared buffer.
  betta_buffer[threadIdx.x][threadIdx.y] = dbetta;
  gamma_buffer[threadIdx.x][threadIdx.y] = dgamma;
  __syncthreads();
  float s1 = betta_buffer[threadIdx.y][threadIdx.x];
  float s2 = gamma_buffer[threadIdx.y][threadIdx.x];
  __syncthreads();

  for (int i = 1; i < TILE_DIM; i <<= 1) {
    s1 += g.shfl_down(s1, i);
    s2 += g.shfl_down(s2, i);
  }

  int pos = blockIdx.x * TILE_DIM + threadIdx.y;
  if (threadIdx.x == 0 && idx < width) {
    betta_grad[pos] = s1;
    gamma_grad[pos] = s2;
  }
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad,
                               const T *residual_grad, const T *inp_or_out,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, int hidden_dim) {
  int offset = blockIdx.x * hidden_dim + threadIdx.x;
  float4 dxhat, xhat;
  float var_rsqrt;

  if (threadIdx.x < hidden_dim) {
    // step 0. dxhat = dout * gamma
    dxhat = ((const float4 *)out_grad)[offset];
    float4 vgamma = ((const float4 *)gamma)[threadIdx.x];
    dxhat.x *= vgamma.x;
    dxhat.y *= vgamma.y;
    dxhat.z *= vgamma.z;
    dxhat.w *= vgamma.w;

    /*
    step 1. xhat = (output - betta) / gamma or
    (input - mean) * rsqrtf(var)
    */
    xhat = ((const float4 *)inp_or_out)[offset];
    var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);
    if (means == nullptr) {
      // inp_or_out is output, xhat = (output - betta) / gamma
      float4 vbetta = ((const float4 *)betta)[threadIdx.x];
      xhat.x = (xhat.x - vbetta.x) / add_eps(vgamma.x);
      xhat.y = (xhat.y - vbetta.y) / add_eps(vgamma.y);
      xhat.z = (xhat.z - vbetta.z) / add_eps(vgamma.z);
      xhat.w = (xhat.w - vbetta.w) / add_eps(vgamma.w);
    } else {
      // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
      float fmean = (float)means[blockIdx.x];
      xhat.x = (xhat.x - fmean) * var_rsqrt;
      xhat.y = (xhat.y - fmean) * var_rsqrt;
      xhat.z = (xhat.z - fmean) * var_rsqrt;
      xhat.w = (xhat.w - fmean) * var_rsqrt;
    }
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  float reduce_val[2] = {0.f, 0.f};
  if (threadIdx.x < hidden_dim) {
    reduce_val[0] = dxhat.x + dxhat.y + dxhat.z + dxhat.w;
    reduce_val[1] = dxhat.x * xhat.x + dxhat.y * xhat.y + dxhat.z * xhat.z +
                    dxhat.w * xhat.w;
  }
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float mean_dim = hidden_dim * 4;
    s_sum_dxhat = reduce_val[0] / mean_dim;
    s_sum_dxhat_xhat = reduce_val[1] / mean_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / mean_dim) * rsqrt(var)
  */
  if (threadIdx.x >= hidden_dim) {
    return;
  }
  dxhat.x = (dxhat.x - s_sum_dxhat - xhat.x * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.y = (dxhat.y - s_sum_dxhat - xhat.y * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.z = (dxhat.z - s_sum_dxhat - xhat.z * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.w = (dxhat.w - s_sum_dxhat - xhat.w * s_sum_dxhat_xhat) * var_rsqrt;
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    dxhat.x += dresidual.x;
    dxhat.y += dresidual.y;
    dxhat.z += dresidual.z;
    dxhat.w += dresidual.w;
  }
  ((float4 *)inp_grad)[offset] = dxhat;
}

template <>
__global__ void ker_ln_bw_dinp<__half>(__half *inp_grad, const __half *out_grad,
                                       const __half *residual_grad,
                                       const __half *inp_or_out,
                                       const __half *gamma, const __half *betta,
                                       const __half *vars, const __half *means,
                                       int hidden_dim) {
  int offset = blockIdx.x * hidden_dim + threadIdx.x;

  float2 dxhat[4], xhat[4];
  float var_rsqrt;
  float4 vtmp;
  __half2 *tmp_h2;
  float reduce_val[2] = {0.f, 0.f};

  if (threadIdx.x < hidden_dim) {
    // step 0. dxhat = dout * gamma
    vtmp = ((const float4 *)out_grad)[offset];
    tmp_h2 = reinterpret_cast<__half2 *>(&vtmp);
    float4 gamma_f4 = ((const float4 *)gamma)[threadIdx.x];
    __half2 *gamma_h2 = reinterpret_cast<__half2 *>(&gamma_f4);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 vdout = __half22float2(tmp_h2[i]);
      float2 vgamma = __half22float2(gamma_h2[i]);
      dxhat[i].x = vdout.x * vgamma.x;
      dxhat[i].y = vdout.y * vgamma.y;
      reduce_val[0] += dxhat[i].x + dxhat[i].y;
    }

    /*
    step 1. xhat = (output - betta) / gamma or
    (input - mean) * rsqrtf(var)
    */
    vtmp = ((const float4 *)inp_or_out)[offset];
    var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);
    if (means == nullptr) {
      // inp_or_out is output, xhat = (output - betta) / gamma
      float4 vbetta = ((const float4 *)betta)[threadIdx.x];
      __half2 *betta_h2 = reinterpret_cast<__half2 *>(&vbetta);
#pragma unroll
      for (int i = 0; i < 4; i++) {
        float2 vout = __half22float2(tmp_h2[i]);
        float2 vgamma = __half22float2(gamma_h2[i]);
        float2 vbetta = __half22float2(betta_h2[i]);
        xhat[i].x = (vout.x - vbetta.x) / add_eps(vgamma.x);
        xhat[i].y = (vout.y - vbetta.y) / add_eps(vgamma.y);
        reduce_val[1] += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
      }
    } else {
      // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
      float fmean = (float)means[blockIdx.x];
#pragma unroll
      for (int i = 0; i < 4; i++) {
        float2 vinp = __half22float2(tmp_h2[i]);
        xhat[i].x = (vinp.x - fmean) * var_rsqrt;
        xhat[i].y = (vinp.y - fmean) * var_rsqrt;
        reduce_val[1] += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
      }
    }
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float mean_dim = hidden_dim * 8;
    s_sum_dxhat = reduce_val[0] / mean_dim;
    s_sum_dxhat_xhat = reduce_val[1] / mean_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / mean_dim) * rsqrt(var)
  */
  if (threadIdx.x >= hidden_dim) {
    return;
  }
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    __half *hdres = reinterpret_cast<__half *>(&dresidual);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i]));
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i + 1]));
    }
  } else {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
    }
  }
  ((float4 *)inp_grad)[offset] = vtmp;
}

__global__ void ker_ln_bw_dinp_x2(__half *inp_grad, const __half *out_grad,
                                  const __half *residual_grad,
                                  const __half *inp_or_out, const __half *gamma,
                                  const __half *betta, const __half *vars,
                                  const __half *means, int hidden_dim) {
  int offset = blockIdx.x * hidden_dim * 2 + threadIdx.x * 2;

  float2 dxhat[4], xhat[4];
  float2 dxhat_1[4], xhat_1[4];
  float var_rsqrt;
  float4 vtmp, vtmp_1;
  __half2 *tmp_h2;
  __half2 *tmp_h2_1;
  float reduce_val[2] = {0.f, 0.f};

  if (threadIdx.x < hidden_dim) {
    // step 0. dxhat = dout * gamma
    vtmp = ((const float4 *)out_grad)[offset];
    vtmp_1 = ((const float4 *)out_grad)[offset + 1];
    tmp_h2 = reinterpret_cast<__half2 *>(&vtmp);
    tmp_h2_1 = reinterpret_cast<__half2 *>(&vtmp_1);
    float4 gamma_f4 = ((const float4 *)gamma)[threadIdx.x * 2];
    float4 gamma_f4_1 = ((const float4 *)gamma)[threadIdx.x * 2 + 1];
    __half2 *gamma_h2 = reinterpret_cast<__half2 *>(&gamma_f4);
    __half2 *gamma_h2_1 = reinterpret_cast<__half2 *>(&gamma_f4_1);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 vdout = __half22float2(tmp_h2[i]);
      float2 vdout_1 = __half22float2(tmp_h2_1[i]);
      float2 vgamma = __half22float2(gamma_h2[i]);
      float2 vgamma_1 = __half22float2(gamma_h2_1[i]);
      dxhat[i].x = vdout.x * vgamma.x;
      dxhat[i].y = vdout.y * vgamma.y;
      dxhat_1[i].x = vdout_1.x * vgamma_1.x;
      dxhat_1[i].y = vdout_1.y * vgamma_1.y;
      reduce_val[0] += dxhat[i].x + dxhat[i].y + dxhat_1[i].x + dxhat_1[i].y;
    }

    /*
    step 1. xhat = (output - betta) / gamma or
    (input - mean) * rsqrtf(var)
    */
    vtmp = ((const float4 *)inp_or_out)[offset];
    vtmp_1 = ((const float4 *)inp_or_out)[offset + 1];
    var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);
    if (means == nullptr) {
      // inp_or_out is output, xhat = (output - betta) / gamma
      float4 vbetta = ((const float4 *)betta)[2 * threadIdx.x];
      float4 vbetta_1 = ((const float4 *)betta)[2 * threadIdx.x + 1];
      __half2 *betta_h2 = reinterpret_cast<__half2 *>(&vbetta);
      __half2 *betta_h2_1 = reinterpret_cast<__half2 *>(&vbetta_1);
#pragma unroll
      for (int i = 0; i < 4; i++) {
        float2 vout = __half22float2(tmp_h2[i]);
        float2 vout_1 = __half22float2(tmp_h2_1[i]);
        float2 vgamma = __half22float2(gamma_h2[i]);
        float2 vgamma_1 = __half22float2(gamma_h2_1[i]);
        float2 vbetta = __half22float2(betta_h2[i]);
        float2 vbetta_1 = __half22float2(betta_h2_1[i]);
        xhat[i].x = (vout.x - vbetta.x) / add_eps(vgamma.x);
        xhat_1[i].x = (vout_1.x - vbetta_1.x) / add_eps(vgamma_1.x);
        xhat[i].y = (vout.y - vbetta.y) / add_eps(vgamma.y);
        xhat_1[i].y = (vout_1.y - vbetta_1.y) / add_eps(vgamma_1.y);
        reduce_val[1] += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
        reduce_val[1] +=
            xhat_1[i].x * dxhat_1[i].x + xhat_1[i].y * dxhat_1[i].y;
      }
    } else {
      // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
      float fmean = (float)means[blockIdx.x];
#pragma unroll
      for (int i = 0; i < 4; i++) {
        float2 vinp = __half22float2(tmp_h2[i]);
        float2 vinp_1 = __half22float2(tmp_h2_1[i]);
        xhat[i].x = (vinp.x - fmean) * var_rsqrt;
        xhat_1[i].x = (vinp_1.x - fmean) * var_rsqrt;
        xhat[i].y = (vinp.y - fmean) * var_rsqrt;
        xhat_1[i].y = (vinp_1.y - fmean) * var_rsqrt;
        reduce_val[1] += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
        reduce_val[1] +=
            xhat_1[i].x * dxhat_1[i].x + xhat_1[i].y * dxhat_1[i].y;
      }
    }
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float mean_dim = hidden_dim * 8 * 2;
    s_sum_dxhat = reduce_val[0] / mean_dim;
    s_sum_dxhat_xhat = reduce_val[1] / mean_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / mean_dim) * rsqrt(var)
  */
  if (threadIdx.x >= hidden_dim) {
    return;
  }
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    float4 dresidual_1 = ((const float4 *)residual_grad)[offset + 1];
    __half *hdres = reinterpret_cast<__half *>(&dresidual);
    __half *hdres_1 = reinterpret_cast<__half *>(&dresidual_1);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i]));
      tmp_h2_1[i].x = __float2half(
          (dxhat_1[i].x - s_sum_dxhat - xhat_1[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_1[2 * i]));
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i + 1]));
      tmp_h2_1[i].y = __float2half(
          (dxhat_1[i].y - s_sum_dxhat - xhat_1[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_1[2 * i + 1]));
    }
  } else {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_1[i].x = __float2half(
          (dxhat_1[i].x - s_sum_dxhat - xhat_1[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_1[i].y = __float2half(
          (dxhat_1[i].y - s_sum_dxhat - xhat_1[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
    }
  }
  ((float4 *)inp_grad)[offset] = vtmp;
  ((float4 *)inp_grad)[offset + 1] = vtmp_1;
}

__global__ void ker_ln_bw_dinp_x4(__half *inp_grad, const __half *out_grad,
                                  const __half *residual_grad,
                                  const __half *inp_or_out, const __half *gamma,
                                  const __half *betta, const __half *vars,
                                  const __half *means, int hidden_dim) {
  int offset = blockIdx.x * hidden_dim * 4 + threadIdx.x * 4;

  float2 dxhat[4], xhat[4];
  float2 dxhat_1[4], xhat_1[4];
  float2 dxhat_2[4], xhat_2[4];
  float2 dxhat_3[4], xhat_3[4];
  float var_rsqrt;
  float4 vtmp, vtmp_1, vtmp_2, vtmp_3;
  __half2 *tmp_h2;
  __half2 *tmp_h2_1;
  __half2 *tmp_h2_2;
  __half2 *tmp_h2_3;
  float reduce_val[2] = {0.f, 0.f};

  if (threadIdx.x < hidden_dim) {
    // step 0. dxhat = dout * gamma
    vtmp = ((const float4 *)out_grad)[offset];
    vtmp_1 = ((const float4 *)out_grad)[offset + 1];
    vtmp_2 = ((const float4 *)out_grad)[offset + 2];
    vtmp_3 = ((const float4 *)out_grad)[offset + 3];
    tmp_h2 = reinterpret_cast<__half2 *>(&vtmp);
    tmp_h2_1 = reinterpret_cast<__half2 *>(&vtmp_1);
    tmp_h2_2 = reinterpret_cast<__half2 *>(&vtmp_2);
    tmp_h2_3 = reinterpret_cast<__half2 *>(&vtmp_3);
    float4 gamma_f4 = ((const float4 *)gamma)[threadIdx.x * 4];
    float4 gamma_f4_1 = ((const float4 *)gamma)[threadIdx.x * 4 + 1];
    float4 gamma_f4_2 = ((const float4 *)gamma)[threadIdx.x * 4 + 2];
    float4 gamma_f4_3 = ((const float4 *)gamma)[threadIdx.x * 4 + 3];
    __half2 *gamma_h2 = reinterpret_cast<__half2 *>(&gamma_f4);
    __half2 *gamma_h2_1 = reinterpret_cast<__half2 *>(&gamma_f4_1);
    __half2 *gamma_h2_2 = reinterpret_cast<__half2 *>(&gamma_f4_2);
    __half2 *gamma_h2_3 = reinterpret_cast<__half2 *>(&gamma_f4_3);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 vdout = __half22float2(tmp_h2[i]);
      float2 vdout_1 = __half22float2(tmp_h2_1[i]);
      float2 vdout_2 = __half22float2(tmp_h2_2[i]);
      float2 vdout_3 = __half22float2(tmp_h2_3[i]);
      float2 vgamma = __half22float2(gamma_h2[i]);
      float2 vgamma_1 = __half22float2(gamma_h2_1[i]);
      float2 vgamma_2 = __half22float2(gamma_h2_2[i]);
      float2 vgamma_3 = __half22float2(gamma_h2_3[i]);
      dxhat[i].x = vdout.x * vgamma.x;
      dxhat[i].y = vdout.y * vgamma.y;
      dxhat_1[i].x = vdout_1.x * vgamma_1.x;
      dxhat_1[i].y = vdout_1.y * vgamma_1.y;
      dxhat_2[i].x = vdout_2.x * vgamma_2.x;
      dxhat_2[i].y = vdout_2.y * vgamma_2.y;
      dxhat_3[i].x = vdout_3.x * vgamma_3.x;
      dxhat_3[i].y = vdout_3.y * vgamma_3.y;
      reduce_val[0] += dxhat[i].x + dxhat[i].y + dxhat_1[i].x + dxhat_1[i].y +
                       dxhat_2[i].x + dxhat_2[i].y + dxhat_3[i].x +
                       dxhat_3[i].y;
    }

    /*
    step 1. xhat = (output - betta) / gamma or
    (input - mean) * rsqrtf(var)
    */
    vtmp = ((const float4 *)inp_or_out)[offset];
    vtmp_1 = ((const float4 *)inp_or_out)[offset + 1];
    vtmp_2 = ((const float4 *)inp_or_out)[offset + 2];
    vtmp_3 = ((const float4 *)inp_or_out)[offset + 3];
    var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);
    if (means == nullptr) {
      // inp_or_out is output, xhat = (output - betta) / gamma
      float4 vbetta = ((const float4 *)betta)[4 * threadIdx.x];
      float4 vbetta_1 = ((const float4 *)betta)[4 * threadIdx.x + 1];
      float4 vbetta_2 = ((const float4 *)betta)[4 * threadIdx.x + 2];
      float4 vbetta_3 = ((const float4 *)betta)[4 * threadIdx.x + 3];
      __half2 *betta_h2 = reinterpret_cast<__half2 *>(&vbetta);
      __half2 *betta_h2_1 = reinterpret_cast<__half2 *>(&vbetta_1);
      __half2 *betta_h2_2 = reinterpret_cast<__half2 *>(&vbetta_2);
      __half2 *betta_h2_3 = reinterpret_cast<__half2 *>(&vbetta_3);
#pragma unroll
      for (int i = 0; i < 4; i++) {
        float2 vout = __half22float2(tmp_h2[i]);
        float2 vout_1 = __half22float2(tmp_h2_1[i]);
        float2 vout_2 = __half22float2(tmp_h2_2[i]);
        float2 vout_3 = __half22float2(tmp_h2_3[i]);
        float2 vgamma = __half22float2(gamma_h2[i]);
        float2 vgamma_1 = __half22float2(gamma_h2_1[i]);
        float2 vgamma_2 = __half22float2(gamma_h2_2[i]);
        float2 vgamma_3 = __half22float2(gamma_h2_3[i]);
        float2 vbetta = __half22float2(betta_h2[i]);
        float2 vbetta_1 = __half22float2(betta_h2_1[i]);
        float2 vbetta_2 = __half22float2(betta_h2_2[i]);
        float2 vbetta_3 = __half22float2(betta_h2_3[i]);
        xhat[i].x = (vout.x - vbetta.x) / add_eps(vgamma.x);
        xhat_1[i].x = (vout_1.x - vbetta_1.x) / add_eps(vgamma_1.x);
        xhat_2[i].x = (vout_2.x - vbetta_2.x) / add_eps(vgamma_2.x);
        xhat_3[i].x = (vout_3.x - vbetta_3.x) / add_eps(vgamma_3.x);
        xhat[i].y = (vout.y - vbetta.y) / add_eps(vgamma.y);
        xhat_1[i].y = (vout_1.y - vbetta_1.y) / add_eps(vgamma_1.y);
        xhat_2[i].y = (vout_2.y - vbetta_2.y) / add_eps(vgamma_2.y);
        xhat_3[i].y = (vout_3.y - vbetta_3.y) / add_eps(vgamma_3.y);
        reduce_val[1] += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
        reduce_val[1] +=
            xhat_1[i].x * dxhat_1[i].x + xhat_1[i].y * dxhat_1[i].y;
        reduce_val[1] +=
            xhat_2[i].x * dxhat_2[i].x + xhat_2[i].y * dxhat_2[i].y;
        reduce_val[1] +=
            xhat_3[i].x * dxhat_3[i].x + xhat_3[i].y * dxhat_3[i].y;
      }
    } else {
      // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
      float fmean = (float)means[blockIdx.x];
#pragma unroll
      for (int i = 0; i < 4; i++) {
        float2 vinp = __half22float2(tmp_h2[i]);
        float2 vinp_1 = __half22float2(tmp_h2_1[i]);
        float2 vinp_2 = __half22float2(tmp_h2_2[i]);
        float2 vinp_3 = __half22float2(tmp_h2_3[i]);
        xhat[i].x = (vinp.x - fmean) * var_rsqrt;
        xhat_1[i].x = (vinp_1.x - fmean) * var_rsqrt;
        xhat_2[i].x = (vinp_2.x - fmean) * var_rsqrt;
        xhat_3[i].x = (vinp_3.x - fmean) * var_rsqrt;
        xhat[i].y = (vinp.y - fmean) * var_rsqrt;
        xhat_1[i].y = (vinp_1.y - fmean) * var_rsqrt;
        xhat_2[i].y = (vinp_2.y - fmean) * var_rsqrt;
        xhat_3[i].y = (vinp_3.y - fmean) * var_rsqrt;
        reduce_val[1] += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
        reduce_val[1] +=
            xhat_1[i].x * dxhat_1[i].x + xhat_1[i].y * dxhat_1[i].y;
        reduce_val[1] +=
            xhat_2[i].x * dxhat_2[i].x + xhat_2[i].y * dxhat_2[i].y;
        reduce_val[1] +=
            xhat_3[i].x * dxhat_3[i].x + xhat_3[i].y * dxhat_3[i].y;
      }
    }
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float mean_dim = hidden_dim * 8 * 4;
    s_sum_dxhat = reduce_val[0] / mean_dim;
    s_sum_dxhat_xhat = reduce_val[1] / mean_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / mean_dim) * rsqrt(var)
  */
  if (threadIdx.x >= hidden_dim) {
    return;
  }
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    float4 dresidual_1 = ((const float4 *)residual_grad)[offset + 1];
    float4 dresidual_2 = ((const float4 *)residual_grad)[offset + 2];
    float4 dresidual_3 = ((const float4 *)residual_grad)[offset + 3];
    __half *hdres = reinterpret_cast<__half *>(&dresidual);
    __half *hdres_1 = reinterpret_cast<__half *>(&dresidual_1);
    __half *hdres_2 = reinterpret_cast<__half *>(&dresidual_2);
    __half *hdres_3 = reinterpret_cast<__half *>(&dresidual_3);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i]));
      tmp_h2_1[i].x = __float2half(
          (dxhat_1[i].x - s_sum_dxhat - xhat_1[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_1[2 * i]));
      tmp_h2_2[i].x = __float2half(
          (dxhat_2[i].x - s_sum_dxhat - xhat_2[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_2[2 * i]));
      tmp_h2_3[i].x = __float2half(
          (dxhat_3[i].x - s_sum_dxhat - xhat_3[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_3[2 * i]));
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i + 1]));
      tmp_h2_1[i].y = __float2half(
          (dxhat_1[i].y - s_sum_dxhat - xhat_1[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_1[2 * i + 1]));
      tmp_h2_2[i].y = __float2half(
          (dxhat_2[i].y - s_sum_dxhat - xhat_2[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_1[2 * i + 1]));
      tmp_h2_3[i].y = __float2half(
          (dxhat_3[i].y - s_sum_dxhat - xhat_3[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres_1[2 * i + 1]));
    }
  } else {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_1[i].x = __float2half(
          (dxhat_1[i].x - s_sum_dxhat - xhat_1[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_2[i].x = __float2half(
          (dxhat_2[i].x - s_sum_dxhat - xhat_2[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_3[i].x = __float2half(
          (dxhat_3[i].x - s_sum_dxhat - xhat_3[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_1[i].y = __float2half(
          (dxhat_1[i].y - s_sum_dxhat - xhat_1[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_2[i].y = __float2half(
          (dxhat_2[i].y - s_sum_dxhat - xhat_2[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2_3[i].y = __float2half(
          (dxhat_3[i].y - s_sum_dxhat - xhat_3[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
    }
  }
  ((float4 *)inp_grad)[offset] = vtmp;
  ((float4 *)inp_grad)[offset + 1] = vtmp_1;
  ((float4 *)inp_grad)[offset + 2] = vtmp_2;
  ((float4 *)inp_grad)[offset + 3] = vtmp_3;
}

/**
Layer norm backword,
  compute the gradient of gamma, betta and input.
dbetta = sum(dout, dim=0)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
  (output - betta) / gamma if mean is nullptr
dgamma = sum(xhat * dout, dim=0)
dxhat = dout * gamma
dinp = (dxhat - (sum(dxhat, 1) + xhat * sum(dxhat * xhat, 1)) / hidden_dim)
  * rsqrt(var)

residual_grad, means, betta can be nullptr.
residual_grad will be added to dinp if it is not nullptr
  which is useful in transformer layer when pre-ln
means and betta are only used to compute xhat,
  (means == nullptr) ^ (betta == nullptr) should be true
*/
template <>
void launch_ln_bw<float>(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *residual_grad,
                         const float *inp_or_out, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch, int hidden_dim,
                         hipStream_t stream[2]) {
  // compute grad of gamma and betta
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream[0]>>>(
      gamma_grad, betta_grad, out_grad, inp_or_out, gamma, betta, vars, means,
      batch, hidden_dim);

  // compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch, nthread, 0, stream[1]>>>(
      inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars, means,
      hidden_dim);
}

template <>
void launch_ln_bw<__half>(__half *gamma_grad, __half *betta_grad,
                          __half *inp_grad, const __half *out_grad,
                          const __half *residual_grad, const __half *inp_or_out,
                          const __half *gamma, const __half *betta,
                          const __half *vars, const __half *means, int batch,
                          int hidden_dim, hipStream_t stream[2]) {
  // compute grad of gamma and betta
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
      gamma_grad, betta_grad, out_grad, inp_or_out, gamma, betta, vars, means,
      batch, hidden_dim);

  // compute grad of input
  if (hidden_dim % 8 != 0) {
    throw std::runtime_error("hidden_dim % 8 != 0");
  }
  hidden_dim >>= 3;

  if (hidden_dim * 8 <= 8192) {
    int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
    ker_ln_bw_dinp<<<batch, nthread, 0, stream[1]>>>(
        inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars,
        means, hidden_dim);
  } else if (hidden_dim * 8 > 8192 && hidden_dim * 8 <= 8192 * 2) {
    hidden_dim >>= 1;
    int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
    ker_ln_bw_dinp_x2<<<batch, nthread, 0, stream[1]>>>(
        inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars,
        means, hidden_dim);
  } else if (hidden_dim * 8 > 2 * 8192 && hidden_dim * 8 <= 8192 * 4) {
    hidden_dim >>= 2;
    int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
    ker_ln_bw_dinp_x4<<<batch, nthread, 0, stream[1]>>>(
        inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars,
        means, hidden_dim);
  } else {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 32768");
  }
}

#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <math.h>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>

#include "block_reduce.h"
#include "kernels.h"

namespace cg = cooperative_groups;
const float EPSILON = 1e-8f;

/**
@brief: softmax_kernel
Softmax forward kernel for
  enc-self-attn, dec-self-attn, encdec-attn

@thread
gridDim.x = dynamic
gridDim.y = batch_size
gridDim.z = nhead
blockDim.x = from_len

@param
inp: [batch_size, nhead, from_len, to_len], softmax input.
attn_mask: [batch_size, to_len], padding tokens are -inf,
  non padding tokens are 0.
  attn_mask!=nullptr for enc-self-attn and enc-dec-attn
  attn_mask=nullptr and mask_future=ture for dec-self-attn training
  attn_mask=nullptr and mask_future=false for dec-self-attn infer
*/
template <typename T, int block_dim, int ele_per_thread>
__global__ void ker_attn_softmax(T *inp, const T *attn_mask, int from_len,
                                 int to_len, bool mask_future) {
  int batch_id = blockIdx.y;
  int head_id = blockIdx.z;
  const int nhead = gridDim.z;
  const int token_per_reduce = 1;
  typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;
  typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  T mval[ele_per_thread];
  if (attn_mask) {
    attn_mask += batch_id * to_len;
    BlockLoad(ts_load).Load(attn_mask, mval, to_len, REDUCE_FLOAT_INF_NEG);
  }

  inp += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  for (int token_id = blockIdx.x * token_per_reduce; token_id < from_len;
       token_id += gridDim.x * token_per_reduce) {
    T inp_val[token_per_reduce][ele_per_thread];
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      BlockLoad(ts_load).Load(inp + (token_id + i) * to_len, inp_val[i], to_len,
                              REDUCE_FLOAT_INF_NEG);
    }

    /* step 1. compute max */
    // thread local max
    float val[token_per_reduce][ele_per_thread];
    float l_max[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_max[i] = REDUCE_FLOAT_INF_NEG;
      for (int j = 0; j < ele_per_thread; j++) {
        if (attn_mask) {
          val[i][j] = (float)inp_val[i][j] + (float)mval[j];
        } else {
          if (mask_future && ele_per_thread * threadIdx.x + j > token_id + i) {
            val[i][j] = REDUCE_FLOAT_INF_NEG;
          } else {
            val[i][j] = (float)inp_val[i][j];
          }
        }
        l_max[i] = fmaxf(l_max[i], val[i][j]);
      }
    }
    // block reduce max
    blockReduce<ReduceType::kMax, token_per_reduce>(l_max);
    // write shared
    __shared__ float s_max[token_per_reduce];
    if (threadIdx.x == 0) {
      for (int i = 0; i < token_per_reduce; i++) {
        s_max[i] = l_max[i];
      }
    }
    __syncthreads();

    /* step 2. compute sum */
    // thread local sum
    float l_sum[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_sum[i] = 0.f;
      for (int j = 0; j < ele_per_thread; j++) {
        val[i][j] = __expf(val[i][j] - s_max[i]);
        l_sum[i] += val[i][j];
      }
    }
    // block reduce sum
    blockReduce<ReduceType::kSum, token_per_reduce>(l_sum);
    // write shared
    __shared__ float s_sum[token_per_reduce];
    if (threadIdx.x == 0) {
      for (int i = 0; i < token_per_reduce; i++) {
        s_sum[i] = __fdividef(1.0f, l_sum[i] + EPSILON);
      }
    }
    __syncthreads();

    /* step 3. compute final result */
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      for (int j = 0; j < ele_per_thread; j++) {
        inp_val[i][j] = (T)(val[i][j] * s_sum[i]);
      }
      BlockStore(ts_store).Store(inp + (token_id + i) * to_len, inp_val[i],
                                 to_len);
    }
  }  // blockIdx.x
}

template <typename T, int block_dim, int ele_per_thread>
__global__ void ker_attn_softmax_lt32(T *inp, const T *attn_mask, int from_len,
                                      int to_len, bool mask_future) {
  int batch_id = blockIdx.y;
  int head_id = blockIdx.z;
  const int nhead = gridDim.z;
  const int token_per_reduce = 1;
  typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;
  typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  T mval[ele_per_thread];
  if (attn_mask) {
    attn_mask += batch_id * to_len;
    BlockLoad(ts_load).Load(attn_mask, mval, to_len, REDUCE_FLOAT_INF_NEG);
  }

  inp += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  for (int token_id = blockIdx.x * token_per_reduce; token_id < from_len;
       token_id += gridDim.x * token_per_reduce) {
    T inp_val[token_per_reduce][ele_per_thread];
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      BlockLoad(ts_load).Load(inp + (token_id + i) * to_len, inp_val[i], to_len,
                              REDUCE_FLOAT_INF_NEG);
    }

    /* step 1. compute max */
    // thread local max
    float val[token_per_reduce][ele_per_thread];
    float l_max[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_max[i] = REDUCE_FLOAT_INF_NEG;
      for (int j = 0; j < ele_per_thread; j++) {
        if (attn_mask) {
          val[i][j] = (float)inp_val[i][j] + (float)mval[j];
        } else {
          if (mask_future && ele_per_thread * threadIdx.x + j > token_id + i) {
            val[i][j] = REDUCE_FLOAT_INF_NEG;
          } else {
            val[i][j] = (float)inp_val[i][j];
          }
        }
        l_max[i] = fmaxf(l_max[i], val[i][j]);
      }
    }
    // warp reduce max
    warpReduce<ReduceType::kMax, token_per_reduce>(l_max);

    /* step 2. compute sum */
    // thread local sum
    float l_sum[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_sum[i] = 0.f;
      for (int j = 0; j < ele_per_thread; j++) {
        val[i][j] = __expf(val[i][j] - l_max[i]);
        l_sum[i] += val[i][j];
      }
    }
    // warp reduce sum
    warpReduce<ReduceType::kSum, token_per_reduce>(l_sum);

    /* step 3. compute final result */
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      l_sum[i] = __fdividef(1.0f, l_sum[i] + EPSILON);
      for (int j = 0; j < ele_per_thread; j++) {
        inp_val[i][j] = (T)(val[i][j] * l_sum[i]);
      }
      BlockStore(ts_store).Store(inp + (token_id + i) * to_len, inp_val[i],
                                 to_len);
    }
  }  // blockIdx.x
}

/*
  attn_mask!=nullptr for enc-self-attn and enc-dec-attn
  attn_mask=nullptr and mask_future=ture for dec-self-attn training
  attn_mask=nullptr and mask_future=false for dec-self-attn infer
*/
template <>
void launch_attn_softmax<float>(float *inp, const float *attn_mask,
                                int batch_size, int nhead, int from_len,
                                int to_len, bool mask_future,
                                hipStream_t stream) {
  dim3 grid_dim(1, batch_size, nhead);
  if (to_len <= 32) {
    ker_attn_softmax_lt32<float, 32, 1><<<grid_dim, 32, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 64) {
    ker_attn_softmax_lt32<float, 32, 2><<<grid_dim, 32, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 128) {
    grid_dim.x = 16;
    ker_attn_softmax<float, 64, 2><<<grid_dim, 64, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 256) {
    grid_dim.x = 32;
    ker_attn_softmax<float, 128, 2><<<grid_dim, 128, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 512) {
    grid_dim.x = 64;
    ker_attn_softmax<float, 256, 2><<<grid_dim, 256, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else {
    throw std::runtime_error(
        "Sequence length greater than 512 is currently not supported");
  }
}

template <>
void launch_attn_softmax<__half>(__half *inp, const __half *attn_mask,
                                 int batch_size, int nhead, int from_len,
                                 int to_len, bool mask_future,
                                 hipStream_t stream) {
  dim3 grid_dim(1, batch_size, nhead);
  if (to_len <= 32) {
    ker_attn_softmax_lt32<__half, 32, 1><<<grid_dim, 32, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 64) {
    ker_attn_softmax_lt32<__half, 32, 2><<<grid_dim, 32, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 128) {
    grid_dim.x = 8;
    ker_attn_softmax<__half, 64, 2><<<grid_dim, 64, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 256) {
    grid_dim.x = 16;
    ker_attn_softmax<__half, 128, 2><<<grid_dim, 128, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 512) {
    grid_dim.x = 32;
    ker_attn_softmax<__half, 256, 2><<<grid_dim, 256, 0, stream>>>(
        inp, attn_mask, from_len, to_len, mask_future);
  } else {
    throw std::runtime_error(
        "Sequence length greater than 512 is currently not supported");
  }
}

/**
@brief: ker_attn_softmax_bw
Softmax backward in self attention.

@thread
gridDim.x = batch_size * nhead * seq_len / warps_per_block
blockDim.x = WARP_SIZE
blockDim.y = warps_per_block

@param
grad: [batch_size, nhead, seq_len, seq_len], output grad.
output: [batch_size, nhead, seq_len, seq_len], output of softmax forward.
*/
template <typename T, int ITERATIONS>
__global__ void ker_attn_softmax_bw(T *grad, const T *inp, int softmax_length) {
  int batch_idx = blockIdx.x * blockDim.y + threadIdx.y;
  int offset = batch_idx * softmax_length + threadIdx.x;

  grad += offset;
  inp += offset;

  T grad_reg[ITERATIONS];
  T inp_reg[ITERATIONS];
  float sum = 0.0;

#pragma unroll
  for (int i = 0; i < ITERATIONS; ++i) {
    int curr_idx = threadIdx.x + i * WARP_SIZE;
    if (curr_idx < softmax_length) {
      grad_reg[i] = grad[i * WARP_SIZE];
      inp_reg[i] = inp[i * WARP_SIZE];
      sum += (float)grad_reg[i] * (float)inp_reg[i];
    }
  }

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_xor(sum, i);

#pragma unroll
  for (int i = 0; i < ITERATIONS; ++i) {
    int curr_idx = threadIdx.x + i * WARP_SIZE;
    if (curr_idx < softmax_length)
      grad[i * WARP_SIZE] = (T)((float)inp_reg[i] * ((float)grad_reg[i] - sum));
  }
}

template <typename T>
void launch_attn_softmax_bw(T *out_grad, const T *soft_inp, int rows,
                            int softmax_len, hipStream_t stream) {
  const int warps_per_block = 4;
  // rows = batch_size * nhead * from_len
  dim3 grid_dim(rows / warps_per_block);
  dim3 block_dim(WARP_SIZE, warps_per_block);

  if (softmax_len <= 32)
    ker_attn_softmax_bw<T, 1>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 64)
    ker_attn_softmax_bw<T, 2>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 128)
    ker_attn_softmax_bw<T, 4>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 256)
    ker_attn_softmax_bw<T, 8>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 384)
    ker_attn_softmax_bw<T, 12>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 512)
    ker_attn_softmax_bw<T, 16>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 768)
    ker_attn_softmax_bw<T, 24>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 1024)
    ker_attn_softmax_bw<T, 32>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else if (softmax_len <= 2048)
    ker_attn_softmax_bw<T, 64>
        <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, softmax_len);
  else
    throw std::runtime_error(
        std::string(
            "Special sequence length found in softmax backward, seq_len: ") +
        std::to_string(softmax_len));
}

template void launch_attn_softmax_bw<__half>(__half *out_grad,
                                             const __half *soft_inp, int rows,
                                             int softmax_len,
                                             hipStream_t stream);
template void launch_attn_softmax_bw<float>(float *out_grad,
                                            const float *soft_inp, int rows,
                                            int softmax_len,
                                            hipStream_t stream);

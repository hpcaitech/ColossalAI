#include "hip/hip_runtime.h"
#include <chrono>
#include <ctime>

#include "kernels.h"

#include <hip/hip_cooperative_groups.h>


namespace cg = cooperative_groups;

hiprandStatePhilox4_32_10_t *curandstate;

/**
 * @brief element-wise activation function on device, like Relu, Gelu
 *
 * @tparam enum class ActivationType, kRelu, kGelu
 * @tparam input type
 * @param any shape of float and __half2
 * @return same shape and type with input
 */
template <ActivationType, typename T>
__forceinline__ __device__ T activation_kernel(T x);

template <>
__device__ float activation_kernel<ActivationType::kGelu, float>(float x) {
  float cdf =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

template <>
__device__ __half2
activation_kernel<ActivationType::kGelu, __half2>(__half2 val) {
  __half2 val_pow3 = __hmul2(val, __hmul2(val, val));
  float2 tmp_pow = __half22float2(val_pow3);
  float2 tmp = __half22float2(val);

  tmp.x =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (tmp.x + 0.044715f * tmp_pow.x))));
  tmp.y =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (tmp.y + 0.044715f * tmp_pow.y))));
  return __hmul2(val, __float22half2_rn(tmp));
}

template <>
__device__ float activation_kernel<ActivationType::kRelu, float>(float x) {
  return fmaxf(x, 0);
}

template <>
__device__ __half2
activation_kernel<ActivationType::kRelu, __half2>(__half2 x) {
  return __floats2half2_rn(fmaxf(0.f, __half2float(x.x)),
                           fmaxf(0.f, __half2float(x.y)));
}

/**
 * @brief element-wise activation backward function on device
 *
 * @tparam enum class ActivationType
 * @tparam input type
 * @param any shape of float and __half2
 * @return same shape of input
 */
template <ActivationType, typename T>
__forceinline__ __device__ T activation_bwd_kernel(T grad, T x);

template <>
__device__ float activation_bwd_kernel<ActivationType::kGelu, float>(float grad,
                                                                     float x) {
  const float sqrt_param = 0.79788456080286535587989211986876f;
  const float mul_param = 0.044715;

  float x2mul = x * x * mul_param;
  float tan_h = tanhf(sqrt_param * (x + x * x2mul));
  float dg1 = 0.5f * (1.0f + tan_h);
  float dg2 = x * 0.5f * sqrt_param * (1 - tan_h * tan_h);
  float dg3 = dg2 * 3 * x2mul;
  return grad * (dg1 + dg2 + dg3);
}

template <>
__device__ __half activation_bwd_kernel<ActivationType::kGelu, __half>(
    __half grad, __half x_half) {
  float x = __half2float(x_half);
  const float sqrt_param = 0.79788456080286535587989211986876f;
  const float mul_param = 0.044715;

  float x2mul = x * x * mul_param;
  float tan_h = tanhf(sqrt_param * (x + x * x2mul));
  float dg1 = 0.5f * (1.0f + tan_h);
  float dg2 = x * 0.5f * sqrt_param * (1 - tan_h * tan_h);
  float dg3 = dg2 * 3 * x2mul;
  return grad * __float2half(dg1 + dg2 + dg3);
}

template <>
__device__ float activation_bwd_kernel<ActivationType::kRelu, float>(float grad,
                                                                     float x) {
  return x > 0.f ? grad : 0.f;
}

template <>
__device__ __half
activation_bwd_kernel<ActivationType::kRelu, __half>(__half grad, __half x) {
  const __half half_zero = __float2half(0.f);
  return x > half_zero ? grad : half_zero;
}

template <>
__device__ __half2 activation_bwd_kernel<ActivationType::kRelu, __half2>(
    __half2 grad2, __half2 x_half2) {
  const __half half_zero = __float2half(0.f);
  return __floats2half2_rn(x_half2.x > half_zero ? grad2.x : half_zero,
                           x_half2.y > half_zero ? grad2.y : half_zero);
}

/**
 * @brief init hiprand states in global memory
 *
 * @thread grid_dim * block*dim to suuport any size of states
 * @param state persistant hiprand states
 * @param seed seed to init states
 * @return void
 */
__global__ void curand_init_kernel(hiprandStatePhilox4_32_10_t *state,
                                   int seed) {
  /* Each thread gets same seed, a different sequence
     number, no offset */
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, id, 0, &state[id]);
}

void launch_curand_init(int total_count, int dim, hipStream_t stream) {
  hipMalloc(&curandstate, total_count * sizeof(hiprandStatePhilox4_32_10_t));
  int grid_dim = total_count >> 9;
  curand_init_kernel<<<grid_dim, 512, 0, stream>>>(
      curandstate, std::chrono::duration_cast<std::chrono::microseconds>(
                       std::chrono::system_clock::now().time_since_epoch())
                       .count());
}

/**
 * @brief element-wise dropout, store dropped position in mask, it's not
 * in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out any size of float and __half
 * @param in same with out
 * @param mask uint8 type, same size with out
 * @param seed seed to hiprand
 * @return void
 */
__global__ void ls_dropout_kernel(const int total_count, const float ratio,
                                  float *__restrict__ out,
                                  const float *__restrict__ in,
                                  uint8_t *__restrict__ mask, const int seed) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *data4 = reinterpret_cast<const float4 *>(in);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);

  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] = m4[0];

  float4 input4 = data4[i];
  float4 res4;
  res4.x = input4.x * scale * m[0];
  res4.y = input4.y * scale * m[1];
  res4.z = input4.z * scale * m[2];
  res4.w = input4.w * scale * m[3];
  out4[i] = res4;
}

__global__ void ls_dropout_kernel(const int total_count, const float ratio,
                                  __half *__restrict__ out,
                                  const __half *__restrict__ in,
                                  uint8_t *__restrict__ mask, const int seed) {
  const float scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = (uint8_t)(rand.x > ratio);
  m[5] = (uint8_t)(rand.y > ratio);
  m[6] = (uint8_t)(rand.z > ratio);
  m[7] = (uint8_t)(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] = *m8;

  float4 val_float4 = vals_float4[i];
  float4 out_float4;
  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  __half2 scale_mask_1 = __floats2half2_rn(scale * m[0], scale * m[1]);
  __half2 scale_mask_2 = __floats2half2_rn(scale * m[2], scale * m[3]);
  __half2 scale_mask_3 = __floats2half2_rn(scale * m[4], scale * m[5]);
  __half2 scale_mask_4 = __floats2half2_rn(scale * m[6], scale * m[7]);
  out_half2[0] = __hmul2(val_half2[0], scale_mask_1);
  out_half2[1] = __hmul2(val_half2[1], scale_mask_2);
  out_half2[2] = __hmul2(val_half2[2], scale_mask_3);
  out_half2[3] = __hmul2(val_half2[3], scale_mask_4);
  outs_float4[i] = out_float4;
}

/**
 * @brief element-wise dropout backward with dropout mask, it's
 * not in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param in any size of float and __half
 * @param mask uint8 type, same size with in
 * @return void
 */
__global__ void ls_dropout_bwd_kernel(const int total_count, const float ratio,
                                      float *out, const float *in,
                                      const uint8_t *__restrict__ mask) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *in4 = reinterpret_cast<const float4 *>(in);
  const uint32_t *mask4 = reinterpret_cast<const uint32_t *>(mask);

  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  m4[0] = mask4[i];

  float4 input4 = in4[i];
  float4 res4;
  res4.x = input4.x * scale * static_cast<float>(m[0]);
  res4.y = input4.y * scale * static_cast<float>(m[1]);
  res4.z = input4.z * scale * static_cast<float>(m[2]);
  res4.w = input4.w * scale * static_cast<float>(m[3]);
  out4[i] = res4;
}

__global__ void ls_dropout_bwd_kernel(const int total_count, const float ratio,
                                      __half *out, const __half *in,
                                      const uint8_t *__restrict__ mask) {
  const __half scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  const uint64_t *mask8 = reinterpret_cast<const uint64_t *>(mask);

  uint8_t m[8];
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  m8[0] = mask8[i];

  float4 val_float4 = vals_float4[i];
  float4 out_float4;
  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  __half2 scale_mask_1 =
      __halves2half2(scale * __float2half(m[0]), scale * __float2half(m[1]));
  __half2 scale_mask_2 =
      __halves2half2(scale * __float2half(m[2]), scale * __float2half(m[3]));
  __half2 scale_mask_3 =
      __halves2half2(scale * __float2half(m[4]), scale * __float2half(m[5]));
  __half2 scale_mask_4 =
      __halves2half2(scale * __float2half(m[6]), scale * __float2half(m[7]));
  out_half2[0] = __hmul2(val_half2[0], scale_mask_1);
  out_half2[1] = __hmul2(val_half2[1], scale_mask_2);
  out_half2[2] = __hmul2(val_half2[2], scale_mask_3);
  out_half2[3] = __hmul2(val_half2[3], scale_mask_4);
  out4[i] = out_float4;
}

template <>
void launch_ls_dropout<float>(float *out, const float *vals, uint8_t *mask,
                              int total_count, float ratio, hipStream_t stream,
                              bool backward) {
  int grid_dim = total_count >> 12;
  if (!backward) {
    ls_dropout_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
        total_count, ratio, out, vals, mask,
        std::chrono::duration_cast<std::chrono::microseconds>(
            std::chrono::system_clock::now().time_since_epoch())
            .count());
  } else {
    ls_dropout_bwd_kernel<<<grid_dim + 1, 1024, 0, stream>>>(total_count, ratio,
                                                             out, vals, mask);
  }
}

template <>
void launch_ls_dropout<__half>(__half *out, const __half *vals, uint8_t *mask,
                               int total_count, float ratio,
                               hipStream_t stream, bool backward) {
  int grid_dim = total_count >> 13;
  if (!backward) {
    ls_dropout_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
        total_count, ratio, out, vals, mask,
        std::chrono::duration_cast<std::chrono::microseconds>(
            std::chrono::system_clock::now().time_since_epoch())
            .count());
  } else {
    ls_dropout_bwd_kernel<<<grid_dim + 1, 1024, 0, stream>>>(total_count, ratio,
                                                             out, vals, mask);
  }
}

/**
 * @brief fused bias, dropout, and residual at the end of Attention and FFN,
 * store dropped position in mask, it's not in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param residual [batch_size, seq_len, hidden_size], float and __half
 * @param seed seed to hiprand
 * @param hidden_size hidden size
 * @return void
 */
__global__ void ls_dropout_res_bias_kernel(
    const int total_count, const float ratio, float *__restrict__ out,
    const float *__restrict__ in, uint8_t *__restrict__ mask,
    const float *__restrict__ bias, const float *__restrict__ residual,
    const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *data4 = reinterpret_cast<const float4 *>(in);
  const float4 *residual4 = reinterpret_cast<const float4 *>(residual);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = static_cast<uint8_t>(rand.x > ratio);
  m[1] = static_cast<uint8_t>(rand.y > ratio);
  m[2] = static_cast<uint8_t>(rand.z > ratio);
  m[3] = static_cast<uint8_t>(rand.w > ratio);

  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] = m4[0];
  const float4 input4 = data4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  const float4 res4 = residual4[i];
  float4 output4;

  output4.x = (input4.x + b4.x) * scale * m[0] + res4.x;
  output4.y = (input4.y + b4.y) * scale * m[1] + res4.y;
  output4.z = (input4.z + b4.z) * scale * m[2] + res4.z;
  output4.w = (input4.w + b4.w) * scale * m[3] + res4.w;

  out4[i] = output4;
}

__global__ void ls_dropout_res_bias_kernel(
    const int total_count, const float ratio, __half *__restrict__ out,
    const __half *__restrict__ in, uint8_t *__restrict__ mask,
    const __half *__restrict__ bias, const __half *__restrict__ residual,
    const int seed, const int hidden_size) {
  const __half scale = 1. / (1. - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  const float4 *residual4 = reinterpret_cast<const float4 *>(residual);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = static_cast<uint8_t>(rand.x > ratio);
  m[1] = static_cast<uint8_t>(rand.y > ratio);
  m[2] = static_cast<uint8_t>(rand.z > ratio);
  m[3] = static_cast<uint8_t>(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = static_cast<uint8_t>(rand.x > ratio);
  m[5] = static_cast<uint8_t>(rand.y > ratio);
  m[6] = static_cast<uint8_t>(rand.z > ratio);
  m[7] = static_cast<uint8_t>(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] = m8[0];

  int bias_i = i % (hidden_size >> 3);
  float4 val_float4 = vals_float4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  const float4 res4 = residual4[i];
  float4 out_float4;

  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);
  const __half2 *res_half2 = reinterpret_cast<const __half2 *>(&res4);
  __half2 scale_mask_1 =
      __halves2half2(scale * __float2half(m[0]), scale * __float2half(m[1]));
  __half2 scale_mask_2 =
      __halves2half2(scale * __float2half(m[2]), scale * __float2half(m[3]));
  __half2 scale_mask_3 =
      __halves2half2(scale * __float2half(m[4]), scale * __float2half(m[5]));
  __half2 scale_mask_4 =
      __halves2half2(scale * __float2half(m[6]), scale * __float2half(m[7]));
  out_half2[0] =
      __hfma2(__hadd2(val_half2[0], b_half2[0]), scale_mask_1, res_half2[0]);
  out_half2[1] =
      __hfma2(__hadd2(val_half2[1], b_half2[1]), scale_mask_2, res_half2[1]);
  out_half2[2] =
      __hfma2(__hadd2(val_half2[2], b_half2[2]), scale_mask_3, res_half2[2]);
  out_half2[3] =
      __hfma2(__hadd2(val_half2[3], b_half2[3]), scale_mask_4, res_half2[3]);
  outs_float4[i] = out_float4;
}

template <>
void launch_ls_dropout_res_bias<float>(float *out, const float *vals,
                                       uint8_t *mask, const float *bias,
                                       const float *residual, int total_count,
                                       int dim, float ratio,
                                       hipStream_t stream) {
  int grid_dim = total_count >> 12;
  ls_dropout_res_bias_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      total_count, ratio, out, vals, mask, bias, residual,
      std::chrono::duration_cast<std::chrono::microseconds>(
          std::chrono::system_clock::now().time_since_epoch())
          .count(),
      dim);
}

template <>
void launch_ls_dropout_res_bias<__half>(__half *out, const __half *vals,
                                        uint8_t *mask, const __half *bias,
                                        const __half *residual, int total_count,
                                        int dim, float ratio,
                                        hipStream_t stream) {
  int grid_dim = total_count >> 13;
  ls_dropout_res_bias_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      total_count, ratio, out, vals, mask, bias, residual,
      std::chrono::duration_cast<std::chrono::microseconds>(
          std::chrono::system_clock::now().time_since_epoch())
          .count(),
      dim);
}

/**
 * @brief fused bias and dropout backward at the end of Attention and FFN
 *
 * @thread
 * gridDim.x = hidden_size / 8
 * blockDim.x = 8
 * blockDim.y = 1024 / 8 = 128
 *
 * @param row_size batch_size * seq_len
 * @param ratio dropout ratio
 * @param in_grad [batch_size, seq_len, hidden_size], input grad
 * @param bias_grad [hidden_size], bias grad
 * @param out_grad [batch_size, seq_len, hidden_size], output grad
 * @param mask [batch_size, seq_len, hidden_size], dropout mask
 * @param hidden_size
 * @return void
 */
__global__ void ls_dropout_bias_bwd_kernel(
    const int row_size, const float ratio, float *__restrict__ in_grad,
    float *__restrict__ bias_grad, const float *__restrict__ out_grad,
    const uint8_t *__restrict__ mask, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  // every block generate 8 bias result
  __shared__ float tile[8][129];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, 8);
  int stride = hidden_size * 128;
  float local_sum = 0;

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
  for (int r = threadIdx.y; r < row_size; r += 128) {
    float val = out_grad[idx];
    val *= scale * static_cast<float>(mask[idx]);
    local_sum += val;
    in_grad[idx] = val;
    idx += stride;
  }

  tile[threadIdx.x][threadIdx.y] = local_sum;
  __syncthreads();

  float sum = 0;
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int x = tid >> 7;
  int y = tid & (127);
  if (y < 32) {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      sum += tile[x][y + i * 32];
    }
  }
  __syncthreads();

  for (int i = 1; i < 32; i <<= 1) sum += g.shfl_down(sum, i);

  if (y == 0) tile[0][x] = sum;
  __syncthreads();

  if (threadIdx.x < 8) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, 8);
    bias_grad[pos] = tile[0][threadIdx.x];
  }
}

__global__ void ls_dropout_bias_bwd_kernel(
    const int row_size, const float ratio, __half *__restrict__ in_grad,
    __half *__restrict__ bias_grad, const __half *__restrict__ out_grad,
    const uint8_t *__restrict__ mask, const int hidden_size) {
  const __half2 scale = __float2half2_rn(1.f / (1.f - ratio));
  __shared__ __half2 tile[8][129];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  __half2 *in_grad2 = reinterpret_cast<__half2 *>(in_grad);
  const __half2 *out_grad2 = reinterpret_cast<const __half2 *>(out_grad);
  __half2 *bias_grad2 = reinterpret_cast<__half2 *>(bias_grad);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, 8);
  int stride = hidden_size * 128;
  __half2 local_sum = __float2half2_rn(0.f);

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
  for (int r = threadIdx.y; r < row_size; r += 128) {
    __half2 val = out_grad2[idx];
    __half2 m2 = __floats2half2_rn(mask[2 * idx], mask[2 * idx + 1]);
    val *= scale * m2;
    local_sum += val;
    in_grad2[idx] = val;
    idx += stride;
  }

  tile[threadIdx.x][threadIdx.y] = local_sum;
  __syncthreads();

  __half2 sum = __float2half2_rn(0.f);
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int x = tid >> 7;
  int y = tid & (127);
  if (y < 32) {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      sum += tile[x][y + i * 32];
    }
  }
  __syncthreads();

  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (y == 0) tile[0][x] = sum;
  __syncthreads();

  if (threadIdx.x < 8) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, 8);
    bias_grad2[pos] = tile[0][threadIdx.x];
  }
}

template <typename T>
void launch_ls_dropout_bias_bwd(T *in_grad, T *bias_grad, const T *out_grad,
                                const uint8_t *mask, int row_size, int dim,
                                float ratio, hipStream_t stream) {
  dim3 grid_dim((dim - 1) / 8 + 1);
  dim3 block_dim(8, 128);
  ls_dropout_bias_bwd_kernel<<<grid_dim, block_dim, 0, stream>>>(
      row_size, ratio, in_grad, bias_grad, out_grad, mask, dim);
}

template <>
void launch_ls_dropout_bias_bwd(__half *in_grad, __half *bias_grad,
                                const __half *out_grad, const uint8_t *mask,
                                int row_size, int dim, float ratio,
                                hipStream_t stream) {
  dim >>= 1;
  dim3 grid_dim((dim - 1) / 8 + 1);
  dim3 block_dim(8, 128);
  ls_dropout_bias_bwd_kernel<<<grid_dim, block_dim, 0, stream>>>(
      row_size, ratio, in_grad, bias_grad, out_grad, mask, dim);
}

template void launch_ls_dropout_bias_bwd(float *in_grad, float *bias_grad,
                                         const float *out_grad,
                                         const uint8_t *mask, int row_size,
                                         int dim, float ratio,
                                         hipStream_t stream);

/**
 * @brief fused bias, activation, and dropout at the end of first ffn
 *
 * @thread
 * gridDim.x = hidden_size / 8
 * blockDim.x = 8
 * blockDim.y = 1024 / 8 = 128
 *
 * @tparam act_type activation function, like kRelu, kGelu
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param seed seed to hiprand
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type>
__global__ void ls_dropout_act_bias_kernel(
    const int total_count, const float ratio, float *__restrict__ out,
    const float *__restrict__ in, uint8_t *__restrict__ mask,
    const float *__restrict__ bias, const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *data4 = reinterpret_cast<const float4 *>(in);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);

  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] = m4[0];
  const float4 input4 = data4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  float4 output4;

  output4.x =
      activation_kernel<act_type, float>(input4.x + b4.x) * scale * m[0];
  output4.y =
      activation_kernel<act_type, float>(input4.y + b4.y) * scale * m[1];
  output4.z =
      activation_kernel<act_type, float>(input4.z + b4.z) * scale * m[2];
  output4.w =
      activation_kernel<act_type, float>(input4.w + b4.w) * scale * m[3];

  out4[i] = output4;
}

template <ActivationType act_type>
__global__ void ls_dropout_act_bias_kernel(
    const int total_count, const float ratio, __half *__restrict__ out,
    const __half *__restrict__ in, uint8_t *__restrict__ mask,
    const __half *__restrict__ bias, const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = (uint8_t)(rand.x > ratio);
  m[5] = (uint8_t)(rand.y > ratio);
  m[6] = (uint8_t)(rand.z > ratio);
  m[7] = (uint8_t)(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] = *m8;

  int bias_i = i % (hidden_size >> 3);
  float4 val_float4 = vals_float4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  float4 out_float4;

  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);

  __half2 scale_mask_1 = __floats2half2_rn(scale * m[0], scale * m[1]);
  __half2 scale_mask_2 = __floats2half2_rn(scale * m[2], scale * m[3]);
  __half2 scale_mask_3 = __floats2half2_rn(scale * m[4], scale * m[5]);
  __half2 scale_mask_4 = __floats2half2_rn(scale * m[6], scale * m[7]);
  out_half2[0] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[0], b_half2[0])),
      scale_mask_1);
  out_half2[1] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[1], b_half2[1])),
      scale_mask_2);
  out_half2[2] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[2], b_half2[2])),
      scale_mask_3);
  out_half2[3] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[3], b_half2[3])),
      scale_mask_4);
  outs_float4[i] = out_float4;
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kGelu, float>(
    float *out, const float *vals, uint8_t *mask, const float *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 10;
  ls_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kGelu, __half>(
    __half *out, const __half *vals, uint8_t *mask, const __half *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 11;
  ls_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kRelu, float>(
    float *out, const float *vals, uint8_t *mask, const float *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 10;
  ls_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kRelu, __half>(
    __half *out, const __half *vals, uint8_t *mask, const __half *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 11;
  ls_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

/**
 * @brief fused bias, activation, and dropout backward
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @tparam act_type kRelu
 * @param row_size batch_size * seq_len
 * @param ratio dropout ratio
 * @param in_grad [batch_size, seq_len, hidden_size], input grad
 * @param bias_grad [hidden_size], bias grad
 * @param out_grad [batch_size, seq_len, hidden_size], output grad
 * @param mask [batch_size, seq_len, hidden_size], dropout mask
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type, typename T>
__global__ void ls_dropout_act_bias_bwd_kernel(
    const int row_size, const float ratio, T *in_grad,
    T *__restrict__ bias_grad, const T *__restrict__ input,
    const T *__restrict__ bias, const T *out_grad,
    const uint8_t *__restrict__ mask, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  __shared__ float tile[WARP_SIZE][WARP_SIZE + 1];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);

  int stride = hidden_size * WARP_SIZE;
  float local_sum = 0;

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
  if (col_idx < hidden_size) {
    for (int r = threadIdx.y; r < row_size; r += WARP_SIZE) {
      float val = out_grad[idx];
      float in = input[idx];
      float b = bias[idx % hidden_size];
      val = activation_bwd_kernel<act_type, float>(
          val * scale * static_cast<float>(mask[idx]), in + b);
      local_sum += val;
      in_grad[idx] = val;
      idx += stride;
    }
  }

  tile[threadIdx.x][threadIdx.y] = local_sum;
  __syncthreads();
  float sum = tile[threadIdx.y][threadIdx.x];
  __syncthreads();

  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (threadIdx.x == 0) tile[0][threadIdx.y] = sum;
  __syncthreads();

  if (threadIdx.y == 0) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
    bias_grad[pos] = tile[0][threadIdx.x];
  }
}

// @brief fused bias, activation, and dropout backward
// It is deprecated for precision reason. Keep it for future optimization.
//
// template <ActivationType act_type>
// __global__ void ls_dropout_act_bias_bwd_kernel(
//     const int row_size, const float ratio, __half * in_grad,
//     __half *__restrict__ bias_grad, const __half *__restrict__ input, const
//     __half *__restrict__ bias, const __half * out_grad, const uint8_t
//     *__restrict__ mask, const int hidden_size) {
//   const __half2 scale = __float2half2_rn(1.f / (1.f - ratio));
//   __shared__ __half2 tile[WARP_SIZE][WARP_SIZE + 1];

//   cg::thread_block b = cg::this_thread_block();
//   cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

//   __half2 *in_grad2 = reinterpret_cast<__half2 *>(in_grad);
//   __half2 *bias_grad2 = reinterpret_cast<__half2 *>(bias_grad);
//   const __half2 *out_grad2 = reinterpret_cast<const __half2 *>(out_grad);
//   const __half2 *input2 = reinterpret_cast<const __half2 *>(input);
//   const __half2 *bias2 = reinterpret_cast<const __half2 *>(bias);

//   int col_idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);

//   int stride = hidden_size * WARP_SIZE;
//   __half2 local_sum = __float2half2_rn(0.f);

//   int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
//   if (col_idx < hidden_size) {
//     for (int r = threadIdx.y; r < row_size; r += WARP_SIZE) {
//       __half2 val = out_grad2[idx];
//       __half2 in2 = input2[idx];
//       __half2 b2 = bias2[idx % hidden_size ];
//       __half2 m2 = __floats2half2_rn(mask[2 * idx], mask[2 * idx + 1]);
//       val = activation_bwd_kernel<ActivationType::kRelu, __half2>(val * scale
//       *
//                                                                   m2,
//                                                                   in2+b2);
//       local_sum += val;
//       in_grad2[idx] = val;
//       idx += stride;
//     }
//   }

//   tile[threadIdx.x][threadIdx.y] = local_sum;
//   __syncthreads();
//   __half2 sum = tile[threadIdx.y][threadIdx.x];
//   __syncthreads();

//   for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

//   if (threadIdx.x == 0) tile[0][threadIdx.y] = sum;
//   __syncthreads();

//   if (threadIdx.y == 0) {
//     int pos = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
//     bias_grad2[pos] = tile[0][threadIdx.x];
//   }
// }

template <ActivationType act_type, typename T>
void launch_ls_dropout_act_bias_bwd(T *in_grad, T *bias_grad, const T *input,
                                    const T *bias, const T *out_grad,
                                    const uint8_t *mask, int row_size, int dim,
                                    float ratio, hipStream_t stream) {
  dim3 grid_dim((dim - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);
  ls_dropout_act_bias_bwd_kernel<act_type><<<grid_dim, block_dim, 0, stream>>>(
      row_size, ratio, in_grad, bias_grad, input, bias, out_grad, mask, dim);
}

// template <>
// void launch_ls_dropout_act_bias_bwd<ActivationType::kRelu, __half>(
//     __half *in_grad, __half *bias_grad,const __half *input, const __half
//     *bias, const __half *out_grad, const uint8_t *mask, int row_size, int
//     dim, float ratio, hipStream_t stream) {
//   dim >>= 1;
//   dim3 grid_dim((dim - 1) / WARP_SIZE + 1);
//   dim3 block_dim(WARP_SIZE, WARP_SIZE);
//   ls_dropout_act_bias_bwd_kernel<ActivationType::kRelu>
//       <<<grid_dim, block_dim, 0, stream>>>(row_size, ratio, in_grad,
//       bias_grad,
//                                            input, bias,out_grad, mask, dim);
// }

template void launch_ls_dropout_act_bias_bwd<ActivationType::kRelu, float>(
    float *in_grad, float *bias_grad, const float *input, const float *bias,
    const float *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void launch_ls_dropout_act_bias_bwd<ActivationType::kRelu, __half>(
    __half *in_grad, __half *bias_grad, const __half *input, const __half *bias,
    const __half *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void launch_ls_dropout_act_bias_bwd<ActivationType::kGelu, float>(
    float *in_grad, float *bias_grad, const float *input, const float *bias,
    const float *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void launch_ls_dropout_act_bias_bwd<ActivationType::kGelu, __half>(
    __half *in_grad, __half *bias_grad, const __half *input, const __half *bias,
    const __half *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

#include "hip/hip_runtime.h"
#include "kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

/**
@brief: fuse_transpose_bias
Calculate the sum of elements in each column of the matrix.

@thread
gridDim.x = ceil(cols / WARP_SIZE)
blockDim.x = WARP_SIZE
blockDim.y = WARP_SIZE

@param
inp: [rows, cols]
out: [cols]
rows: the number of rows in the matrix
cols: the number of cols in the matrix
*/
template <typename T>
__global__ void column_sum_reduce(const T *__restrict__ inp,
                                  T *__restrict__ out, int rows, int cols) {
  __shared__ float tile[WARP_SIZE][WARP_SIZE];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
  int y_stride = cols * WARP_SIZE;
  float localSum = 0;

  // Loop across matrix row
  // TODO: optimize to log complexity
  if (idx < cols) {
    int offset = flat_2dim(threadIdx.y, idx, cols);
    for (int r = threadIdx.y; r < rows; r += WARP_SIZE) {
      localSum += (float)inp[offset];
      offset += y_stride;
    }
  }

  // The sum of a row in tile is equal to the sum of a col in original matrix
  tile[threadIdx.x][threadIdx.y] = localSum;

  __syncthreads();

  // Sum the shared buffer.
  // The change of threadIdx.x is continuous
  float sum = tile[threadIdx.y][threadIdx.x];

  __syncthreads();

  // Calculate the sum of a row in tile
  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (threadIdx.x == 0) {
    int pos = flat_2dim(blockIdx.x, threadIdx.y, WARP_SIZE);
    if (pos < cols) out[pos] = sum;
  }
}

// [r, c] -> [c]
template <>
void launch_fuse_transpose_bias_kernel<float>(const float *inp, float *out,
                                              int rows, int cols,
                                              hipStream_t stream) {
  dim3 grid_dim((cols - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);

  column_sum_reduce<float>
      <<<grid_dim, block_dim, 0, stream>>>(inp, out, rows, cols);
}

template <>
void launch_fuse_transpose_bias_kernel<__half>(const __half *inp, __half *out,
                                               int rows, int cols,
                                               hipStream_t stream) {
  dim3 grid_dim((cols - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);

  column_sum_reduce<__half>
      <<<grid_dim, block_dim, 0, stream>>>(inp, out, rows, cols);
}

/**
@brief: fused_add2
Add two matrix inp1 and inp2 to out.

@thread
gridDim.x = batch_size * seq_len
blockDim.x = min(hidden_dim, MAX_THREADS)

@param
inp1: [batch_size, seq_len, hidden_dim]
inp2: [batch_size, seq_len, hidden_dim]
out: [batch_size, seq_len, hidden_dim]
batch_size: the size of the current batch
seq_len: the sequence length of the current batch
hidden_dim: dim of the hidden tensor
*/
template <typename T>
__global__ void fused_add2_kernel(T *out, const T *inp1, const T *inp2,
                                  int hidden_dim);

template <>
__global__ void fused_add2_kernel<float>(float *out, const float *inp1,
                                         const float *inp2, int hidden_dim) {
  int row_id = blockIdx.x;
  int offset = flat_2dim(row_id, 0, hidden_dim);

  const float4 *inp1_4 = reinterpret_cast<const float4 *>(inp1);
  const float4 *inp2_4 = reinterpret_cast<const float4 *>(inp2);
  float4 *out_4 = reinterpret_cast<float4 *>(out);
  float4 vinp1;
  float4 vinp2;
  float4 val;

  for (std::size_t i = threadIdx.x; i < hidden_dim; i += blockDim.x) {
    vinp1 = inp1_4[offset + i];
    vinp2 = inp2_4[offset + i];
    val.x = vinp1.x + vinp2.x;
    val.y = vinp1.y + vinp2.y;
    val.z = vinp1.z + vinp2.z;
    val.w = vinp1.w + vinp2.w;
    out_4[offset + i] = val;
  }
}

template <>
__global__ void fused_add2_kernel<__half>(__half *out, const __half *inp1,
                                          const __half *inp2, int hidden_dim) {
  int row_id = blockIdx.x;
  int offset = flat_2dim(row_id, 0, hidden_dim);

  const float4 *inp1_4 = reinterpret_cast<const float4 *>(inp1);
  const float4 *inp2_4 = reinterpret_cast<const float4 *>(inp2);
  float4 *out_4 = reinterpret_cast<float4 *>(out);
  float4 vinp1;
  float4 vinp2;
  float4 val;
  __half2 *h2_inp1 = reinterpret_cast<__half2 *>(&vinp1);
  __half2 *h2_inp2 = reinterpret_cast<__half2 *>(&vinp2);
  __half2 *h2_val = reinterpret_cast<__half2 *>(&val);

  for (std::size_t i = threadIdx.x; i < hidden_dim; i += blockDim.x) {
    vinp1 = inp1_4[offset + i];
    vinp2 = inp2_4[offset + i];
    h2_val[0] = __hadd2(h2_inp1[0], h2_inp2[0]);
    h2_val[1] = __hadd2(h2_inp1[1], h2_inp2[1]);
    h2_val[2] = __hadd2(h2_inp1[2], h2_inp2[2]);
    h2_val[3] = __hadd2(h2_inp1[3], h2_inp2[3]);
    out_4[offset + i] = val;
  }
}

//[b, s, h] -> [b, s, h]
template <>
void launch_fused_add2<float>(float *out, const float *inp1, const float *inp2,
                              int batch_size, int seq_len, int hidden_dim,
                              hipStream_t &stream) {
  hidden_dim >>= 2;

  dim3 grid_dim(batch_size * seq_len);
  dim3 block_dim(min(hidden_dim, MAX_THREADS));

  fused_add2_kernel<<<grid_dim, block_dim, 0, stream>>>(out, inp1, inp2,
                                                        hidden_dim);
}

template <>
void launch_fused_add2<__half>(__half *out, const __half *inp1,
                               const __half *inp2, int batch_size, int seq_len,
                               int hidden_dim, hipStream_t &stream) {
  hidden_dim >>= 3;

  dim3 grid_dim(batch_size * seq_len);
  dim3 block_dim(min(hidden_dim, MAX_THREADS));

  fused_add2_kernel<<<grid_dim, block_dim, 0, stream>>>(out, inp1, inp2,
                                                        hidden_dim);
}

template <typename T>
__global__ void kernel_concat3_dim1(const T *inp1, const T *inp2, T *output,
                                    int sz0, int sz2, int sz1_1, int sz1_2) {
  int nele = sz0 * sz2 * (sz1_1 + sz1_2);
  int idx = flat_2dim(blockIdx.x, threadIdx.x, blockDim.x);
  if (idx >= nele) {
    return;
  }
  float4 *dst_ptr = (float4 *)output + idx;
  int idx2 = idx % sz2;
  idx = idx / sz2;
  int idx1 = idx % (sz1_1 + sz1_2);
  int idx0 = idx / (sz1_1 + sz1_2);
  float4 *src_ptr = nullptr;
  int sz1 = 0;
  if (idx1 < sz1_1) {
    sz1 = sz1_1;
    src_ptr = (float4 *)inp1;
  } else {
    idx1 -= sz1_1;
    sz1 = sz1_2;
    src_ptr = (float4 *)inp2;
  }
  src_ptr += flat_3dim(idx0, idx1, idx2, sz1, sz2);
  dst_ptr[0] = src_ptr[0];
}

template <>
void launch_concat3_dim1<float>(const float *inp1, const float *inp2,
                                float *output, int sz0, int sz2, int sz1_1,
                                int sz1_2, hipStream_t stream) {
  sz2 >>= 2;
  int nele = sz0 * sz2 * (sz1_1 + sz1_2);
  int nblock = (nele + MAX_THREADS - 1) / MAX_THREADS;
  kernel_concat3_dim1<<<nblock, MAX_THREADS, 0, stream>>>(
      inp1, inp2, output, sz0, sz2, sz1_1, sz1_2);
}

template <>
void launch_concat3_dim1<__half>(const __half *inp1, const __half *inp2,
                                 __half *output, int sz0, int sz2, int sz1_1,
                                 int sz1_2, hipStream_t stream) {
  sz2 >>= 3;
  int nele = sz0 * sz2 * (sz1_1 + sz1_2);
  int nblock = (nele + MAX_THREADS - 1) / MAX_THREADS;
  kernel_concat3_dim1<<<nblock, MAX_THREADS, 0, stream>>>(
      inp1, inp2, output, sz0, sz2, sz1_1, sz1_2);
}

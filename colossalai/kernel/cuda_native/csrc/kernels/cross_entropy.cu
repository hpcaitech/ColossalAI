#include "hip/hip_runtime.h"
#include "block_reduce.h"
#include "cuda_util.h"
#include "kernels.h"
#include "ls_cub.cuh"

ls::hipcub::CachingDeviceAllocator g_allocator(true);

template <typename T>
__global__ void ls_cross_entropy_fw_kernel(
    const T *__restrict__ inputs, const int *__restrict__ targets,
    float *__restrict__ outputs, float *__restrict__ nll_loss_outputs,
    const int padding_idx, const float epsilon, const int vocab_size) {
  /* step1: compute each thread's max_logit and sum_exp_logit, store in
   * max_input, sum_exp_logit */
  const int block_start = blockIdx.x * vocab_size;
  const int left_idx = block_start + threadIdx.x;
  const int right_idx = (blockIdx.x + 1) * vocab_size;
  float max_input[1] = {REDUCE_FLOAT_INF_NEG};
  float sum_logits[2] = {0.f, 0.f};  // logit and logit exp
  int target_tid = targets[blockIdx.x];

  if (target_tid == padding_idx) {
    if (threadIdx.x == 0) {
      nll_loss_outputs[blockIdx.x] = 0.f;
      outputs[blockIdx.x] = 0.f;
    }
    return;
  }

  for (int i = left_idx; i < right_idx; i += blockDim.x) {
    max_input[0] = fmaxf(max_input[0], static_cast<float>(inputs[i]));
  }
  blockReduce<ReduceType::kMax, 1>(max_input);
  __shared__ float s_max_input;
  if (threadIdx.x == 0) {
    s_max_input = max_input[0];
  }
  __syncthreads();

  for (int i = left_idx; i < right_idx; i += blockDim.x) {
    float logit = static_cast<float>(inputs[i]) - s_max_input;
    sum_logits[0] += logit;
    sum_logits[1] += expf(logit);
  }

  blockReduce<ReduceType::kSum, 2>(sum_logits);
  __shared__ float s_sum_logit;
  __shared__ float s_sum_exp;
  if (threadIdx.x == 0) {
    s_sum_logit = sum_logits[0];
    s_sum_exp = sum_logits[1];
  }
  __syncthreads();

  float eps_i = epsilon / (vocab_size - 1);
  if (threadIdx.x == 0) {
    // neg_log_prob = log(sum(exp(x - x_max))) - (x - x_max)
    float nll_loss = logf(s_sum_exp) -
                     static_cast<float>(inputs[block_start + target_tid]) +
                     s_max_input;
    nll_loss_outputs[blockIdx.x] = nll_loss;
    float sum_nll_loss = vocab_size * logf(s_sum_exp) - s_sum_logit;
    outputs[blockIdx.x] =
        (1.f - epsilon - eps_i) * nll_loss + eps_i * sum_nll_loss;
  }
}

template <typename T>
__global__ void ls_cross_entropy_bw_kernel(
    const float *__restrict__ grad_outputs, const T *__restrict__ inputs,
    const int *__restrict__ targets, T *__restrict__ grad_inputs,
    const int padding_idx, const float epsilon, const int vocab_size) {
  /* step1: compute each thread's max_logit and sum_exp_logit, store in
   * max_input, sum_exp_logit */
  const int block_start = blockIdx.x * vocab_size;
  const int left_idx = block_start + threadIdx.x;
  const int right_idx = (blockIdx.x + 1) * vocab_size;
  float max_input[1] = {REDUCE_FLOAT_INF_NEG};
  float sum_logits[1] = {0.f};
  const float grad_out = static_cast<float>(grad_outputs[0]);
  int target_tid = targets[blockIdx.x];

  if (target_tid == padding_idx) {
    for (int i = left_idx; i < right_idx; i += blockDim.x) {
      grad_inputs[i] = 0.f;
    }
    return;
  }

  for (int i = left_idx; i < right_idx; i += blockDim.x) {
    max_input[0] = fmaxf(max_input[0], static_cast<float>(inputs[i]));
  }
  blockReduce<ReduceType::kMax, 1>(max_input);
  __shared__ float s_max_input;
  if (threadIdx.x == 0) {
    s_max_input = max_input[0];
  }
  __syncthreads();

  for (int i = left_idx; i < right_idx; i += blockDim.x) {
    float logit = static_cast<float>(inputs[i]) - s_max_input;
    sum_logits[0] += expf(logit);
  }

  blockReduce<ReduceType::kSum, 1>(sum_logits);
  __shared__ float s_sum_exp;
  if (threadIdx.x == 0) {
    s_sum_exp = sum_logits[0];
  }
  __syncthreads();

  float eps_i = epsilon / (vocab_size - 1);
  float nll_weight = 1.0 - epsilon - eps_i;

  for (int i = left_idx; i < right_idx; i += blockDim.x) {
    float prob = expf(static_cast<float>(inputs[i]) - s_max_input) / s_sum_exp;
    float grad = 0;
    grad += (vocab_size * prob - 1) * eps_i;
    grad += prob * nll_weight;
    if ((i - block_start) == target_tid) {
      grad -= nll_weight;
    }
    grad_inputs[i] = grad_out * grad;
  }
}

template <typename T>
void launch_cross_entropy_fw(const T *inputs_ptr, const int *targets_ptr,
                             float *outputs_ptr, float *nll_loss_ptr,
                             float *loss_buffer, const int padding_idx,
                             const float epsilon, const int batch_size,
                             const int seq_len, const int vocab_size,
                             hipStream_t stream) {
  int grid_dim = batch_size * seq_len;
  float *nll_loss_buffer = loss_buffer + grid_dim;
  ls_cross_entropy_fw_kernel<<<grid_dim, MAX_THREADS, 0, stream>>>(
      inputs_ptr, targets_ptr, loss_buffer, nll_loss_buffer, padding_idx,
      epsilon, vocab_size);

  int num_items = grid_dim;
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CHECK_GPU_ERROR(ls::hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                                             loss_buffer, outputs_ptr,
                                             num_items, stream));
  CHECK_GPU_ERROR(
      g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
  CHECK_GPU_ERROR(ls::hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                                             loss_buffer, outputs_ptr,
                                             num_items, stream));
  CHECK_GPU_ERROR(ls::hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                                             nll_loss_buffer, nll_loss_ptr,
                                             num_items, stream));
  CHECK_GPU_ERROR(g_allocator.DeviceFree(d_temp_storage));
}

template void launch_cross_entropy_fw<float>(
    const float *inputs_ptr, const int *targets_ptr, float *outputs_ptr,
    float *nll_loss_ptr, float *loss_buffer, const int padding_idx,
    const float epsilon, const int batch_size, const int seq_len,
    const int vocab_size, hipStream_t stream);

template void launch_cross_entropy_fw<__half>(
    const __half *inputs_ptr, const int *targets_ptr, float *outputs_ptr,
    float *nll_loss_ptr, float *loss_buffer, const int padding_idx,
    const float epsilon, const int batch_size, const int seq_len,
    const int vocab_size, hipStream_t stream);

template <typename T>
void launch_cross_entropy_bw(const float *grad_outputs_ptr, const T *inputs_ptr,
                             const int *targets_ptr, T *grad_inputs_ptr,
                             const int padding_idx, const float epsilon,
                             const int batch_size, const int seq_len,
                             const int vocab_size, hipStream_t stream) {
  int grid_dim = batch_size * seq_len;
  ls_cross_entropy_bw_kernel<<<grid_dim, MAX_THREADS, 0, stream>>>(
      grad_outputs_ptr, inputs_ptr, targets_ptr, grad_inputs_ptr, padding_idx,
      epsilon, vocab_size);
}

template void launch_cross_entropy_bw<float>(
    const float *grad_outputs_ptr, const float *inputs_ptr,
    const int *targets_ptr, float *grad_inputs_ptr, const int padding_idx,
    const float epsilon, const int batch_size, const int seq_len,
    const int vocab_size, hipStream_t stream);

template void launch_cross_entropy_bw<__half>(
    const float *grad_outputs_ptr, const __half *inputs_ptr,
    const int *targets_ptr, __half *grad_inputs_ptr, const int padding_idx,
    const float epsilon, const int batch_size, const int seq_len,
    const int vocab_size, hipStream_t stream);

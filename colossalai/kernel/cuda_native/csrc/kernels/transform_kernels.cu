#include "hip/hip_runtime.h"
#include <cub/block/block_load.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/block/block_store.cuh>

#include "kernels.h"

using namespace hipcub;

/**
@brief: transform_0213
Split the attention heads and reshape input
during backward progress of encoder self-attention

@thread
gridDim.x = batch_size
gridDim.y = seq_len
blockDim.x = min(hidden_dim, MAX_THREADS)

@param
input: [batch_size, seq_len, hidden_dim]
output: [batch_size, nhead, seq_len, head_dim]
batch_size: the size of the current batch
seq_len: the sequence length of the current batch
hidden_dim: dim of the hidden tensor
nhead: number of attention heads
*/

template <typename T>
__global__ void transform_0213(T *output, const T *input, int hidden_dim,
                               int head_dim);

template <>
__global__ void transform_0213<float>(float *output, const float *input,
                                      int hidden_dim, int head_dim) {
  int batch_id = blockIdx.x;
  int token_id = blockIdx.y;
  int seq_len = gridDim.y;
  int nhead = hidden_dim / head_dim;

  // [b, s, h]
  int src_offset = flat_3dim(batch_id, token_id, 0, seq_len, hidden_dim);
  // [b, nh, s, ad]
  int trg_offset =
      flat_4dim(batch_id, 0, token_id, 0, nhead, seq_len, head_dim);

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  float4 *res4 = reinterpret_cast<float4 *>(output);
  float4 vinput4;

  for (std::size_t i = threadIdx.x; i < hidden_dim; i += blockDim.x) {
    vinput4 = input4[src_offset + i];

    int head_id = i / head_dim;
    int dim_id = i % head_dim;
    int cur_trg_offset = flat_3dim(head_id, 0, dim_id, seq_len, head_dim);
    res4[trg_offset + cur_trg_offset] = vinput4;
  }
}

template <>
__global__ void transform_0213<__half>(__half *output, const __half *input,
                                       int hidden_dim, int head_dim) {
  int batch_id = blockIdx.x;
  int token_id = blockIdx.y;
  int seq_len = gridDim.y;
  int nhead = hidden_dim / head_dim;

  // [b, s, h]
  int src_offset = flat_3dim(batch_id, token_id, 0, seq_len, hidden_dim);
  // [b, nh, s, ad]
  int trg_offset =
      flat_4dim(batch_id, 0, token_id, 0, nhead, seq_len, head_dim);

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  float4 *res4 = reinterpret_cast<float4 *>(output);
  float4 vinput4;

  for (std::size_t i = threadIdx.x; i < hidden_dim; i += blockDim.x) {
    vinput4 = input4[src_offset + i];

    int head_id = i / head_dim;
    int dim_id = i % head_dim;
    int cur_trg_offset = flat_3dim(head_id, 0, dim_id, seq_len, head_dim);
    res4[trg_offset + cur_trg_offset] = vinput4;
  }
}

// [b, s, h] -> [b, nh, s, ad]
template <>
void launch_transform_0213<float>(float *output, const float *input,
                                  int batch_size, int seq_len, int hidden_dim,
                                  int nhead, hipStream_t stream) {
  hidden_dim >>= 2;
  int head_dim = hidden_dim / nhead;

  dim3 grid_dim(batch_size, seq_len);
  dim3 block_dim(min(hidden_dim, MAX_THREADS));

  transform_0213<float>
      <<<grid_dim, block_dim, 0, stream>>>(output, input, hidden_dim, head_dim);
}

template <>
void launch_transform_0213<__half>(__half *output, const __half *input,
                                   int batch_size, int seq_len, int hidden_dim,
                                   int nhead, hipStream_t stream) {
  hidden_dim >>= 3;
  int head_dim = hidden_dim / nhead;

  dim3 grid_dim(batch_size, seq_len);
  dim3 block_dim(min(hidden_dim, MAX_THREADS));

  transform_0213<__half>
      <<<grid_dim, block_dim, 0, stream>>>(output, input, hidden_dim, head_dim);
}

/**
@brief: bias_add_transform_20314
Add bias to input, transform from
[0, 1, 2, 3, 4] to [2, 0, 3, 1, 4]

@thread
gridDim.x = dim_0
gridDim.y = dim_1
gridDim.z = dim_2
blockDim.x = min(dim_3 * dim_4, MAX_THREADS)

@param
input: [dim_0, dim_1, dim_2, dim_3, dim_4]
bias: [dim_2, dim_3, dim_4]
output: [dim_2, dim_0, dim_3, dim_1, dim_4]
*/
template <typename T>
__global__ void bias_add_transform_20314(T *output, const T *input,
                                         const T *bias, int dim_3, int dim_4);

template <>
__global__ void bias_add_transform_20314<float>(float *output,
                                                const float *input,
                                                const float *bias, int dim_3,
                                                int dim_4) {
  int id0 = blockIdx.x;
  int id1 = blockIdx.y;
  int id2 = blockIdx.z;
  int dim_0 = gridDim.x;
  int dim_1 = gridDim.y;
  int dim_2 = gridDim.z;
  int dim_34 = dim_3 * dim_4;

  int src_offset = flat_4dim(id0, id1, id2, 0, dim_1, dim_2, dim_34);
  int trg_offset = flat_5dim(id2, id0, 0, id1, 0, dim_0, dim_3, dim_1, dim_4);
  int bias_offset = flat_2dim(id2, 0, dim_34);

  const float4 *qkv4 = reinterpret_cast<const float4 *>(input);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  float4 *res4 = reinterpret_cast<float4 *>(output);
  float4 vqkv4;
  float4 vbias4;
  float4 vres4;

  for (std::size_t i = threadIdx.x; i < dim_34; i += blockDim.x) {
    vqkv4 = qkv4[src_offset + i];
    vbias4 = bias4[bias_offset + i];
    vres4.x = vqkv4.x + vbias4.x;
    vres4.y = vqkv4.y + vbias4.y;
    vres4.z = vqkv4.z + vbias4.z;
    vres4.w = vqkv4.w + vbias4.w;

    int id3 = i / dim_4;
    int id4 = i % dim_4;
    int cur_trg_offset = flat_3dim(id3, 0, id4, dim_1, dim_4);
    res4[trg_offset + cur_trg_offset] = vres4;
  }
}

template <>
__global__ void bias_add_transform_20314<__half>(__half *output,
                                                 const __half *input,
                                                 const __half *bias, int dim_3,
                                                 int dim_4) {
  int id0 = blockIdx.x;
  int id1 = blockIdx.y;
  int id2 = blockIdx.z;
  int dim_0 = gridDim.x;
  int dim_1 = gridDim.y;
  int dim_2 = gridDim.z;
  int dim_34 = dim_3 * dim_4;

  int src_offset = flat_4dim(id0, id1, id2, 0, dim_1, dim_2, dim_34);
  int trg_offset = flat_5dim(id2, id0, 0, id1, 0, dim_0, dim_3, dim_1, dim_4);
  int bias_offset = flat_2dim(id2, 0, dim_34);

  const float4 *qkv4 = reinterpret_cast<const float4 *>(input);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  float4 *res4 = reinterpret_cast<float4 *>(output);
  float4 vqkv4;
  float4 vbias4;
  float4 vres4;
  __half2 *h2_qkv = reinterpret_cast<__half2 *>(&vqkv4);
  __half2 *h2_bias = reinterpret_cast<__half2 *>(&vbias4);
  __half2 *h2_res = reinterpret_cast<__half2 *>(&vres4);

  for (std::size_t i = threadIdx.x; i < dim_34; i += blockDim.x) {
    vqkv4 = qkv4[src_offset + i];
    vbias4 = bias4[bias_offset + i];
    h2_res[0] = __hadd2(h2_qkv[0], h2_bias[0]);
    h2_res[1] = __hadd2(h2_qkv[1], h2_bias[1]);
    h2_res[2] = __hadd2(h2_qkv[2], h2_bias[2]);
    h2_res[3] = __hadd2(h2_qkv[3], h2_bias[3]);

    int id3 = i / dim_4;
    int id4 = i % dim_4;
    int cur_trg_offset = flat_3dim(id3, 0, id4, dim_1, dim_4);
    res4[trg_offset + cur_trg_offset] = vres4;
  }
}

// [b, s, 3, h] -> [3, b, nh, s, ad]
template <>
void launch_bias_add_transform_20314<float>(float *output, const float *input,
                                            const float *bias, int dim_0,
                                            int dim_1, int dim_2, int dim_3,
                                            int dim_4, hipStream_t stream) {
  dim_4 >>= 2;

  dim3 grid_dim(dim_0, dim_1, dim_2);
  dim3 block_dim(min(dim_3 * dim_4, MAX_THREADS));

  bias_add_transform_20314<float>
      <<<grid_dim, block_dim, 0, stream>>>(output, input, bias, dim_3, dim_4);
}

template <>
void launch_bias_add_transform_20314<__half>(__half *output,
                                             const __half *input,
                                             const __half *bias, int dim_0,
                                             int dim_1, int dim_2, int dim_3,
                                             int dim_4, hipStream_t stream) {
  dim_4 >>= 3;

  dim3 grid_dim(dim_0, dim_1, dim_2);
  dim3 block_dim(min(dim_3 * dim_4, MAX_THREADS));

  bias_add_transform_20314<__half>
      <<<grid_dim, block_dim, 0, stream>>>(output, input, bias, dim_3, dim_4);
}

/**
@brief: transform4d_0213
Reshape the input matrix to merge the heads

@thread
gridDim.x = (num_all + max_block_thread - 1) / max_block_thread
blockDim.x = max_block_thread

@param
input: [trans_count, batch_size, nhead, seq_len, head_dim]
output: [batch_size, seq_len, trans_count, nhead, head_dim]
batch_size: the size of the current batch
seq_len: the sequence length of the current batch
hidden_dim: dim of the hidden tensor
nhead: number of attention heads
trans_count: 1 or 3, the count of matrice need to be transformed
*/
template <typename T>
__global__ void transform4d_0213(T *output, const T *input, int batch_size,
                                 int seq_len, int trans_count, int nhead,
                                 int head_dim, int num_all) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= num_all) {
    return;
  }
  int trans_id, batch_id, head_id, token_id, dim_id;
  decompose_5dim(offset, batch_size, nhead, seq_len, head_dim, &trans_id,
                 &batch_id, &head_id, &token_id, &dim_id);
  // [b, s, tc, nh, ad]
  int trg_offset = flat_5dim(batch_id, token_id, trans_id, head_id, dim_id,
                             seq_len, trans_count, nhead, head_dim);

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  float4 *res4 = reinterpret_cast<float4 *>(output);
  res4[trg_offset] = input4[offset];
}

// [tc, b, nh, s, ad] -> [b, s, tc, nh, ad]
template <>
void launch_transform4d_0213<float>(float *output, const float *input,
                                    int batch_size, int seq_len, int hidden_dim,
                                    int nhead, int trans_count,
                                    hipStream_t stream) {
  hidden_dim >>= 2;
  int head_dim = hidden_dim / nhead;
  int num_all = batch_size * seq_len * trans_count * hidden_dim;
  int nblock = (num_all + MAX_THREADS - 1) / MAX_THREADS;

  transform4d_0213<float><<<nblock, MAX_THREADS, 0, stream>>>(
      output, input, batch_size, seq_len, trans_count, nhead, head_dim,
      num_all);
}

template <>
void launch_transform4d_0213<__half>(__half *output, const __half *input,
                                     int batch_size, int seq_len,
                                     int hidden_dim, int nhead, int trans_count,
                                     hipStream_t stream) {
  hidden_dim >>= 3;
  int head_dim = hidden_dim / nhead;
  int num_all = batch_size * seq_len * trans_count * hidden_dim;
  int nblock = (num_all + MAX_THREADS - 1) / MAX_THREADS;

  transform4d_0213<__half><<<nblock, MAX_THREADS, 0, stream>>>(
      output, input, batch_size, seq_len, trans_count, nhead, head_dim,
      num_all);
}

#include "hip/hip_runtime.h"
/* Copyright 2021 The LightSeq Team
   Copyright Microsoft DeepSpeed
   This file is adapted from Microsoft DeepSpeed
   Licensed under the MIT License.
*/
#include "cublas_wrappers.h"

int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transa,
                   hipblasOperation_t transb, int m, int n, int k,
                   const float *alpha, const float *beta, const float *A,
                   const float *B, float *C, hipblasGemmAlgo_t algo) {
  hipblasStatus_t status =
      hipblasGemmEx(handle, transa, transb, m, n, k, (const void *)alpha,
                   (const void *)A, HIP_R_32F, (transa == HIPBLAS_OP_N) ? m : k,
                   (const void *)B, HIP_R_32F, (transb == HIPBLAS_OP_N) ? k : n,
                   (const void *)beta, C, HIP_R_32F, m, HIP_R_32F, algo);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
            m, n, k, (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transa,
                   hipblasOperation_t transb, int m, int n, int k,
                   const float *alpha, const float *beta, const __half *A,
                   const __half *B, __half *C, hipblasGemmAlgo_t algo) {
  hipblasStatus_t status = hipblasGemmEx(
      handle, transa, transb, m, n, k, (const void *)alpha, (const void *)A,
      HIP_R_16F, (transa == HIPBLAS_OP_N) ? m : k, (const void *)B, HIP_R_16F,
      (transb == HIPBLAS_OP_N) ? k : n, (const void *)beta, (void *)C,
      HIP_R_16F, m, HIP_R_32F, algo);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
            m, n, k, (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_strided_batched_gemm(hipblasHandle_t handle, int m, int n, int k,
                                const float *alpha, const float *beta,
                                const float *A, const float *B, float *C,
                                hipblasOperation_t op_A, hipblasOperation_t op_B,
                                int stride_A, int stride_B, int stride_C,
                                int batch, hipblasGemmAlgo_t algo) {
  hipblasStatus_t status = hipblasGemmStridedBatchedEx(
      handle, op_A, op_B, m, n, k, alpha, A, HIP_R_32F,
      (op_A == HIPBLAS_OP_N) ? m : k, stride_A, B, HIP_R_32F,
      (op_B == HIPBLAS_OP_N) ? k : n, stride_B, beta, C, HIP_R_32F, m, stride_C,
      batch, HIP_R_32F, algo);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (batch: %d, m: %d, n: %d, k: %d, "
            "error: %d) \n",
            batch, m, n, k, (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_strided_batched_gemm(hipblasHandle_t handle, int m, int n, int k,
                                const float *alpha, const float *beta,
                                const __half *A, const __half *B, __half *C,
                                hipblasOperation_t op_A, hipblasOperation_t op_B,
                                int stride_A, int stride_B, int stride_C,
                                int batch, hipblasGemmAlgo_t algo) {
  hipblasStatus_t status = hipblasGemmStridedBatchedEx(
      handle, op_A, op_B, m, n, k, alpha, A, HIP_R_16F,
      (op_A == HIPBLAS_OP_N) ? m : k, stride_A, B, HIP_R_16F,
      (op_B == HIPBLAS_OP_N) ? k : n, stride_B, beta, C, HIP_R_16F, m, stride_C,
      batch, HIP_R_32F, algo);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
            m, n, k, (int)status);
    return EXIT_FAILURE;
  }

  return 0;
}

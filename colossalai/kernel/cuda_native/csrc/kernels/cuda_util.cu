#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "cuda_util.h"

/* GPU function guard */
std::string _cudaGetErrorString(hipError_t error) {
  return hipGetErrorString(error);
}

std::string _cudaGetErrorString(hipblasStatus_t error) {
  switch (error) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";

    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";

    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";

    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";

    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";

    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";

    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";

    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";

    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";

    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "CUBLAS_UNKNOW";
}

template <typename T>
void check_gpu_error(T result, char const *const func, const char *const file,
                     int const line) {
  if (result) {
    throw std::runtime_error(std::string("[CUDA][ERROR] ") + +file + "(" +
                             std::to_string(line) +
                             "): " + (_cudaGetErrorString(result)) + "\n");
  }
}

template void check_gpu_error<hipError_t>(hipError_t result,
                                           char const *const func,
                                           const char *const file,
                                           int const line);
template void check_gpu_error<hipblasStatus_t>(hipblasStatus_t result,
                                              char const *const func,
                                              const char *const file,
                                              int const line);

template <typename T>
void print_vec(const T *outv, std::string outn, int num_output_ele) {
  std::cout << outn << ": ";
  std::vector<T> hout(num_output_ele, (T)0);
  hipMemcpy(hout.data(), outv, num_output_ele * sizeof(T),
             hipMemcpyDeviceToHost);
  for (int i = 0; i < num_output_ele; i++) {
    std::cout << hout[i] << ", ";
  }
  std::cout << std::endl;
}

template <>
void print_vec<__half>(const __half *outv, std::string outn,
                       int num_output_ele) {
  std::cout << outn << ": ";
  std::vector<__half> hout(num_output_ele, (__half)0.f);
  hipMemcpy(hout.data(), outv, num_output_ele * sizeof(__half),
             hipMemcpyDeviceToHost);
  for (int i = 0; i < num_output_ele; i++) {
    std::cout << __half2float(hout[i]) << ", ";
  }
  std::cout << std::endl;
}

template void print_vec<float>(const float *outv, std::string outn,
                               int num_output_ele);

template void print_vec<int>(const int *outv, std::string outn,
                             int num_output_ele);

template void print_vec<__half>(const __half *outv, std::string outn,
                                int num_output_ele);

template <typename T>
T *cuda_malloc(size_t ele_num) {
  size_t byte_size = ele_num * sizeof(T);
  T *pdata = nullptr;
  CHECK_GPU_ERROR(hipMalloc((void **)&pdata, byte_size));
  return pdata;
}

template float *cuda_malloc<float>(size_t ele_num);

template __half *cuda_malloc<__half>(size_t ele_num);

template uint8_t *cuda_malloc<uint8_t>(size_t ele_num);

void cuda_free(void *pdata) {
  if (pdata != nullptr) {
    hipFree(pdata);
  }
}

template <typename T>
struct _isnan {
  __device__ bool operator()(T a) const { return isnan(a); }
};

template <>
struct _isnan<__half> {
  __device__ bool operator()(const __half a) const { return __hisnan(a); }
};

template <typename T>
struct _isinf {
  __device__ bool operator()(T a) const { return isinf(a); }
};

template <>
struct _isinf<__half> {
  __device__ bool operator()(const __half a) const { return __hisinf(a); }
};

template <typename T>
void check_nan_inf(const T *data_ptr, int dsize, bool check_nan_inf,
                   std::string file, int line, hipStream_t stream) {
  // check_nan_inf = 0 for checking nan
  // check_nan_inf = 1 for checking inf
  bool res = false;
  std::string msg = file + "(" + std::to_string(line) + "): ";
  if (check_nan_inf) {
    msg += "nan.";
    res = thrust::transform_reduce(thrust::cuda::par.on(stream), data_ptr,
                                   data_ptr + dsize, _isnan<T>(), false,
                                   thrust::logical_or<bool>());
  } else {
    msg += "inf.";
    res = thrust::transform_reduce(thrust::cuda::par.on(stream), data_ptr,
                                   data_ptr + dsize, _isinf<T>(), false,
                                   thrust::logical_or<bool>());
  }
  if (res) {
    throw std::runtime_error(msg);
  }
  std::cout << msg << " [check pass]." << std::endl;
}

template void check_nan_inf<float>(const float *data_ptr, int dsize,
                                   bool check_nan_inf, std::string file,
                                   int line, hipStream_t stream);

template void check_nan_inf<__half>(const __half *data_ptr, int dsize,
                                    bool check_nan_inf, std::string file,
                                    int line, hipStream_t stream);

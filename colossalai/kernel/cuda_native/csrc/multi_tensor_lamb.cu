#include "hip/hip_runtime.h"
// modified from
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_lamb.cu
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "multi_tensor_apply.cuh"
#include "type_shim.h"

#define BLOCK_SIZE 512
#define ILP 4

template <typename T>
__device__ __forceinline__ bool is_aligned(T *p) {
  return ((uint64_t)p) % (ILP * sizeof(T)) == 0;
}

template <typename T>
__device__ __forceinline__ void load_store(T *dst, T *src, int dst_offset,
                                           int src_offset) {
  typedef
      typename std::aligned_storage<ILP * sizeof(T), ILP * alignof(T)>::type LT;
  ((LT *)dst)[dst_offset] = ((LT *)src)[src_offset];
}

typedef enum {
  MOMENT_MODE_0 = 0,  // L2 regularization mode
  MOMENT_MODE_1 = 1   // Decoupled weight decay mode
} adamMode_t;

std::tuple<at::Tensor, at::Tensor> multi_tensor_l2norm_cuda(
    int chunk_size, at::Tensor noop_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists,
    at::optional<bool> per_tensor_python);

using MATH_T = float;

template <typename T>
struct LAMBStage1Functor {
  __device__ __forceinline__ void operator()(
      int chunk_size, volatile int *noop_gmem, TensorListMetadata<4> &tl,
      const float beta1, const float beta2, const float beta3,
      const float beta1_correction, const float beta2_correction,
      const float epsilon, adamMode_t mode, const float decay,
      const float *global_grad_norm, const float max_global_grad_norm) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float clipped_global_grad_norm =
        (*global_grad_norm) > max_global_grad_norm
            ? (*global_grad_norm) / max_global_grad_norm
            : 1.0f;

    T *g = (T *)tl.addresses[0][tensor_loc];
    g += chunk_idx * chunk_size;

    T *p = (T *)tl.addresses[1][tensor_loc];
    p += chunk_idx * chunk_size;

    T *m = (T *)tl.addresses[2][tensor_loc];
    m += chunk_idx * chunk_size;

    T *v = (T *)tl.addresses[3][tensor_loc];
    v += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    MATH_T r_g[ILP];
    MATH_T r_p[ILP];
    MATH_T r_m[ILP];
    MATH_T r_v[ILP];
    // to make things simple, we put aligned case in a different code path
    if (n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(g) &&
        is_aligned(p) && is_aligned(m) && is_aligned(v)) {
      T l_g[ILP];
      T l_p[ILP];
      T l_m[ILP];
      T l_v[ILP];
      for (int i_start = threadIdx.x;
           i_start * ILP < n && i_start * ILP < chunk_size;
           i_start += blockDim.x) {
        // load
        load_store(l_g, g, 0, i_start);
        if (decay != 0) load_store(l_p, p, 0, i_start);
        load_store(l_m, m, 0, i_start);
        load_store(l_v, v, 0, i_start);
        // unpack
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          r_g[ii] = l_g[ii];
          if (decay == 0) {
            r_p[ii] = MATH_T(0);
          } else {
            r_p[ii] = l_p[ii];
          }
          r_m[ii] = l_m[ii];
          r_v[ii] = l_v[ii];
        }
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay * r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1 - beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          } else {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1 - beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased / denom) + (decay * r_p[ii]);
          }
        }
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          l_p[ii] = r_p[ii];
          l_m[ii] = r_m[ii];
          l_v[ii] = r_v[ii];
        }
        // store
        load_store(g, l_p, i_start, 0);
        load_store(m, l_m, i_start, 0);
        load_store(v, l_v, i_start, 0);
      }
    } else {
      // see note in multi_tensor_scale_kernel.cu
      for (int i_start = 0; i_start < n && i_start < chunk_size;
           i_start += blockDim.x * ILP) {
        MATH_T r_g[ILP];
        MATH_T r_p[ILP];
        MATH_T r_m[ILP];
        MATH_T r_v[ILP];
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            r_g[ii] = g[i];
            // special ?optimization? for lamb stage 1
            if (decay == 0) {
              r_p[ii] = MATH_T(0);
            } else {
              r_p[ii] = p[i];
            }
            r_m[ii] = m[i];
            r_v[ii] = v[i];
          } else {
            r_g[ii] = MATH_T(0);
            r_p[ii] = MATH_T(0);
            r_m[ii] = MATH_T(0);
            r_v[ii] = MATH_T(0);
          }
        }
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay * r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1 - beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          } else {
            MATH_T scaled_grad = r_g[ii] / clipped_global_grad_norm;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1 - beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased / denom) + (decay * r_p[ii]);
          }
        }
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            g[i] = r_p[ii];
            m[i] = r_m[ii];
            v[i] = r_v[ii];
          }
        }
      }
    }
  }
};

// Step 2 reads in 'update' value and per-tensor param_norm and update_norm.
// It computes new parameter value.
template <typename T>
struct LAMBStage2Functor {
  __device__ __forceinline__ void operator()(
      int chunk_size, volatile int *noop_gmem, TensorListMetadata<2> &tl,
      const float *per_tensor_param_norm, const float *per_tensor_update_norm,
      const float learning_rate, const float decay, bool use_nvlamb) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    MATH_T ratio = learning_rate;
    // nvlamb: apply adaptive learning rate to all parameters
    // otherwise, only apply to those with non-zero weight decay
    if (use_nvlamb || (decay != 0.0)) {
      float param_norm = per_tensor_param_norm[tensor_num];
      float update_norm = per_tensor_update_norm[tensor_num];
      ratio = (update_norm != 0.0f && param_norm != 0.0f)
                  ? learning_rate * (param_norm / update_norm)
                  : learning_rate;
    }

    T *update = (T *)tl.addresses[0][tensor_loc];
    update += chunk_idx * chunk_size;

    T *p = (T *)tl.addresses[1][tensor_loc];
    p += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // to make things simple, we put aligned case in a different code path
    if (n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(p) &&
        is_aligned(update)) {
      T r_p[ILP];
      T r_update[ILP];
      for (int i_start = threadIdx.x;
           i_start * ILP < n && i_start * ILP < chunk_size;
           i_start += blockDim.x) {
        // load
        load_store(r_p, p, 0, i_start);
        load_store(r_update, update, 0, i_start);
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          r_p[ii] = static_cast<MATH_T>(r_p[ii]) -
                    (ratio * static_cast<MATH_T>(r_update[ii]));
        }
        load_store(p, r_p, i_start, 0);
      }
    } else {
      for (int i_start = 0; i_start < n && i_start < chunk_size;
           i_start += blockDim.x * ILP) {
        MATH_T r_p[ILP];
        MATH_T r_update[ILP];
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            r_p[ii] = p[i];
            r_update[ii] = update[i];
          }
        }
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          r_p[ii] = r_p[ii] - (ratio * r_update[ii]);
        }
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            p[i] = r_p[ii];
          }
        }
      }
    }
  }
};

void multi_tensor_lamb_cuda(int chunk_size, at::Tensor noop_flag,
                            std::vector<std::vector<at::Tensor>> tensor_lists,
                            const float lr, const float beta1,
                            const float beta2, const float epsilon,
                            const int step, const int bias_correction,
                            const float weight_decay, const int grad_averaging,
                            const int mode, at::Tensor global_grad_norm,
                            const float max_grad_norm,
                            at::optional<bool> use_nvlamb_python) {
  using namespace at;
  // Master weight and 32bit momentum(potentially changing) is not handled by
  // this So we assume every tensor are all in the same type

  bool use_nvlamb =
      use_nvlamb_python.has_value() ? use_nvlamb_python.value() : false;

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  // Handle grad averaging mode
  float beta3 = 1.0f;
  if (grad_averaging == 1) beta3 = 1 - beta1;

  std::vector<std::vector<at::Tensor>> grad_list(tensor_lists.begin(),
                                                 tensor_lists.begin() + 1);
  std::vector<std::vector<at::Tensor>> param_list(tensor_lists.begin() + 1,
                                                  tensor_lists.begin() + 2);

  // Compute per tensor param norm
  auto param_norm_tuple =
      multi_tensor_l2norm_cuda(chunk_size, noop_flag, param_list, true);

  // We now in-place modify grad to store update before compute its norm
  // Generally this is not a issue since people modify grad in step() method all
  // the time We can also grab list of empty tensor to avoid this, but I'd like
  // to save space/cpu code
  DISPATCH_FLOAT_AND_HALF(
      tensor_lists[0][0].scalar_type(), 0, "lamb_stage_1",
      multi_tensor_apply<4>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            LAMBStage1Functor<scalar_t_0>(), beta1, beta2,
                            beta3,  // 1-beta1 or 1 depends on averaging mode
                            bias_correction1, bias_correction2, epsilon,
                            (adamMode_t)mode, weight_decay,
                            global_grad_norm.DATA_PTR<float>(), max_grad_norm);)

  // Compute update norms
  auto update_norm_tuple =
      multi_tensor_l2norm_cuda(chunk_size, noop_flag, grad_list, true);

  std::vector<std::vector<at::Tensor>> grad_param_list(
      tensor_lists.begin(), tensor_lists.begin() + 2);

  DISPATCH_FLOAT_AND_HALF(
      tensor_lists[0][0].scalar_type(), 0, "lamb_stage_2",
      multi_tensor_apply<2>(BLOCK_SIZE, chunk_size, noop_flag, grad_param_list,
                            LAMBStage2Functor<scalar_t_0>(),
                            std::get<1>(param_norm_tuple).DATA_PTR<float>(),
                            std::get<1>(update_norm_tuple).DATA_PTR<float>(),
                            lr, weight_decay, use_nvlamb);)

  AT_CUDA_CHECK(hipGetLastError());
}

#include "hip/hip_runtime.h"
// modified from
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_sgd_kernel.cu
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "compat.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

/**
 * Perform fused SGD on multiple buffers
 * N: number of tensors
 * tl[0] : gradients
 * tl[1] : weights
 * tl[2] : momentum buffers
 * tl[3] : fp16 weights (if appropriate)
 * wd : weight_decay (scalar)
 * momentum : momentum (scalar)
 * dampening : momentum dampening (scalar)
 * lr : learning rate (scalar)
 * nesterov : enable nesterov (bool)
 * first run : necessary for proper momentum handling & init
 * wd_after_momentum : apply weight decay _after_ momentum instead of before
 **/
template <typename T_grad, typename T_weight>
struct SGDFunctor {
  __device__ __forceinline__ void operator()(
      int chunk_size, volatile int *noop_gmem, TensorListMetadata<3> &tl,
      float wd, float momentum, float dampening, float lr, bool nesterov,
      bool first_run, bool wd_after_momentum, float scale) {
    // Early exit if we don't need to do anything
    if (*noop_gmem) return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T_grad *grad_in = (T_grad *)tl.addresses[0][tensor_loc];
    grad_in += chunk_idx * chunk_size;

    T_weight *weight_in = (T_weight *)tl.addresses[1][tensor_loc];
    weight_in += chunk_idx * chunk_size;

    T_weight *mom_in = (T_weight *)tl.addresses[2][tensor_loc];
    mom_in += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // Non-divergent exit condition for the __syncthreads
    float incoming_grads[ILP];
    float incoming_weights[ILP];
    float incoming_moms[ILP];
    for (int i_start = 0; i_start < n && i_start < chunk_size;
         i_start += blockDim.x * ILP) {
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        incoming_grads[ii] = 0;
        incoming_weights[ii] = 0;
        incoming_moms[ii] = 0;
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          incoming_grads[ii] = static_cast<float>(grad_in[i]) * scale;
          incoming_weights[ii] = static_cast<float>(weight_in[i]);
          incoming_moms[ii] = static_cast<float>(mom_in[i]);
        }
      }

// note for clarification to future michael:
// From a pure memory dependency perspective, there's likely no point unrolling
// the write loop, since writes just fire off once their LDGs arrive.
// Put another way, the STGs are dependent on the LDGs, but not on each other.
// There is still compute ILP benefit from unrolling the loop though.
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          // apply weight decay before momentum if necessary
          if (wd != 0.f && !wd_after_momentum)
            incoming_grads[ii] += wd * incoming_weights[ii];

          if (momentum != 0.f) {
            if (!first_run)
              incoming_moms[ii] = incoming_moms[ii] * momentum +
                                  (1.f - dampening) * incoming_grads[ii];
            else  // initialize momentums to current incoming grads
              incoming_moms[ii] = incoming_grads[ii];

            if (nesterov)
              incoming_grads[ii] += momentum * incoming_moms[ii];
            else
              incoming_grads[ii] = incoming_moms[ii];
          }

          // Apply WD after momentum if desired
          if (wd != 0.f && wd_after_momentum)
            incoming_grads[ii] += wd * incoming_weights[ii];

          // adjust the weight and write out
          weight_in[i] += (-lr * incoming_grads[ii]);

          // also write out the new momentum
          if (momentum != 0.f) mom_in[i] = incoming_moms[ii];
        }
      }
    }
  }
};

void multi_tensor_sgd_cuda(int chunk_size, at::Tensor noop_flag,
                           std::vector<std::vector<at::Tensor>> tensor_lists,
                           float wd, float momentum, float dampening, float lr,
                           bool nesterov, bool first_run,
                           bool wd_after_momentum, float scale) {
  auto num_tensors = tensor_lists.size();
  auto grad_type = tensor_lists[0][0].scalar_type();
  auto weight_type = tensor_lists[1][0].scalar_type();

  TORCH_CHECK(noop_flag.device() == tensor_lists[0][0].device(),
              "expected noop flag to be on the same device as tensors");

  // We have 3 possibilities to handle here, in terms of
  // grad_type, param_type, momentum_type
  // 1. fp16, fp16, fp16
  // 2. fp32, fp32, fp32
  // 3. fp16, fp32, fp32
  // It's easier to hardcode these possibilities than to use
  // switches etc. to handle the cross-product of cases where
  // we don't want the majority of them.

  // Case 1. fp16, fp16, fp16, No
  if (grad_type == at::ScalarType::Half &&
      weight_type == at::ScalarType::Half && num_tensors == 3) {
    multi_tensor_apply<3>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                          SGDFunctor<at::Half, at::Half>(), wd, momentum,
                          dampening, lr, nesterov, first_run, wd_after_momentum,
                          scale);
  }
  // Case 2. fp32, fp32, fp32
  else if (grad_type == at::ScalarType::Float &&
           weight_type == at::ScalarType::Float && num_tensors == 3) {
    multi_tensor_apply<3>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                          SGDFunctor<float, float>(), wd, momentum, dampening,
                          lr, nesterov, first_run, wd_after_momentum, scale);
  }
  // Case 3. fp16, fp32, fp32
  else if (grad_type == at::ScalarType::Half &&
           weight_type == at::ScalarType::Float && num_tensors == 3) {
    multi_tensor_apply<3>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                          SGDFunctor<at::Half, float>(), wd, momentum,
                          dampening, lr, nesterov, first_run, wd_after_momentum,
                          scale);
  } else {
    AT_ERROR(
        "multi_tensor_sgd only supports some combinations of gradient & weight "
        "types. Given: ",
        "gradient: ", grad_type, ", weight: ", weight_type,
        ", num_lists: ", num_tensors);
  }

  AT_CUDA_CHECK(hipGetLastError());
}
